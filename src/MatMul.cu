#include "hip/hip_runtime.h"
#include "CKKS/LinearTransform.cuh"
#include "MatMul.cuh"

namespace FIDESlib::CKKS {

#if LOW_MEM

void CCMMSquare_GPU(FIDESlib::CKKS::Ciphertext& cMat1, FIDESlib::CKKS::Ciphertext& cMat2, uint32_t rowSize,
                    FIDESlib::CKKS::Ciphertext& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp) {

    // Carlos A.D. TODO: Equalize levels (+ noise degree) of inputs to reduce unnecessary complexity in computations

    FIDESlib::CKKS::Context& cc = cMat1.cc;
    // Step 1-1: Linear transform for first matrix
    FIDESlib::CKKS::Ciphertext linearTransform1(cc);

#if BSGS
    linearTransform1.copy(cMat1);

    // std::cout << "#1: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;
    LinearTransform(linearTransform1, 2 * rowSize - 1, precomp.bStep, precomp.pts_1, 1, -(int)rowSize + 1);
    // std::cout << "#1: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;

#else
    linearTransform1.multPt(cMat1, precomp.sigmaPlaintexts[0][0]);

    FIDESlib::CKKS::Ciphertext prev_rotation(cc);
    FIDESlib::CKKS::Ciphertext productCt(cc);

    prev_rotation.copy(cMat1);
    for (size_t i = 1; i < rowSize; i++) {
        prev_rotation.rotate(1, cc.GetRotationKey(1));

        productCt.multPt(prev_rotation, precomp.sigmaPlaintexts[0][i]);
        linearTransform1.add(productCt);
    }

    prev_rotation.copy(cMat1);
    for (size_t i = 1; i < rowSize; i++) {
        prev_rotation.rotate(-1, cc.GetRotationKey(-1));

        productCt.multPt(prev_rotation, precomp.sigmaPlaintexts[1][i - 1]);
        linearTransform1.add(productCt);
    }
#endif

    // Step 1-2: Linear transform for second matrix
    FIDESlib::CKKS::Ciphertext linearTransform2(cc);

#if BSGS
    linearTransform2.copy(cMat2);

    // std::cout << "#2: " << linearTransform2.getLevel() << ", " << linearTransform2.NoiseLevel << std::endl;
    LinearTransform(linearTransform2, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0);
    // std::cout << "#2: " << linearTransform2.getLevel() << ", " << linearTransform2.NoiseLevel << std::endl;
#else
    linearTransform2.multPt(cMat2, precomp.tauPlaintexts[0]);

    prev_rotation.copy(cMat2);

    for (size_t i = 1; i < rowSize; i++) {
        prev_rotation.rotate(rowSize, cc.GetRotationKey(rowSize));

        productCt.multPt(prev_rotation, precomp.tauPlaintexts[i /* * rowSize */]);
        linearTransform2.add(productCt);
    }
#endif

    // cProduct.copy(linearTransform1);
    //
    // Steps 2 and 3: Initial computation

#if BSGS
    Ciphertext aux(cc);
    aux.rotate(linearTransform1, -(int)rowSize /*+ 1*/, cc.GetRotationKey(-(int)rowSize /*+ 1*/));
    LinearTransformSpecial(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1,
                           precomp.pts_3_2, 1, rowSize);

    // std::cout << "#3: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;

    cProduct.copy(linearTransform1);
#else
    cProduct.mult(linearTransform1, linearTransform2, cc.GetEvalKey());
    prev_rotation.copy(linearTransform1);
    FIDESlib::CKKS::Ciphertext prev_rotation_minus_row(cc);
    prev_rotation_minus_row.rotate(prev_rotation, -(int)rowSize, cc.GetRotationKey(-(int)rowSize));
    FIDESlib::CKKS::Ciphertext productCt2(cc);

    FIDESlib::CKKS::Ciphertext prev_rotation2(cc);
    prev_rotation2.copy(linearTransform2);

    for (size_t i = 1; i < rowSize; i++) {
        // Step 2
        prev_rotation.rotate(1, linearTransform1.cc.GetRotationKey(1));

        productCt.multPt(prev_rotation, precomp.phiPlaintexts[i][0]);

        prev_rotation_minus_row.rotate(1, linearTransform1.cc.GetRotationKey(1));

        productCt2.multPt(prev_rotation_minus_row, precomp.phiPlaintexts[i][1]);

        productCt.add(productCt2);

        prev_rotation2.rotate(rowSize, cc.GetRotationKey(rowSize));

        // Step 3
        productCt.mult(prev_rotation2, cc.GetEvalKey());
        cProduct.add(productCt);
    }

#endif
}

#else

void CCMMSquare_GPU(FIDESlib::CKKS::Ciphertext& cMat1, FIDESlib::CKKS::Ciphertext& cMat2, uint32_t rowSize,
                    FIDESlib::CKKS::Ciphertext& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp) {

    // Step 1-1: Linear transform for first matrix
    FIDESlib::CKKS::Ciphertext linearTransform1(cMat1.cc);

    for (size_t i = 1; i < rowSize; i++) {
        FIDESlib::CKKS::Ciphertext rotatedCt(cMat1.cc);
        rotatedCt.copy(cMat1);
        rotatedCt.rotate(i, cMat1.cc.GetRotationKey(i));

        FIDESlib::CKKS::Ciphertext productCt(rotatedCt.cc);
        productCt.multPt(rotatedCt, precomp.sigmaPlaintexts[0][i]);
        linearTransform1.add(productCt);

        rotatedCt.copy(cMat1);
        rotatedCt.rotate(-i, cMat1.cc.GetRotationKey(-i));

        productCt.multPt(rotatedCt, precomp.sigmaPlaintexts[0][rowSize * rowSize - i]);
        linearTransform1.add(productCt);
    }

    // Step 1-2: Linear transform for second matrix
    FIDESlib::CKKS::Ciphertext linearTransform2(cMat2.cc);
    linearTransform2.multPt(cMat2, precomp.tauPlaintexts[0]);

    for (size_t i = 1; i < rowSize; i++) {
        FIDESlib::CKKS::Ciphertext rotatedCt(cMat2.cc);
        rotatedCt.copy(cMat2);
        rotatedCt.rotate(i * rowSize, cMat2.cc.GetRotationKey(i * rowSize));

        FIDESlib::CKKS::Ciphertext productCt(rotatedCt.cc);
        productCt.multPt(rotatedCt, precomp.tauPlaintexts[i * rowSize]);
        linearTransform2.add(productCt);
    }

    // Steps 2 and 3: Initial computation
    cProduct.mult(linearTransform1, linearTransform2, linearTransform2.cc.GetEvalKey());

    for (size_t i = 1; i < rowSize; i++) {
        // Step 2
        FIDESlib::CKKS::Ciphertext rotatedCt(linearTransform1.cc);
        rotatedCt.copy(linearTransform1);
        rotatedCt.rotate(i, linearTransform1.cc.GetRotationKey(i));

        FIDESlib::CKKS::Ciphertext productCt1(rotatedCt.cc);
        productCt1.multPt(rotatedCt, precomp.phiPlaintexts[i][0]);

        rotatedCt.copy(linearTransform1);
        rotatedCt.rotate(i - rowSize, linearTransform1.cc.GetRotationKey(i - rowSize));

        FIDESlib::CKKS::Ciphertext productCt2(rotatedCt.cc);
        productCt2.multPt(rotatedCt, precomp.phiPlaintexts[i][1]);

        FIDESlib::CKKS::Ciphertext linearTransformPhi(productCt1.cc);
        linearTransformPhi.add(productCt1, productCt2);

        FIDESlib::CKKS::Ciphertext linearTransformPsi(linearTransform2.cc);
        linearTransformPsi.copy(linearTransform2);
        linearTransformPsi.rotate(i * rowSize, linearTransform2.cc.GetRotationKey(i * rowSize));

        // Step 3
        FIDESlib::CKKS::Ciphertext tempProduct(linearTransformPhi.cc);
        tempProduct.mult(linearTransformPhi, linearTransformPsi, linearTransformPsi.cc.GetEvalKey());
        cProduct.add(tempProduct);
    }
}

#endif

// matrix multiplication
void CCMM_GPU(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp) {

    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());

        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            CCMMSquare_GPU(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                CCMMSquare_GPU(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }
}

void CCMMSquare_GPU_masked(FIDESlib::CKKS::Ciphertext& cMat1, FIDESlib::CKKS::Ciphertext& cMat2, uint32_t rowSize,
                    FIDESlib::CKKS::Ciphertext& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp, Plaintext& mask_pt) {

    FIDESlib::CKKS::Context& cc = cMat1.cc;
    // Step 1-1: Linear transform for first matrix
    FIDESlib::CKKS::Ciphertext linearTransform1(cc);

    linearTransform1.copy(cMat1);

    // std::cout << "#1: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;
    LinearTransform(linearTransform1, 2 * rowSize - 1, precomp.bStep, precomp.pts_1, 1, -(int)rowSize + 1);
    // std::cout << "#1: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;

    // Step 1-2: Linear transform for second matrix
    FIDESlib::CKKS::Ciphertext linearTransform2(cc);
    linearTransform2.copy(cMat2);

    // std::cout << "#2: " << linearTransform2.getLevel() << ", " << linearTransform2.NoiseLevel << std::endl;
    LinearTransform(linearTransform2, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0);
    // std::cout << "#2: " << linearTransform2.getLevel() << ", " << linearTransform2.NoiseLevel << std::endl;

    // Steps 2 and 3: Initial computation
    Ciphertext aux(cc);
    aux.rotate(linearTransform1, -(int)rowSize /*+ 1*/, cc.GetRotationKey(-(int)rowSize /*+ 1*/));
    LinearTransformSpecial(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1,
                           precomp.pts_3_2, 1, rowSize, mask_pt);

    // std::cout << "#3: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;

    cProduct.copy(linearTransform1);

}

// masked matrix multiplication
void CCMM_GPU_masked(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp, Plaintext& mask_pt) {
    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> cProduct;
    cProduct.reserve(matrix1.size());
    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());

        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            CCMMSquare_GPU_masked(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp, mask_pt);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                CCMMSquare_GPU_masked(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp, mask_pt);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        cProduct.emplace_back(std::move(row));
    }
    product.emplace_back(std::move(cProduct));
}



// double masking, in and out
void CCMMSquare_GPU_double_mask(FIDESlib::CKKS::Ciphertext& cMat1, FIDESlib::CKKS::Ciphertext& cMat2, uint32_t rowSize,
                    FIDESlib::CKKS::Ciphertext& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp, 
                    PtMasks_GPU& masks, bool head_no, int token_length, bool if_transpose) {

    FIDESlib::CKKS::Context& cc = cMat1.cc;
    // Step 1-1: Linear transform for first matrix
    FIDESlib::CKKS::Ciphertext linearTransform1(cc);
    linearTransform1.copy(cMat1);

    // std::cout << "#1: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;
    LinearTransform(linearTransform1, 2 * rowSize - 1, precomp.bStep, precomp.pts_1, 1, -(int)rowSize + 1, masks.head_masks[head_no]);
    // std::cout << "#1: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;

    // Step 1-2: Linear transform for second matrix
    FIDESlib::CKKS::Ciphertext linearTransform2(cc);
    linearTransform2.copy(cMat2);

    // std::cout << "#2: " << linearTransform2.getLevel() << ", " << linearTransform2.NoiseLevel << std::endl;
    // LinearTransform(linearTransform2, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0, mask_pt);

    if (if_transpose) {
        LinearTransform(linearTransform2, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0);
    }
    else {
        LinearTransform(linearTransform2, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0, masks.head_masks[head_no]);
    }

    // std::cout << "#2: " << linearTransform2.getLevel() << ", " << linearTransform2.NoiseLevel << std::endl;

    // Steps 2 and 3: Initial computation
    Ciphertext aux(cc);
    aux.rotate(linearTransform1, -(int)rowSize /*+ 1*/, cc.GetRotationKey(-(int)rowSize /*+ 1*/));
    // LinearTransformSpecial(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1,
    //                        precomp.pts_3_2, 1, rowSize);

    Plaintext masks_double(cc);
    masks_double.copy(masks.head_masks[0]);
    masks_double.multPt(masks_double, masks.row_masks[token_length], true);

    LinearTransformSpecial(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1,
                           precomp.pts_3_2, 1, rowSize, masks_double);

    // std::cout << "#3: " << linearTransform1.getLevel() << ", " << linearTransform1.NoiseLevel << std::endl;
    
    cProduct.copy(linearTransform1);

}

// masked matrix multiplication
void CCMM_GPU_double_mask(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp, PtMasks_GPU& masks, bool head_no, int token_length, bool if_transpose) {
    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> cProduct;
    cProduct.reserve(matrix1.size());
    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());

        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            CCMMSquare_GPU_double_mask(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp, masks, head_no, token_length, if_transpose);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                CCMMSquare_GPU_double_mask(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp, masks, head_no, token_length, if_transpose);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        cProduct.emplace_back(std::move(row));
    }
    product.emplace_back(std::move(cProduct));
}


void PCMMSquare_GPU(FIDESlib::CKKS::Ciphertext& cMat1, const FIDESlib::CKKS::Plaintext& pMat2, uint32_t rowSize,
                    FIDESlib::CKKS::Ciphertext& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp) {

    FIDESlib::CKKS::Context& cc = cMat1.cc;
    // Step 1-1: Linear transform for first matrix
    FIDESlib::CKKS::Ciphertext linearTransform1(cc);
    linearTransform1.copy(cMat1);
    LinearTransform(linearTransform1, 2 * rowSize - 1, precomp.bStep, precomp.pts_1, 1, -(int)rowSize + 1);

    // Step 1-2: Linear transform for second matrix
    FIDESlib::CKKS::Plaintext linearTransform2(cc);
    linearTransform2.copy(pMat2);

    //LinearTransformPt(linearTransform2, cc, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0);

    // Steps 2 and 3: Initial computation
    Ciphertext aux(cc);
    aux.rotate(linearTransform1, -(int)rowSize /*+ 1*/, cc.GetRotationKey(-(int)rowSize /*+ 1*/));
    LinearTransformSpecialPt(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1,
                             precomp.pts_3_2, 1,
                             rowSize);  // LT2 is now plaintext
    if (linearTransform1.NoiseLevel == 2)
        linearTransform1.rescale();
    // linearTransform1.dropToLevel(cMat1.getLevel() - 3);
    cProduct.copy(linearTransform1);
}

// PCMM with bias
void PCMM_GPU(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias) {
    product.clear();
    product.reserve(matrix1.size());
    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());
        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            PCMMSquare_GPU(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);
            dotProd.addPt(bias[0][j]);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                PCMMSquare_GPU(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }

}


// PCMM with (masked) bias
void PCMM_GPU(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias,
              Plaintext& mask_row) {
    product.clear();
    product.reserve(matrix1.size());
    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());
        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            PCMMSquare_GPU(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);

            Plaintext bias_masked(matrix1[i][0].cc);
            bias_masked.copy(bias[0][j]);
            if (mask_row.NoiseLevel > 1) { mask_row.rescale(); }
            bias_masked.multPt(bias_masked, mask_row, true);

            dotProd.addPt(bias_masked);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                PCMMSquare_GPU(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }

}



// void PCMMSquare_GPU_masked(FIDESlib::CKKS::Ciphertext& cMat1, const FIDESlib::CKKS::Plaintext& pMat2, uint32_t rowSize,
//                     std::vector<FIDESlib::CKKS::Ciphertext>& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp, std::vector<Plaintext>& mask_pt) {

//     FIDESlib::CKKS::Context& cc = cMat1.cc;
//     // Step 1-1: Linear transform for first matrix
//     FIDESlib::CKKS::Ciphertext linearTransform1(cc);
//     linearTransform1.copy(cMat1);
//     LinearTransform(linearTransform1, 2 * rowSize - 1, precomp.bStep, precomp.pts_1, 1, -(int)rowSize + 1);

//     // Step 1-2: Linear transform for second matrix
//     FIDESlib::CKKS::Plaintext linearTransform2(cc);
//     linearTransform2.copy(pMat2);

//     //LinearTransformPt(linearTransform2, cc, rowSize, precomp.bStep, precomp.pts_2, rowSize, 0);

//     // Steps 2 and 3: Initial computation
//     Ciphertext aux(cc);
//     aux.rotate(linearTransform1, -(int)rowSize /*+ 1*/, cc.GetRotationKey(-(int)rowSize /*+ 1*/));
    
    
//     std::vector<FIDESlib::CKKS::Ciphertext> results;
//     results.reserve(mask_pt.size());
//     LinearTransformSpecialPt(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1,
//                              precomp.pts_3_2, 1, rowSize, mask_pt, results);  

//     for (int i = 0; i < results.size(); i++){
//         if (results[i].NoiseLevel == 2)
//             results[i].rescale();
//     }

//     // linearTransform1.dropToLevel(cMat1.getLevel() - 3);
//     cProduct.reserve(results.size());
//     for (int i = 0; i < results.size(); i++){
//         cProduct.emplace_back(std::move(results[i]));
//     }
// }

// // PCMM with bias, masked
// void PCMM_GPU_masked(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
//               std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
//               std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>>& product,
//               const MatrixMatrixProductPrecomputations_GPU& precomp,
//               std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias, std::vector<Plaintext>& mask_pt) {

//     product.reserve(matrix1.size());

//     for (size_t i = 0; i < matrix1.size(); i++) {
//         std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> row;
//         row.reserve(matrix2[0].size());
//         for (size_t j = 0; j < matrix2[0].size(); j++) {
//             std::vector<FIDESlib::CKKS::Ciphertext> dotProd;
//             PCMMSquare_GPU_masked(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp, mask_pt);            
//             for (size_t k = 1; k < matrix2.size(); k++) {
//                 std::vector<FIDESlib::CKKS::Ciphertext> dotProdNew;
//                 PCMMSquare_GPU_masked(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp, mask_pt);
//                 for (int m = 0; m < dotProd.size(); m++){
//                     dotProd[m].add(dotProdNew[m]);
//                 }
//             }
//             for (int m = 0; m < dotProd.size(); m++){
//                 dotProd[m].addPt(bias[0][j]);
//             }
//             row.emplace_back(std::move(dotProd));
//         }
//         product.emplace_back(std::move(row));
//     }
// }

///// Q, K, V merged 
void PCMMSquare_GPU_merged(FIDESlib::CKKS::Ciphertext& cMat1, const std::vector<FIDESlib::CKKS::Plaintext>& pMat2, uint32_t rowSize,
                    std::vector<FIDESlib::CKKS::Ciphertext>& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp) {

    FIDESlib::CKKS::Context& cc = cMat1.cc;

    // std::cout << "Step 1-1: ";
    // auto start_gpu = std::chrono::high_resolution_clock::now();

    // Step 1-1: Linear transform for first matrix
    FIDESlib::CKKS::Ciphertext linearTransform1(cc);
    linearTransform1.copy(cMat1);
    LinearTransform(linearTransform1, 2 * rowSize - 1, precomp.bStep, precomp.pts_1, 1, -(int)rowSize + 1);

    // Step 1-2: Linear transform for second matrix
    std::vector<FIDESlib::CKKS::Plaintext> linearTransform2;
    for (int i = 0; i < pMat2.size(); i++){
        Plaintext lt(cc);
        lt.copy(pMat2[i]);
        linearTransform2.emplace_back(std::move(lt));
    }

    // auto end_gpu = std::chrono::high_resolution_clock::now();
    // double time1 = (std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count());
        
    // std::cout << "Step 2 and 3: ";
    // start_gpu = std::chrono::high_resolution_clock::now();

    // Steps 2 and 3: Initial computation
    Ciphertext aux(cc);
    aux.rotate(linearTransform1, -(int)rowSize /*+ 1*/, cc.GetRotationKey(-(int)rowSize /*+ 1*/));
    
    FIDESlib::CKKS::Ciphertext cProduct1(cc), cProduct2(cc), cProduct3(cc);

    cProduct.reserve(pMat2.size());
    // K
    LinearTransformSpecialPt(linearTransform1, aux, linearTransform2[0], rowSize, precomp.bStep, precomp.pts_3_1,
                             precomp.pts_3_2, 1, rowSize, cProduct1);  
    cProduct.emplace_back(std::move(cProduct1));

    // end_gpu = std::chrono::high_resolution_clock::now();
    // double time2 = (std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count());

    // std::cout << "ratio: " << time1 / time2 << std::endl;
        
    // Q
    LinearTransformSpecialPt(linearTransform1, aux, linearTransform2[1], rowSize, precomp.bStep, precomp.pts_3_1,
                             precomp.pts_3_2, 1, rowSize, cProduct2);  
    cProduct.emplace_back(std::move(cProduct2));
    // V
    LinearTransformSpecialPt(linearTransform1, aux, linearTransform2[2], rowSize, precomp.bStep, precomp.pts_3_1,
                             precomp.pts_3_2, 1, rowSize, cProduct3);  
    cProduct.emplace_back(std::move(cProduct3));

    for (int i = 0; i < cProduct.size(); i++){
        if (cProduct[i].NoiseLevel == 2)
            cProduct[i].rescale();
    }
}

    void PCMM_GPU_QKV_merged(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
                std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>>& matrix2, uint32_t rowSize,
                std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>>& product,
                const MatrixMatrixProductPrecomputations_GPU& precomp,
                std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>>& bias) {

        product.clear();
        product.reserve(3);

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> product1, product2, product3;
        product1.reserve(matrix1.size());
        product2.reserve(matrix1.size());
        product3.reserve(matrix1.size());

        for (size_t i = 0; i < matrix1.size(); i++) {
            std::vector<FIDESlib::CKKS::Ciphertext> row, row2, row3;
            row.reserve(matrix2[0].size());
            row2.reserve(matrix2[0].size());
            row3.reserve(matrix2[0].size());
            for (size_t j = 0; j < matrix2[0].size(); j++) {
                std::vector<FIDESlib::CKKS::Ciphertext> dotProd;
                dotProd.reserve(3);
                PCMMSquare_GPU_merged(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);

                for (size_t m = 0; m < dotProd.size(); ++m)
                    dotProd[m].addPt(bias[0][j][m]);

                for (size_t k = 1; k < matrix2.size(); k++) {
                    std::vector<FIDESlib::CKKS::Ciphertext> dotProdNew;
                    dotProdNew.reserve(3);
                    PCMMSquare_GPU_merged(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);

                    for (size_t m = 0; m < dotProd.size(); ++m)
                        dotProd[m].add(dotProdNew[m]);
                }
                row.emplace_back(std::move(dotProd[0]));
                row2.emplace_back(std::move(dotProd[1]));
                row3.emplace_back(std::move(dotProd[2]));
            }
            product1.emplace_back(std::move(row));
            product2.emplace_back(std::move(row2));
            product3.emplace_back(std::move(row3));
        }
        product.emplace_back(std::move(product1));
        product.emplace_back(std::move(product2));
        product.emplace_back(std::move(product3));
    }


    // masked bias
    void PCMM_GPU_QKV_merged(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
                std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>>& matrix2, uint32_t rowSize,
                std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>>& product,
                const MatrixMatrixProductPrecomputations_GPU& precomp,
                std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>>& bias, Plaintext& mask_row) {

        product.clear();
        product.reserve(3);

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> product1, product2, product3;
        product1.reserve(matrix1.size());
        product2.reserve(matrix1.size());
        product3.reserve(matrix1.size());

        for (size_t i = 0; i < matrix1.size(); i++) {
            std::vector<FIDESlib::CKKS::Ciphertext> row, row2, row3;
            row.reserve(matrix2[0].size());
            row2.reserve(matrix2[0].size());
            row3.reserve(matrix2[0].size());
            for (size_t j = 0; j < matrix2[0].size(); j++) {
                std::vector<FIDESlib::CKKS::Ciphertext> dotProd;
                dotProd.reserve(3);
                PCMMSquare_GPU_merged(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);

                for (size_t m = 0; m < dotProd.size(); ++m){
                    Plaintext bias_masked(matrix1[i][0].cc);
                    bias_masked.copy(bias[0][j][m]);
                    bias_masked.multPt(bias_masked, mask_row, true);
                    
                    dotProd[m].addPt(bias_masked);

                    // dotProd[m].addPt(bias[0][j][m]);

                }
                for (size_t k = 1; k < matrix2.size(); k++) {
                    std::vector<FIDESlib::CKKS::Ciphertext> dotProdNew;
                    dotProdNew.reserve(3);
                    PCMMSquare_GPU_merged(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);

                    for (size_t m = 0; m < dotProd.size(); ++m)
                        dotProd[m].add(dotProdNew[m]);
                }
                row.emplace_back(std::move(dotProd[0]));
                row2.emplace_back(std::move(dotProd[1]));
                row3.emplace_back(std::move(dotProd[2]));
            }
            product1.emplace_back(std::move(row));
            product2.emplace_back(std::move(row2));
            product3.emplace_back(std::move(row3));
        }
        product.emplace_back(std::move(product1));
        product.emplace_back(std::move(product2));
        product.emplace_back(std::move(product3));
    }


// PCMM without bias
void PCMM_GPU_NoBias(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp) {

    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());
        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            PCMMSquare_GPU(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                PCMMSquare_GPU(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }
}

// Function to convert lbcrypto precomputations to GPU precomputations
struct MatrixMatrixProductPrecomputations_GPU convertToGPUPrecomputations(
    FIDESlib::CKKS::Context& GPUcc, lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& context,
    const MatrixMatrixProductPrecomputations& cpuPrecomp, PtMasks_GPU& masks, int bStep, int levelCP, int levelCC,
    bool fuse_boot_prescale_CCMM, const int slots) {

    // Adjustment for the new PCMM
    levelCP = levelCP + 1;
    levelCC = levelCC + 1;

    struct MatrixMatrixProductPrecomputations_GPU gpuPrecomp;
    gpuPrecomp.rowSize = cpuPrecomp.rowSize;

#if BSGS
    gpuPrecomp.bStep = bStep;
#endif

    gpuPrecomp.sigmaPlaintexts.resize(2);
    // Convert sigma plaintexts
    {
        auto pt_rots =
            GetLinearTransformPlaintextRotationIndices(2 * gpuPrecomp.rowSize - 1, bStep, 1, -gpuPrecomp.rowSize + 1);

        for (int i = 0; i < gpuPrecomp.rowSize; ++i) {

            const auto& sigmaPt_ = cpuPrecomp.sigmaPlaintexts[i];
#if BSGS
#if !EXT
            auto sigmaPt = context->MakeCKKSPackedPlaintext(
                Rotate(sigmaPt_->GetCKKSPackedValue(), pt_rots[i + gpuPrecomp.rowSize - 1]), 1, GPUcc.L - levelCP);
#else
            auto sigmaPt =
                encodeExt(context, Rotate(sigmaPt_->GetCKKSPackedValue(), pt_rots[i + gpuPrecomp.rowSize - 1]), 1,
                          GPUcc.L - level);
#endif
#endif
            auto raw_sigma = FIDESlib::CKKS::GetRawPlainText(context, sigmaPt);
            FIDESlib::CKKS::Plaintext sigma_gpu(GPUcc, raw_sigma);
            gpuPrecomp.sigmaPlaintexts[0].emplace_back(std::move(sigma_gpu));
        }

        for (int i = 0; i < gpuPrecomp.rowSize - 1; ++i) {
            const auto& sigmaPt_ = cpuPrecomp.sigmaPlaintexts[gpuPrecomp.rowSize * gpuPrecomp.rowSize - 1 - i];
#if BSGS
#if !EXT
            auto sigmaPt = context->MakeCKKSPackedPlaintext(
                Rotate(sigmaPt_->GetCKKSPackedValue(), pt_rots[gpuPrecomp.rowSize - 2 - i]), 1, GPUcc.L - levelCP);
#else

            auto sigmaPt =
                encodeExt(context, Rotate(sigmaPt_->GetCKKSPackedValue(), pt_rots[gpuPrecomp.rowSize - 2 - i]), 1,
                          GPUcc.L - level + 1);
#endif
#endif
            auto raw_sigma = FIDESlib::CKKS::GetRawPlainText(context, sigmaPt);
            FIDESlib::CKKS::Plaintext sigma_gpu(GPUcc, raw_sigma);
            gpuPrecomp.sigmaPlaintexts[1].emplace_back(std::move(sigma_gpu));
        }
#if BSGS
        gpuPrecomp.pts_1.resize(gpuPrecomp.rowSize * 2 - 1);
        for (int i = 0; i < gpuPrecomp.rowSize - 1; ++i) {
            gpuPrecomp.pts_1[gpuPrecomp.rowSize - 2 - i] = &gpuPrecomp.sigmaPlaintexts[1][i];
        }
        for (int i = 0; i < gpuPrecomp.rowSize; ++i) {
            gpuPrecomp.pts_1[i + gpuPrecomp.rowSize - 1] = &gpuPrecomp.sigmaPlaintexts[0][i];
        }
#endif
    }
    {
        auto pt_rots = GetLinearTransformPlaintextRotationIndices(gpuPrecomp.rowSize, bStep, gpuPrecomp.rowSize);

        for (int i = 0; i < gpuPrecomp.rowSize; ++i) {
            const auto& tauPt_ = cpuPrecomp.tauPlaintexts[i * gpuPrecomp.rowSize];
            auto values = tauPt_->GetCKKSPackedValue();
            if (fuse_boot_prescale_CCMM) {
                double scale = FIDESlib::CKKS::GetPreScaleFactor(GPUcc, slots);
                //std::cout << scale << std::endl;
                for (auto& k : values)
                    k *= scale;
            } else {
                // for (auto& k : values)
                //     k *= 0.001;
            }

#if BSGS
#if !EXT
            auto tauPt = context->MakeCKKSPackedPlaintext(Rotate(values, pt_rots[i]), 1, GPUcc.L - levelCC);
#else

            auto tauPt = encodeExt(context, Rotate(values, pt_rots[i]), 1, GPUcc.L - level);
#endif
#endif
            {
                auto raw_tau = FIDESlib::CKKS::GetRawPlainText(context, tauPt);
                FIDESlib::CKKS::Plaintext tau_gpu(GPUcc, raw_tau);
                gpuPrecomp.tauPlaintexts.emplace_back(std::move(tau_gpu));
            }
        }

#if BSGS
        gpuPrecomp.pts_2.resize(gpuPrecomp.rowSize);
        for (int i = 0; i < gpuPrecomp.rowSize; ++i) {
            gpuPrecomp.pts_2[i] = &gpuPrecomp.tauPlaintexts[i];
        }
#endif
    }

    {
        auto pt_rots = GetLinearTransformPlaintextRotationIndices(gpuPrecomp.rowSize, bStep, 1, 0);
        // Convert phi plaintexts

        int i = 0;
        for (const auto& phiVec : cpuPrecomp.phiPlaintexts) {
            std::vector<FIDESlib::CKKS::Plaintext> phiGpuVec;
            int j = 0;
            for (const auto& phiPt_ : phiVec) {
                const auto& tauPt_ = cpuPrecomp.tauPlaintexts[i * gpuPrecomp.rowSize];
                auto values = phiPt_->GetCKKSPackedValue();
#if BSGS
#if !EXT
                auto phiPt = context->MakeCKKSPackedPlaintext(Rotate(values, pt_rots[i]), 1, GPUcc.L - levelCP + 1);
#else
                auto phiPt = context->MakeCKKSPackedPlaintext(Rotate(phiPt_->GetCKKSPackedValue(), pt_rots[i]), 1,
                                                              GPUcc.L - level);
                //        auto phiPt = encodeExt(context, Rotate(phiPt_->GetCKKSPackedValue(), pt_rots[i]), 1, GPUcc.L - level);
#endif
#endif
                auto raw_phi = FIDESlib::CKKS::GetRawPlainText(context, phiPt);
                FIDESlib::CKKS::Plaintext phi_gpu(GPUcc, raw_phi);

                // NEW PCMM!!!!
                // phi_gpu.multPt(phi_gpu, masks.U_masks[0], true);
                phiGpuVec.emplace_back(std::move(phi_gpu));
                ++j;
            }
            gpuPrecomp.phiPlaintexts.emplace_back(std::move(phiGpuVec));
            ++i;
        }

#if BSGS
        gpuPrecomp.pts_3_2.resize(gpuPrecomp.rowSize);
        for (int i = 1; i < gpuPrecomp.rowSize; ++i) {
            gpuPrecomp.pts_3_2[i] = &gpuPrecomp.phiPlaintexts[i][1];
        }
        gpuPrecomp.pts_3_1.resize(gpuPrecomp.rowSize);
        for (int i = 0; i < gpuPrecomp.rowSize; ++i) {
            gpuPrecomp.pts_3_1[i] = &gpuPrecomp.phiPlaintexts[i][0];
        }
#endif
    }


        {
        auto pt_rots = GetLinearTransformPlaintextRotationIndices(gpuPrecomp.rowSize, bStep, 1, 0);
        // Convert phi plaintexts

        int i = 0;
        for (const auto& phiVec : cpuPrecomp.phiPlaintexts_new) {
            std::vector<FIDESlib::CKKS::Plaintext> phiGpuVec;
            int j = 0;
            for (const auto& phiPt_ : phiVec) {
                const auto& tauPt_ = cpuPrecomp.tauPlaintexts[i * gpuPrecomp.rowSize];
                auto values = phiPt_->GetCKKSPackedValue();
#if BSGS
#if !EXT
                auto phiPt = context->MakeCKKSPackedPlaintext(Rotate(values, pt_rots[i]), 1, GPUcc.L - levelCP + 1);
#else
                auto phiPt = context->MakeCKKSPackedPlaintext(Rotate(phiPt_->GetCKKSPackedValue(), pt_rots[i]), 1,
                                                              GPUcc.L - level);
                //        auto phiPt = encodeExt(context, Rotate(phiPt_->GetCKKSPackedValue(), pt_rots[i]), 1, GPUcc.L - level);
#endif
#endif
                auto raw_phi = FIDESlib::CKKS::GetRawPlainText(context, phiPt);
                FIDESlib::CKKS::Plaintext phi_gpu(GPUcc, raw_phi);
                phiGpuVec.emplace_back(std::move(phi_gpu));
                ++j;
            }
            gpuPrecomp.phiPlaintexts_new.emplace_back(std::move(phiGpuVec));
            ++i;

        }

#if BSGS
        gpuPrecomp.pts_3_2_new.resize(gpuPrecomp.rowSize);
        for (int i = 1; i < gpuPrecomp.rowSize; ++i) {
            gpuPrecomp.pts_3_2_new[i] = &gpuPrecomp.phiPlaintexts_new[i][1];
        }
        gpuPrecomp.pts_3_1_new.resize(gpuPrecomp.rowSize);
        for (int i = 0; i < gpuPrecomp.rowSize; ++i) {
            gpuPrecomp.pts_3_1_new[i] = &gpuPrecomp.phiPlaintexts_new[i][0];
        }
#endif
    }


    //gpuPrecomp.tauVectors.resize(gpuPrecomp.rowSize);
    //for (int i = 0; i < gpuPrecomp.rowSize; ++i) {
    //        gpuPrecomp.tauVectors[i] = cpuPrecomp.tauVectors[i];
    //    }

    return gpuPrecomp;
}

// Direct GPU version of getMatrixMatrixProductPrecomputations
struct MatrixMatrixProductPrecomputations_GPU getMatrixMatrixProductPrecomputations_GPU(
    FIDESlib::CKKS::Context& GPUcc, lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& context, PtMasks_GPU& masks, int rowSize, int bStep,
    const int levelCP, const int levelCC, const bool fuse_boot_prescale_CCMM, const int slots) {

    // First get CPU precomputations
    MatrixMatrixProductPrecomputations cpuPrecomp = getMatrixMatrixProductPrecomputations(context, rowSize);

    // Then convert to GPU precomputations
    return convertToGPUPrecomputations(GPUcc, context, cpuPrecomp, masks, bStep, levelCP, levelCC, fuse_boot_prescale_CCMM,
                                       slots);
}

std::vector<int> GenerateMatMulRotationIndices_GPU(int rowSize, int bStep) {
    std::set<int> indices;
#if LOW_MEM

#if BSGS
    std::vector<int> aux = GetLinearTransformRotationIndices(bStep, 1, -rowSize + 1);
    std::vector<int> aux2 = GetLinearTransformRotationIndices(bStep, rowSize, 0);
    for (auto& i : aux)
        indices.insert(i);
    for (auto& i : aux2)
        indices.insert(i);
    indices.insert(-rowSize);

    // For the special transform for steps 2 and 3
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);
    if (gStep - 1 != 0 && rowSize - 1 != 0)
        indices.insert((gStep - 1) * bStep * (rowSize - 1));
    if (rowSize - 1 != 0)
        indices.insert(-bStep * (rowSize - 1));
#else
    indices.insert(1);
    indices.insert(-1);
    indices.insert(rowSize);
    indices.insert(-rowSize);
#endif

#else
    for (size_t i = 1; i < rowSize; i++) {
        indices.insert(i);
        indices.insert(-i);
        indices.insert(i * rowSize);
        indices.insert(i - rowSize);
    }
#endif

    std::vector<int> indicesList(indices.begin(), indices.end());
    return indicesList;
}

FIDESlib::CKKS::Ciphertext rotsum_GPU(FIDESlib::CKKS::Ciphertext& in, int blockSize, int padding) {
    { std::cout << "Replace RotSum with CKKS::Accumulate " << std::endl; }
    Context& cc = in.cc;

    FIDESlib::CKKS::Ciphertext prev_rotation(cc);
    prev_rotation.copy(in);

    for (int i = 0; i < std::log2(blockSize); ++i) {
        int rot_index = padding * (1 << i);
        FIDESlib::CKKS::Ciphertext rotated(cc);
        rotated.copy(prev_rotation);
        rotated.rotate(rot_index, cc.GetRotationKey(rot_index));
        prev_rotation.add(rotated);
    }

    return prev_rotation;
}

// Combine 2-D grids into [I][J][3]
void CombineQKV_2D_to_3( PtWeights_GPU& weights_layer, FIDESlib::CKKS::Context& cc, 
            std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>>& WeightsQKV, // [I][J][3]
            std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>>& BiasQKV     // [I][J][3]
) {
    using PT = FIDESlib::CKKS::Plaintext;
    const size_t I = weights_layer.Wk.size();
    const size_t J = weights_layer.Wk[0].size();

    WeightsQKV.clear(); BiasQKV.clear();
    WeightsQKV.resize(I);
    BiasQKV.resize(I);

    for (size_t i = 0; i < I; ++i) {
        WeightsQKV[i].resize(J);
        BiasQKV[i].resize(J);
        for (size_t j = 0; j < J; ++j) {
            WeightsQKV[i][j].reserve(3);
            BiasQKV[i][j].reserve(3);

            // K, Q, V
            WeightsQKV[i][j].emplace_back(cc);
            WeightsQKV[i][j].back().copy(weights_layer.Wk[i][j]);

            WeightsQKV[i][j].emplace_back(cc);
            WeightsQKV[i][j].back().copy(weights_layer.Wq[i][j]);

            WeightsQKV[i][j].emplace_back(cc);
            WeightsQKV[i][j].back().copy(weights_layer.Wv[i][j]);

            // bk, bq, bv
            BiasQKV[i][j].emplace_back(cc);
            BiasQKV[i][j].back().copy(weights_layer.bk[i][j]);

            BiasQKV[i][j].emplace_back(cc);
            BiasQKV[i][j].back().copy(weights_layer.bq[i][j]);

            BiasQKV[i][j].emplace_back(cc);
            BiasQKV[i][j].back().copy(weights_layer.bv[i][j]);
        }
    }
}


// Asymmetrical PCMM, square
void PCMMSquare_2(FIDESlib::CKKS::Ciphertext& cMat1, const FIDESlib::CKKS::Plaintext& pMat2, uint32_t rowSize,
                    FIDESlib::CKKS::Ciphertext& cProduct, const MatrixMatrixProductPrecomputations_GPU& precomp) {

    FIDESlib::CKKS::Context& cc = cMat1.cc;

    FIDESlib::CKKS::Plaintext linearTransform2(cc);
    linearTransform2.copy(pMat2);

    // Step 2:
    Ciphertext linearTransform1(cc);
    linearTransform1.copy(cMat1);

    Ciphertext aux(cc);
    aux.rotate(linearTransform1, -(int)rowSize, cc.GetRotationKey(-(int)rowSize));
    
    LinearTransformSpecialPt_2(linearTransform1, aux, linearTransform2, rowSize, precomp.bStep, precomp.pts_3_1_new,
                             precomp.pts_3_2_new, 1, rowSize); 

    if (linearTransform1.NoiseLevel == 2)
        linearTransform1.rescale();

    cProduct.copy(linearTransform1);
}

// Asymmetrical PCMM 
void PCMM_2(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias,
              Plaintext& mask_row) {
    product.clear();
    product.reserve(matrix1.size());
    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());
        for (size_t j = 0; j < matrix2[0].size(); j++) {
            FIDESlib::CKKS::Ciphertext dotProd(matrix1[i][0].cc);
            PCMMSquare_2(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);

            Plaintext bias_masked(matrix1[i][0].cc);
            bias_masked.copy(bias[0][j]);
            if (mask_row.NoiseLevel > 1) { mask_row.rescale(); }
            bias_masked.multPt(bias_masked, mask_row, true);

            dotProd.addPt(bias_masked);
            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                PCMMSquare_2(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }

}

// // Asymmetrical PCMM with delayed normalization and row-masked bias
void PCMM_2_delayedInv(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias,
              Plaintext& mask_row, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& factor) {
    product.clear();
    product.reserve(matrix1.size());

    FIDESlib::CKKS::Context& cc = matrix1[0][0].cc;

    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());
        for (size_t j = 0; j < matrix2[0].size(); j++) {
            auto factor_ind = std::min(i,j);
            FIDESlib::CKKS::Ciphertext dotProd(cc);
            PCMMSquare_2(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);

            Plaintext bias_masked(cc);
            bias_masked.copy(bias[0][j]);
            if (mask_row.NoiseLevel > 1) { mask_row.rescale(); }
            bias_masked.multPt(bias_masked, mask_row, true);

            Ciphertext tmp_bias(cc);
            tmp_bias.copy(factor[factor_ind][factor_ind]);
            tmp_bias.dropToLevel(bias_masked.c0.getLevel());
            tmp_bias.multPt(bias_masked);
            dotProd.dropToLevel(tmp_bias.getLevel());
            dotProd.add(tmp_bias);

            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                PCMMSquare_2(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }

}

// PCMM with delayed Inv (factored and masked) bias
void PCMM_GPU_delayedInv(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix1,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& matrix2, uint32_t rowSize,
              std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& product,
              const MatrixMatrixProductPrecomputations_GPU& precomp,
              std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias,
              Plaintext& mask_row, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& factor) {
    product.clear();
    product.reserve(matrix1.size());

    FIDESlib::CKKS::Context& cc = matrix1[0][0].cc;

    for (size_t i = 0; i < matrix1.size(); i++) {
        std::vector<FIDESlib::CKKS::Ciphertext> row;
        row.reserve(matrix2[0].size());
        for (size_t j = 0; j < matrix2[0].size(); j++) {
            auto factor_ind = std::min(i,j);
            FIDESlib::CKKS::Ciphertext dotProd(cc);
            PCMMSquare_GPU(matrix1[i][0], matrix2[0][j], rowSize, dotProd, precomp);

            Plaintext bias_masked(cc);
            bias_masked.copy(bias[0][j]);
            if (mask_row.NoiseLevel > 1) { mask_row.rescale(); }
            // std::cout << "bias[i][j]: "  << bias[i][j].c0.getLevel() << ", " << bias[i][j].NoiseLevel << std::endl;
            // std::cout << "mask_row: "  << mask_row.c0.getLevel() << ", " << mask_row.NoiseLevel << std::endl;
            bias_masked.multPt(bias_masked, mask_row, true);

            Ciphertext tmp_bias(cc);
            tmp_bias.copy(factor[factor_ind][factor_ind]);
            // std::cout << "bias_masked: "  << bias_masked.c0.getLevel() << ", " << bias_masked.NoiseLevel << std::endl;
            // std::cout << "factor[i][j]: "  << factor[factor_ind][factor_ind].getLevel() << ", " << factor[factor_ind][factor_ind].NoiseLevel << std::endl;
            tmp_bias.dropToLevel(bias_masked.c0.getLevel());
            tmp_bias.multPt(bias_masked);
            dotProd.dropToLevel(tmp_bias.getLevel());
            dotProd.add(tmp_bias);

            for (size_t k = 1; k < matrix2.size(); k++) {
                FIDESlib::CKKS::Ciphertext dotProdNew(matrix1[i][k].cc);
                PCMMSquare_GPU(matrix1[i][k], matrix2[k][j], rowSize, dotProdNew, precomp);
                dotProd.add(dotProdNew);
            }
            row.emplace_back(std::move(dotProd));
        }
        product.emplace_back(std::move(row));
    }

}


}  // namespace FIDESlib::CKKS