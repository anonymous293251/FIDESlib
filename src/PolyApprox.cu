#include "hip/hip_runtime.h"
//
// Created by seyda on 5/19/25.
//
#include "PolyApprox.cuh"

// Softmax
// exp: [-2, 2]
std::vector<double> cheb_coeff_exp_softmax = {22.6038, 19.5189, 12.8444, 6.67455, 2.83255, 1.00945, 0.30893, 0.0826599, 0.0196199, 0.00418051, 0.000807578, 0.000142616, 2.31874e-05, 3.49196e-06, 4.89675e-07, 6.42347e-08, 7.91444e-09, 9.19231e-10, 1.0097e-10, 1.05187e-11, 1.04334e-12, 9.62492e-14, 1.38012e-14, 3.17005e-15, 2.97591e-16, 4.20728e-15, 3.33981e-15, -1.15174e-15};
// 1/x step 1: [1, 1e4]
std::vector<double> cheb_coeff_inv_softmax1 = {15.2662, 3.04038, -1.00686, 0.537829, -0.344851, 0.24387, -0.183359, 0.143772, -0.116229, 0.0961676, -0.0810316, 0.0692861, -0.0599607, 0.0524146, -0.0462096, 0.0410368, -0.0366729, 0.0329531, -0.029753, 0.0269775, -0.0245527, 0.0224202, -0.0205335, 0.0188551, -0.0173546, 0.0160069, -0.0147912, 0.0136901, -0.0126892, 0.011776, -0.0109402, 0.0101726, -0.00946568, 0.00881261, -0.00820761, 0.0076456, -0.00712214, 0.00663329, -0.00617558, 0.00574594, -0.00534162, 0.00496015, -0.00459934, 0.00425718, -0.00393188, 0.0036218, -0.00332545, 0.00304144, -0.00276854, 0.00250556, -0.00225144, 0.00200515, -0.00176577, 0.00153238, -0.00130416, 0.00108029, -0.000860006, 0.000642551, -0.000427205, 0.000213254};
// 1/x step 2: [1, 10]
std::vector<double> cheb_coeff_inv_softmax2 = {0.210884, -0.201002, 0.176099, -0.144081, 0.111611, -0.0827377, 0.0591755, -0.0410896, 0.0278333, -0.0184621, 0.0120277, -0.00771463, 0.00488133, -0.00305179, 0.00188779, -0.00115673, 0.00070276, -0.000423683, 0.000253654, -0.000150898, 8.92474e-05, -5.25035e-05, 3.07357e-05, -1.79111e-05, 1.03937e-05, -6.0078e-06, 3.45998e-06, -1.98587e-06, 1.13616e-06, -6.48079e-07, 3.68631e-07, -2.09124e-07, 1.18339e-07, -6.68079e-08, 3.76318e-08, -2.11524e-08, 1.18657e-08, -6.64345e-09, 3.71282e-09, -2.07139e-09, 1.15372e-09, -6.41587e-10, 3.5625e-10, -1.97527e-10, 1.0937e-10, -6.0477e-11, 3.33993e-11, -1.84223e-11, 1.01497e-11, -5.58507e-12, 3.07035e-12, -1.6857e-12, 9.24767e-13, -5.06924e-13, 2.77302e-13, -1.51643e-13, 8.26928e-14, -4.3749e-14, 2.21008e-14, -8.72618e-15};
// sqrt(x)
// std::vector<double> cheb_coeff_squareroot = {4.03512, 1.33466, -0.262842, 0.110326, -0.0597802, 0.0369779, -0.0248124, 0.0175913, -0.0129753, 0.00985927, -0.00766574, 0.00606939, -0.0048757, 0.00396263, -0.00325056, 0.00268574, -0.00223088, 0.00185942, -0.00155204, 0.00129439, -0.00107563, 0.000887367, -0.000723054, 0.000577433, -0.000446209, 0.000325788, -0.000213075, 0.000105324};
std::vector<double> cheb_coeff_squareroot = {4.02777, 1.34082, -0.267314, 0.114015, -0.0629426, 0.0397433, -0.0272604, 0.0197758, -0.0149357, 0.0116253, -0.00926059, 0.00751176, -0.00618084, 0.00514318, -0.00431709, 0.00364721, -0.00309488, 0.00263245, -0.00223965, 0.00190132, -0.00160591, 0.00134445, -0.00110982, 0.000896257, -0.000699028, 0.000514111, -0.000338015, 0.000167612};
std::vector<double> cheb_coeff_squareroot_13 = {1.27626, 0.421811, -0.082865, 0.0346236, -0.0186233, 0.0113904, -0.00751481, 0.00519526, -0.00369067, 0.00264969, -0.00188728, 0.00129734, -0.000813979, 0.000392673};

// LayerNorm
std::vector<double> cheb_coeff_squareroot_ln = {5.72578, 1.87097, -0.358808, 0.144356, -0.0731997, 0.0405698, -0.0222362, 0.00995092};

// sqrt(x)
std::vector<double> cheb_coeff_inv_layernorm_1_1 = {12.736, 4.24097, -0.846262, 0.361496, -0.200004, 0.126656, -0.0871998, 0.0635537, -0.0482738, 0.0378351, -0.0303914, 0.0248999, -0.0207346, 0.017502, -0.0149441, 0.0128863, -0.0112069, 0.00981909, -0.00865952, 0.00768112, -0.00684831, 0.00613381, -0.00551641, 0.00497942, -0.00450957, 0.00409621, -0.00373069, 0.00340593, -0.00311612, 0.00285641, -0.00262278, 0.00241181, -0.00222064, 0.00204682, -0.00188827, 0.00174318, -0.00161, 0.00148739, -0.00137418, 0.00126934, -0.00117197, 0.00108128, -0.000996555, 0.000917179, -0.000842591, 0.000772289, -0.000705823, 0.000642784, -0.000582804, 0.000525542, -0.00047069, 0.000417962, -0.000367092, 0.000317834, -0.000269955, 0.000223237, -0.000177469, 0.000132453, -8.79949e-05, 4.39056e-05};
// 1/x
std::vector<double> cheb_coeff_inv_layernorm_1_2 = {2, -1.63636, 1.33884, -1.09542, 0.89625, -0.733296, 0.599969, -0.490884, 0.401632, -0.328608, 0.268861, -0.219977, 0.179982, -0.147258, 0.120483, -0.0985774, 0.0806542, -0.0659898, 0.0539917, -0.044175, 0.0361432, -0.0295717, 0.024195, -0.0197959, 0.0161967, -0.0132518, 0.0108424, -0.00887104, 0.00725812, -0.00593845, 0.00485872, -0.00397531, 0.00325251, -0.00266113, 0.00217726, -0.00178137, 0.00145746, -0.00119242, 0.000975573, -0.000798138, 0.000652952, -0.000534147, 0.000436924, -0.000357355, 0.000292224, -0.0002389, 0.000195228, -0.000159445, 0.000130104, -0.000106019, 8.62186e-05, -6.99014e-05, 5.64084e-05, -4.51946e-05, 3.58069e-05, -2.78658e-05, 2.10507e-05, -1.50861e-05, 9.73107e-06, -4.76919e-06};
// // 1 / sqrt(x)
std::vector<double> cheb_coeff_inv_layernorm = {0.746083, -0.491487, 0.406744, -0.356017, 0.319894, -0.2919, 0.26909, -0.249875, 0.233303, -0.218755, 0.205809, -0.19416, 0.183585, -0.173914, 0.165011, -0.156773, 0.149113, -0.141961, 0.135258, -0.128956, 0.123013, -0.117393, 0.112066, -0.107004, 0.102184, -0.0975847, 0.0931888, -0.0889794, 0.0849418, -0.0810629, 0.0773307, -0.0737343, 0.0702639, -0.0669104, 0.0636656, -0.0605218, 0.057472, -0.0545095, 0.0516284, -0.0488229, 0.0460877, -0.0434178, 0.0408085, -0.0382555, 0.0357544, -0.0333013, 0.0308925, -0.0285242, 0.0261931, -0.0238957, 0.021629, -0.0193897, 0.0171749, -0.0149817, 0.0128071, -0.0106485, 0.0085031, -0.00636818, 0.00424111, -0.00211925};

// gelu: [-20, 20] 
std::vector<double> cheb_coeff_relu = {12.7164, 10, 4.25996, -1.12549e-15, -0.864296, -4.29301e-16, 0.378673, 5.11298e-16, -0.216208, -2.86107e-17, 0.141752, -2.53104e-16, -0.101066, -1.36003e-16, 0.0760924, -1.87066e-16, -0.0594185, -7.45384e-16, 0.0475514, 4.08234e-16, -0.0386772, -7.37563e-16, 0.0317833, 6.37343e-16, -0.0262714, 1.22969e-15, 0.0217706, 1.96066e-15, -0.0180414, 5.59383e-16, 0.0149225, -1.00925e-15, -0.0123007, 7.9624e-17, 0.0100929, 6.5449e-16, -0.00823504, 2.93692e-15, 0.00667598, -1.43119e-15, -0.00537299, 2.2065e-15, 0.00428933, 2.10318e-15, -0.00339274, -2.12901e-15, 0.0026544, 1.60021e-15, -0.00204834, -3.0896e-15, 0.00155104, 1.57273e-15, -0.00114103, 3.13517e-15, 0.000798621, 2.02804e-15, -0.000505584, 2.57601e-15, 0.000244821, 1.13086e-14};

// tanh: [-20, 20]
std::vector<double> cheb_coeff_tanh = {1.77636e-16, 1.27193, 1.4803e-17, -0.420508, 2.40548e-16, 0.248214, -7.40149e-18, -0.173038, 2.62753e-16, 0.130341, 3.21965e-16, -0.102516, 1.70234e-16, 0.0827991, 4.07082e-16, -0.0680397, 1.14723e-16, 0.0565678, -4.07082e-17, -0.0474136, -5.44009e-16, 0.0399709, 6.62433e-16, -0.0338387, 1.70234e-16, 0.0287378, 3.84877e-16, -0.0244661, 3.14563e-16, 0.0208721, 8.69675e-16, -0.0178388, 2.70154e-16, 0.015274, -9.62193e-16, -0.0131038, 4.84797e-16, 0.0112679, 3.62673e-16, -0.00971691, -2.22045e-16, 0.00840999, -2.60902e-16, -0.00731326, 4.81097e-17, 0.00639867, 2.59052e-17, -0.00564301, 1.33227e-16, 0.00502725, 5.18104e-16, -0.00453594, -7.03141e-16, 0.00415682, 3.79326e-17, -0.00388044, -5.13941e-16, 0.00369991, 9.77921e-16, -0.00361075};

// Degree 27 coefficients
std::vector<double> cheb_coeff_exp_softmax_4 = {22.6038, 19.5189, 12.8444, 6.67455, 2.83255, 1.00945, 0.30893, 0.0826599, 0.0196199, 0.00418051, 0.000807578, 0.000142616, 2.31874e-05, 3.49196e-06, 4.89675e-07, 6.42347e-08, 7.91444e-09, 9.19231e-10, 1.0097e-10, 1.05187e-11, 1.04334e-12, 9.62492e-14, 1.38012e-14, 3.17005e-15, 2.97591e-16, 4.20728e-15, 3.33981e-15, -1.15174e-15};
std::vector<double> cheb_coeff_exp_softmax_2 = {4.55917, 3.18127, 1.3779, 0.42548, 0.101457, 0.0196514, 0.00320035, 0.000449278, 5.53987e-05, 6.08837e-06, 6.03393e-07, 5.4444e-08, 4.50826e-09, 3.44903e-10, 2.45199e-11, 1.62784e-12, 1.01355e-13, 5.99781e-15, 5.75283e-16, -1.38733e-16, 2.36558e-16, -4.44402e-16, 3.6722e-16, -2.6676e-18, -7.10034e-16, 3.22581e-16, 4.16053e-16, 2.70171e-16}; 
std::vector<double> cheb_coeff_exp_softmax_1 = {2.53213, 1.13032, 0.271495, 0.0443368, 0.00547424, 0.000542926, 4.49773e-05, 3.19844e-06, 1.99212e-07, 1.10368e-08, 5.5059e-10, 2.49795e-11, 1.03914e-12, 3.99088e-14, 1.76514e-15, 1.96862e-16, -1.32079e-17, 1.40569e-16, 1.23913e-16, -1.43947e-16, 3.08643e-17, -2.166e-16, -3.91206e-17, -3.18494e-16, -6.44747e-16, -8.80072e-17, 1.77988e-16, 7.41657e-16};
std::vector<double> cheb_coeff_exp_softmax_05 = {2.12697, 0.515789, 0.0638123, 0.00529022, 0.000329611, 1.64463e-05, 6.84247e-07, 2.44102e-08, 7.62157e-10, 2.11564e-11, 5.28556e-13, 1.19717e-14, 2.53367e-16, 7.55152e-17, 4.66275e-16, 2.90244e-16, -2.72772e-17, 9.7106e-17, 3.80061e-17, -1.27505e-16, -3.30434e-17, -1.65405e-16, -1.86796e-16, -5.13417e-16, -5.78011e-16, -1.71782e-16, 7.43779e-17, 1.08655e-15};

std::vector<double> cheb_coeff_inv_softmax1_27 = {5.44283, 0.596525, -0.241373, 0.141981, -0.0968308, 0.0715189, -0.0555137, 0.0445659, -0.0366489, 0.03068, -0.0260309, 0.0223132, -0.0192746, 0.0167442, -0.0146025, 0.0127633, -0.011163, 0.00975347, -0.0084977, 0.00736661, -0.00633693, 0.00538976, -0.00450947, 0.00368287, -0.00289863, 0.00214682, -0.0014185, 0.000705484};
std::vector<double> cheb_coeff_inv_softmax2_27 = {0.302733, -0.277884, 0.218387, -0.150772, 0.0936511, -0.0533594, 0.0283131, -0.0141566, 0.00673172, -0.00306667, 0.00134625, -0.000572233, 0.000236433, -9.52668e-05, 3.75372e-05, -1.44967e-05, 5.49824e-06, -2.05145e-06, 7.54085e-07, -2.73437e-07, 9.79177e-08, -3.46627e-08, 1.21406e-08, -4.21048e-09, 1.44675e-09, -4.92363e-10, 1.64628e-10, -4.97485e-11};
std::vector<double> cheb_coeff_inv_softmax_27 = {0.0101598, -0.00976182, 0.00936772, -0.00897736, 0.0085906, -0.00820727, 0.00782722, -0.00745031, 0.00707638, -0.00670528, 0.00633686, -0.00597097, 0.00560748, -0.00524623, 0.00488707, -0.00452987, 0.00417449, -0.00382077, 0.00346858, -0.00311778, 0.00276823, -0.00241978, 0.0020723, -0.00172565, 0.0013797, -0.00103429, 0.000689297, -0.000344579};

std::vector<double> cheb_coeff_inv_layernorm_27 = {0.68943, -0.434804, 0.34997, -0.299093, 0.262758, -0.234492, 0.211346, -0.191734, 0.1747, -0.159626, 0.146086, -0.133776, 0.122469, -0.111994, 0.102214, -0.0930209, 0.0843266, -0.0760575, 0.0681518, -0.0605567, 0.0532264, -0.0461205, 0.0392031, -0.0324418, 0.0258069, -0.0192707, 0.0128073, -0.00639178};
std::vector<double> cheb_coeff_relu_27 = {1, 0.635894, 6.10865e-17, -0.210048, -2.29257e-17, 0.123783, -4.54656e-17, -0.0861079, 1.32285e-17, 0.0647187, -4.28239e-17, -0.0508282, 3.02237e-17, 0.0410745, 3.64694e-17, -0.0339037, 8.43072e-18, 0.0285058, 5.4813e-17, -0.0244228, 4.75268e-19, 0.0213817, -1.36629e-16, -0.0192146, -4.53836e-16, 0.0178187, 8.04664e-17, -0.0171345};
std::vector<double> cheb_coeff_relu_13 = {1, 0.633708, -2.88693e-17, -0.203749, 6.61587e-19, 0.114141, -6.73466e-18, -0.0743652, 1.8037e-17, 0.0525471, 5.47986e-17, -0.0403031, -5.0067e-18, 0.0346936};

std::vector<double> cheb_coeff_tanh_27 = {1.90324e-16, 1.27231, 3.17207e-17, -0.421657, 3.17207e-17, 0.250161, 7.93016e-17, -0.17583, 8.72318e-17, 0.134049, 5.55112e-17, -0.107231, 8.72318e-17, 0.0886388, -5.63042e-16, -0.0751497, 6.34413e-17, 0.0651265, 1.74464e-16, -0.0576362, 6.74064e-17, 0.0521168, 0, -0.0482184, -3.80648e-16, 0.0457239, 2.5773e-16, -0.044506};

std::vector<double> cheb_coeff_inv_layernorm1_27 = {3.06073, 0.335451, -0.135734, 0.0798416, -0.0544519, 0.040218, -0.0312177, 0.0250613, -0.0206092, 0.0172527, -0.0146383, 0.0125477, -0.0108389, 0.00941596, -0.00821158, 0.00717733, -0.0062774, 0.00548478, -0.00477861, 0.00414255, -0.00356352, 0.00303089, -0.00253586, 0.00207103, -0.00163002, 0.00120724, -0.000797683, 0.000396723};
std::vector<double> cheb_coeff_inv_layernorm2_27 = {1404.83, -1376.09, 1295.97, -1178.16, 1038.42, -891.04, 747.025, -613.793, 495.539, -393.953, 308.968, -239.415, 183.538, -139.353, 104.891, -78.3319, 58.0792, -42.7789, 31.3152, -22.788, 16.4843, -11.8469, 8.44538, -5.94998, 4.1088, -2.72895, 1.66075, -0.784642};

namespace FIDESlib::CKKS { 

void evalFunction(FIDESlib::CKKS::Ciphertext& ctxt, const KeySwitchingKey& keySwitchingKey,
                  std::vector<double> cheb_coeff, int numSlots, double lower_bound = -1, double upper_bound = 1, bool bts = false) {
    // affine transformation to scale
    if (!(lower_bound == -1.0 && upper_bound == 1.0)) {
        double scale = 2.0 / (upper_bound - lower_bound);
        double shift = -(upper_bound + lower_bound) / (upper_bound - lower_bound);

            if (scale != 1){
                ctxt.multScalar(scale);
            }
            ctxt.addScalar(shift);
            lower_bound = -1;
            upper_bound = 1;
        }

        if (bts == true) {
            Bootstrap(ctxt, numSlots);
        }
        evalChebyshevSeries(ctxt, keySwitchingKey, cheb_coeff, lower_bound, upper_bound);
    }

    void evalTanh(FIDESlib::CKKS::Ciphertext& ctxt, const KeySwitchingKey& keySwitchingKey, int numSlots, double lower_bound, double upper_bound, bool bts){
        
        // evalFunction(ctxt, keySwitchingKey, cheb_coeff_tanh, numSlots, lower_bound, upper_bound, bts);
        evalFunction(ctxt, keySwitchingKey, cheb_coeff_tanh_27, numSlots, lower_bound, upper_bound, bts);
        
    }

    void EvalTanh_Matrix(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& ctxt, const KeySwitchingKey& keySwitchingKey, 
                int numSlots, double lower_bound, double upper_bound, bool bts){
        
        for (size_t i = 0; i < ctxt.size(); i++) {
            for (size_t j = 0; j < ctxt[0].size(); j++) {
                evalTanh(ctxt[i][j], keySwitchingKey, numSlots, lower_bound, upper_bound, bts);
            }
        }
    }

    void evalRelu(FIDESlib::CKKS::Ciphertext& ctxt, const KeySwitchingKey& keySwitchingKey, int numSlots){

        // evalFunction(ctxt, keySwitchingKey, cheb_coeff_relu, numSlots, -20, 20);
        Ciphertext tmp(ctxt.cc);
        tmp.copy(ctxt);
        // prescaled
        // evalFunction(ctxt, keySwitchingKey, cheb_coeff_relu_13, numSlots, -1, 1);   // Approximation to GeLU / x
        evalFunction(ctxt, keySwitchingKey, cheb_coeff_relu_27, numSlots, -20, 20, true);   // Approximation to GeLU / x
        Bootstrap(ctxt, numSlots);
    
        ctxt.mult(ctxt, tmp, keySwitchingKey);     // GeLU

    }

    void EvalSoftmax(FIDESlib::CKKS::Ciphertext& ctxt, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, 
                    lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey, const KeySwitchingKey& keySwitchingKey,
                    FIDESlib::CKKS::Plaintext& mask_token, FIDESlib::CKKS::Plaintext& mask_broadcast, FIDESlib::CKKS::Plaintext& mask_mean, FIDESlib::CKKS::Plaintext& mask_max,         
                    int numSlots, int blockSize, int bStepAcc, int token_length, bool bts, int test_case, int long_input){
        bool print = false;  // for debugging 

        if (print) std::cout << "SOFTMAX " << std::endl;      

        FIDESlib::CKKS::Context& cc = ctxt.cc;
        auto context = ctxt_cpu->GetCryptoContext();    // for debugging 

        int accum_size = 1 << static_cast<int>(std::ceil(std::log2(token_length)));
        // int accum_size = blockSize;

        if (test_case > 0) {      
            ctxt.multPt(mask_token);

            Ciphertext mean(ctxt.cc);
            mean.copy(ctxt);
            Accumulate(mean, bStepAcc, 1, accum_size);

            Plaintext mask_mean_(cc);
            mask_mean_.copy(mask_broadcast);
            mask_mean_.multScalar(1.0 / token_length, true);
            mean.multPt(mask_mean_);
            Broadcast(mean, bStepAcc, 1, accum_size);

            ////////////////////////////////////
            if (print) {
                std::cout << "mu " << std::endl;
                std::cout << "#: " << mean.getLevel() << " " << mean.NoiseLevel << std::endl;            
                FIDESlib::CKKS::RawCipherText raw_res;
                mean.store(cc, raw_res);
                auto result2(ctxt_cpu->Clone());
                GetOpenFHECipherText(result2, raw_res);

                lbcrypto::Plaintext result_pt;
                context->Decrypt(privateKey, result2, &result_pt);
                result_pt->SetLength(4);
                std::cout << result_pt << std::endl;
            }
            ////////////////////////////////////
            
            Ciphertext var(cc);
            var.copy(ctxt);
            var.sub(mean);  // ctxt - mean = var
            var.mult(var, var, keySwitchingKey);

            Ciphertext sum_var(ctxt.cc);
            sum_var.copy(var);
            Accumulate(sum_var, bStepAcc, 1, accum_size);
            sum_var.multPt(mask_mean_);
            Broadcast(sum_var, bStepAcc, 1, accum_size);

            // Square root
            // evalFunction(sum_var, keySwitchingKey, cheb_coeff_squareroot, numSlots, 0.1, 100, bts); // prescaled
            evalFunction(sum_var, keySwitchingKey, cheb_coeff_squareroot_13, numSlots, 0.001, 1, bts); // prescaled

            ////////////////////////////////////
            if (print) {
                std::cout << "sigma " << std::endl;
                std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
                FIDESlib::CKKS::RawCipherText raw_res;
                sum_var.store(cc, raw_res);
                auto result2(ctxt_cpu->Clone());
                GetOpenFHECipherText(result2, raw_res);

                lbcrypto::Plaintext result_pt;
                context->Decrypt(privateKey, result2, &result_pt);
                result_pt->SetLength(4);
                std::cout << result_pt << std::endl;
            }
            ////////////////////////////////////

            Ciphertext musigma(cc);
            musigma.copy(sum_var);
            musigma.sub(sum_var);      // 0


            musigma.add(sum_var);   // 1σ

            // paperdaki sonuclar 
            if (long_input == 1) musigma.multPt(mask_max, true);

            musigma.add(sum_var);   // 1σ

            // if (test_case == 2) {
            //     musigma.add(sum_var);   // 1σ
            // }
            musigma.add(mean);      // μ + 2σ

            ////////////////////////////////////
            if (print) {
                std::cout << "musigma " << std::endl;
                std::cout << "#: " << musigma.getLevel() << " " << musigma.NoiseLevel << std::endl;            
                FIDESlib::CKKS::RawCipherText raw_res;
                musigma.store(cc, raw_res);
                auto result2(ctxt_cpu->Clone());
                GetOpenFHECipherText(result2, raw_res);

                lbcrypto::Plaintext result_pt;
                context->Decrypt(privateKey, result2, &result_pt);
                result_pt->SetLength(4);
                std::cout << result_pt << std::endl;
            }
            ////////////////////////////////////

            ctxt.sub(musigma);
        }

        if (print) std::cout << "accum_size: " << accum_size << std::endl;      

        // Exponential
        evalFunction(ctxt, keySwitchingKey, cheb_coeff_exp_softmax_1, numSlots, -1, 1, bts); // prescaled

         ////////////////////////////////////
        if (print) {
            std::cout << "Step 1 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        if (bts){ Bootstrap(ctxt, numSlots); }
        for (int i=0; i < 4; i++){
            ctxt.mult(ctxt, ctxt, keySwitchingKey);    
        }
        if (bts){ Bootstrap(ctxt, numSlots); }  // ? necessary ?

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 2 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        ctxt.multPt(mask_token);
        Ciphertext scores_sum(ctxt.cc);
        scores_sum.copy(ctxt);
        FIDESlib::CKKS::Accumulate(scores_sum, bStepAcc, 1, accum_size);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 3-1 " << std::endl;
            std::cout << "#: " << scores_sum.getLevel() << " " << scores_sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            scores_sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        scores_sum.multPt(mask_broadcast); 

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 3-2 " << std::endl;
            std::cout << "#: " << scores_sum.getLevel() << " " << scores_sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            scores_sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Broadcast(scores_sum, bStepAcc, 1, accum_size);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 3-3 " << std::endl;
            std::cout << "#: " << scores_sum.getLevel() << " " << scores_sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            scores_sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Ciphertext scores_sum_x(scores_sum.cc);
        scores_sum_x.copy(scores_sum);

        // 1/x step 1
        evalFunction(scores_sum, keySwitchingKey, cheb_coeff_inv_softmax1_27, numSlots, 1, 1e4, bts);
        if (bts){ Bootstrap(scores_sum, numSlots); }

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 4 " << std::endl;
            std::cout << "#: " << scores_sum.getLevel() << " " << scores_sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            scores_sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // // 1/x step 2
        evalFunction(scores_sum, keySwitchingKey, cheb_coeff_inv_softmax2_27, numSlots, 1, 3, bts);
        if (bts){ Bootstrap(scores_sum, numSlots); }

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 5 " << std::endl;
            std::cout << "#: " << scores_sum.getLevel() << " " << scores_sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            scores_sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // NewtonRaphson
        NewtonRaphsonInv(scores_sum_x, scores_sum, keySwitchingKey, 1, ctxt, ctxt_cpu, privateKey);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 6 " << std::endl;
            std::cout << "#: " << scores_sum.getLevel() << " " << scores_sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            scores_sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        ctxt.mult(ctxt, scores_sum, keySwitchingKey);
   
        ////////////////////////////////////
        if (print) {
            std::cout << "Step 7 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // ctxt.copy(scores_sum);
    }
    

    void EvalSoftmax_Matrix(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& ctxt, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey, const KeySwitchingKey& keySwitchingKey, 
                FIDESlib::CKKS::Plaintext& mask_token, FIDESlib::CKKS::Plaintext& mask_broadcast, FIDESlib::CKKS::Plaintext& mask_mean, FIDESlib::CKKS::Plaintext& mask_max, int numSlots, int blockSize, int bStepAcc, int token_length, bool bts, int test_case, int long_input){
        
        for (size_t i = 0; i < ctxt.size(); i++) {
            for (size_t j = 0; j < ctxt[0].size(); j++) {
                EvalSoftmax(ctxt[i][j], ctxt_cpu, privateKey, keySwitchingKey, mask_token, mask_broadcast, mask_mean, mask_max, numSlots, blockSize, bStepAcc, token_length, bts, test_case, long_input);
            }
        }
    }

    void EvalGelu_Matrix(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& ctxt, const KeySwitchingKey& keySwitchingKey, int numSlots){
        
        for (size_t i = 0; i < ctxt.size(); i++) {
            for (size_t j = 0; j < ctxt[0].size(); j++) {
                evalRelu(ctxt[i][j], keySwitchingKey, numSlots);    // GeLU
            }
        }
    }

    void EvalLayerNorm(FIDESlib::CKKS::Ciphertext& ctxt, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey, 
                    const KeySwitchingKey& keySwitchingKey, std::vector<FIDESlib::CKKS::Plaintext>& mask_ln, FIDESlib::CKKS::Plaintext& mask_row,
                    int numSlots, int blockSize, FIDESlib::CKKS::Plaintext& weight, FIDESlib::CKKS::Plaintext& bias, bool bts, const int bStepAcc) {
        bool print = false;

        FIDESlib::CKKS::Context& cc = ctxt.cc;
        auto context = ctxt_cpu->GetCryptoContext();

        if (print) std::cout << "LAYERNORM " << std::endl;      

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 0 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Ciphertext sum(ctxt.cc);
        sum.copy(ctxt);

        Accumulate(sum, bStepAcc, 1, blockSize);


        ////////////////////////////////////
        if (print) {
            std::cout << "Step 1 " << std::endl;
            std::cout << "#: " << sum.getLevel() << " " << sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////
        
        sum.multPt(mask_ln[0]);

        Broadcast(sum, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 2 " << std::endl;
            std::cout << "#: " << sum.getLevel() << " " << sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        FIDESlib::CKKS::Ciphertext var(cc);

        var.copy(ctxt);
        var.sub(sum);  // ctxt - mean = var
        var.mult(var, var, keySwitchingKey);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 3 " << std::endl;
            std::cout << "#: " << var.getLevel() << " " << var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Ciphertext sum_var(ctxt.cc);
        sum_var.copy(var);
        Accumulate(sum_var, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 4 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        sum_var.multPt(mask_ln[1]);
        Broadcast(sum_var, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 5 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////


        // std::cout << "weight: " << weight.c0.getLevel() << ", " << weight.NoiseLevel << std::endl;
        // std::cout << "bias: " << bias.c0.getLevel() << ", " << bias.NoiseLevel << std::endl;
        // std::cout << "mask_row: " << mask_row.c0.getLevel() << ", " << mask_row.NoiseLevel << std::endl;


        if (bts) Bootstrap(sum_var, numSlots);

        Ciphertext sum_var_x(cc);
        sum_var_x.copy(sum_var);
        sum_var.addScalar(-(100+0.01)/(100-0.01));

        // evalFunction(sum_var, keySwitchingKey, cheb_coeff_inv_layernorm1_27, numSlots, -1, 1, bts); 
        evalFunction(sum_var, keySwitchingKey, cheb_coeff_inv_layernorm_27, numSlots, -1, 1); 
        if (bts) Bootstrap(sum_var, numSlots);   

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 6 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // evalFunction(sum_var, keySwitchingKey, cheb_coeff_inv_layernorm2_27, numSlots, 0.1, 2.1, bts); 
        // if (bts) Bootstrap(sum_var, numSlots);   

        double scale = 2.0 / (1e2 - 0.01);
        sum_var_x.multScalar(0.5 / scale);
        NewtonRaphsonInvSqrt(sum_var_x, sum_var, keySwitchingKey, 1);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 7 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////


        ctxt.sub(sum); // ctxt - mean

        // ctxt.mult(ctxt, sum_var, keySwitchingKey);

        Plaintext weight_masked(cc);
        weight_masked.copy(weight);
        weight_masked.multPt(weight_masked, mask_row, true);
        ctxt.multPt(weight_masked);

        ctxt.mult(ctxt, sum_var, keySwitchingKey);

        Plaintext bias_masked(cc);
        bias_masked.copy(bias);
        bias_masked.multPt(bias_masked, mask_row, true);
        ctxt.addPt(bias_masked);
    }

    void EvalLayerNorm_Matrix(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& ctxt, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey,
                            const KeySwitchingKey& keySwitchingKey, std::vector<FIDESlib::CKKS::Plaintext>& mask_ln, FIDESlib::CKKS::Plaintext& mask_row,
                            std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& weight,
                            std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias, int numSlots, int blockSize,
                            const int bStepAcc, bool bts) {

        for (size_t i = 0; i < ctxt.size(); i++) {
            for (size_t j = 0; j < ctxt[0].size(); j++) {
                EvalLayerNorm(ctxt[i][j], ctxt_cpu, privateKey, keySwitchingKey, mask_ln, mask_row, numSlots, blockSize, weight[i][j], bias[i][j], bts, bStepAcc);
            }
        }
    }



    void evalSqrtTaylor(FIDESlib::CKKS::Ciphertext& x, const KeySwitchingKey& ksk) {
        Context& cc = x.cc;

        // h = x - 1
        Ciphertext h(cc); h.copy(x);
        h.addScalar(-1.0);

        // h2 = h*h  (ct-ct mult #1)
        Ciphertext h2(cc);
        h2.mult(h, h, ksk);          

        // e1 = 1 + 0.5*h   
        Ciphertext e1(cc); e1.copy(h);
        e1.multScalar(0.5);
        e1.addScalar(1.0);

        // e2 = -1/8 + (1/16)*h  
        Ciphertext e2(cc); e2.copy(h);
        e2.multScalar(0.0625);        
        e2.addScalar(-0.125);     

        // prod = h2 * e2 
        Ciphertext prod(cc);
        e2.mult(h2, e2, ksk);
        e2.add(e1);
        x.copy(e2);
    }

    void NewtonRaphsonInv(FIDESlib::CKKS::Ciphertext& ctxt, FIDESlib::CKKS::Ciphertext& initial, const KeySwitchingKey& keySwitchingKey, int num_iter, 
                        FIDESlib::CKKS::Ciphertext& final, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey) {

        bool print = false;

        FIDESlib::CKKS::Context& cc = ctxt.cc;
        auto context = ctxt_cpu->GetCryptoContext();    

        FIDESlib::CKKS::Ciphertext ctxt_tmp(cc), ctxt_y(cc), ctxt_z(cc);

        ctxt_y.copy(initial);                 // y
        ctxt_z.copy(ctxt);                    // x
        ctxt_z.mult(initial, keySwitchingKey);           // x.y
        ctxt_z.mult(initial, keySwitchingKey);           // xy^2

        ctxt_tmp.copy(initial);  // y
        ctxt_tmp.add(ctxt_tmp); // 2y

        ctxt_tmp.dropToLevel(ctxt_z.getLevel());

        ctxt_tmp.sub(ctxt_z);  

        // for (int iter = 1; iter < num_iter; iter++) {
        //     ctxt_z.copy(ctxt_x);                           // x
        //     ctxt_y.mult(ctxt_y, ctxt_y, keySwitchingKey);  // y^2
        //     ctxt_z.mult(ctxt_z, ctxt_y, keySwitchingKey);  // xy^2

        //     ctxt_tmp.copy(ctxt_y);      // y
        //     ctxt_tmp.multScalar(2);     // 2y
        //     ctxt_tmp.dropToLevel(ctxt_z.getLevel());

        //     ctxt_y.sub(ctxt_tmp, ctxt_z);     // 2y - xy^2
        // }
        initial.copy(ctxt_tmp);
    }


    void NewtonRaphsonInvSqrt(FIDESlib::CKKS::Ciphertext& ctxt,
                FIDESlib::CKKS::Ciphertext& initial,
                const FIDESlib::CKKS::KeySwitchingKey& keySwitchingKey,
                int num_iter) {

        FIDESlib::CKKS::Context& cc = ctxt.cc;


        FIDESlib::CKKS::Ciphertext ctxt_x(cc), ctxt_y(cc);
        FIDESlib::CKKS::Ciphertext ctxt_y_sq(cc), ctxt_y_cu(cc), ctxt_xy_cu(cc), ctxt_tmp1(cc), ctxt_tmp2(cc);

        ctxt_x.copy(ctxt);      // x
        ctxt_y.copy(initial);   // y0

        for (int iter = 0; iter < num_iter; iter++) {
            ctxt_y_sq.copy(ctxt_y);
            ctxt_y_sq.mult(ctxt_y_sq, ctxt_y, keySwitchingKey);             // y^2


            ctxt_xy_cu.copy(ctxt_x);
            ctxt_xy_cu.mult(ctxt_y, keySwitchingKey);                      // xy
            ctxt_xy_cu.dropToLevel(ctxt_y_sq.getLevel());           

            ctxt_xy_cu.mult(ctxt_xy_cu, ctxt_y_sq, keySwitchingKey);        // x * y^3

            ctxt_tmp1.copy(ctxt_y);
            ctxt_tmp1.multScalar(1.5);                                       // 1.5 * y

            // ctxt_tmp2.multScalar(0.5);                                       // 0.5 * x * y^3
            ctxt_tmp1.dropToLevel(ctxt_xy_cu.getLevel());

            ctxt_y.sub(ctxt_tmp1, ctxt_xy_cu);                                // y = 1.5y - 0.5xy^3
        }

        initial.copy(ctxt_y); // return y
    }



    void EvalLayerNorm_DelayedInv(FIDESlib::CKKS::Ciphertext& ctxt, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey, 
                    const KeySwitchingKey& keySwitchingKey, std::vector<FIDESlib::CKKS::Plaintext>& mask_ln, FIDESlib::CKKS::Plaintext& mask_row,
                    int numSlots, int blockSize, FIDESlib::CKKS::Plaintext& weight, FIDESlib::CKKS::Plaintext& bias, bool bts, const int bStepAcc, FIDESlib::CKKS::Ciphertext& factor) {
        bool print = true;

        FIDESlib::CKKS::Context& cc = ctxt.cc;
        auto context = ctxt_cpu->GetCryptoContext();

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 0 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Ciphertext sum(ctxt.cc);
        sum.copy(ctxt);

        Accumulate(sum, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 1 " << std::endl;
            std::cout << "#: " << sum.getLevel() << " " << sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // sum.multPt(mask_ln[0]);
        
        sum.multPt(mask_ln[0]);

        Broadcast(sum, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 2 " << std::endl;
            std::cout << "#: " << sum.getLevel() << " " << sum.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        FIDESlib::CKKS::Ciphertext var(cc);

        var.copy(ctxt);
        var.sub(sum);  // ctxt - mean = var
        var.mult(var, var, keySwitchingKey);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 3 " << std::endl;
            std::cout << "#: " << var.getLevel() << " " << var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Ciphertext sum_var(ctxt.cc);
        sum_var.copy(var);
        Accumulate(sum_var, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 4 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        sum_var.multPt(mask_ln[0]);
        Broadcast(sum_var, bStepAcc, 1, blockSize);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 5 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // sqrt(csum_var)
        // evalFunction(sum_var, keySwitchingKey, cheb_coeff_squareroot_ln, numSlots, -1, 1);
        evalSqrtTaylor(sum_var, cc.GetEvalKey());
        // evalFunction(sum_var, keySwitchingKey, cheb_coeff_squareroot_ln, numSlots, 0.01, 100, bts);
        if (bts) Bootstrap(sum_var, numSlots); 
        // evalSqrtTaylor(sum_var, keySwitchingKey);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 6 " << std::endl;
            std::cout << "#: " << sum_var.getLevel() << " " << sum_var.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            sum_var.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        ctxt.sub(sum); // ctxt - mean

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 7 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Plaintext weight_masked(cc);
        weight_masked.copy(weight);
        weight_masked.multPt(weight_masked, mask_row, true);
        ctxt.multPt(weight_masked);


        ////////////////////////////////////
        if (print) {
            std::cout << "Step 8 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        Ciphertext tmp_bias(cc);
        tmp_bias.copy(sum_var); // sqrt(var)

        Plaintext bias_masked(cc);
        bias_masked.copy(bias);
        bias_masked.multPt(bias_masked, mask_row, true);

        tmp_bias.multPt(bias_masked);
        ctxt.add(tmp_bias);

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 9 " << std::endl;
            std::cout << "#: " << ctxt.getLevel() << " " << ctxt.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            ctxt.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

        // Ciphertext tmp_bias2(cc);
        // tmp_bias2.copy(sum_var); // 
        // tmp_bias2.sub(sum_var); // 0
        // tmp_bias2.addScalar(2); // 2
        // tmp_bias2.sub(sum_var); // 2 - factor

        factor.copy(sum_var);
        // factor.mult(tmp_bias2, cc.GetEvalKey()); // factor * (2 - factor)

        ////////////////////////////////////
        if (print) {
            std::cout << "Step 10 " << std::endl;
            std::cout << "#: " << factor.getLevel() << " " << factor.NoiseLevel << std::endl;            
            FIDESlib::CKKS::RawCipherText raw_res;
            factor.store(cc, raw_res);
            auto result2(ctxt_cpu->Clone());
            GetOpenFHECipherText(result2, raw_res);

            lbcrypto::Plaintext result_pt;
            context->Decrypt(privateKey, result2, &result_pt);
            result_pt->SetLength(4);
            std::cout << result_pt << std::endl;
        }
        ////////////////////////////////////

    }


    void EvalLayerNorm_Matrix_DelayedInv(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& ctxt, lbcrypto::Ciphertext<lbcrypto::DCRTPoly>& ctxt_cpu, lbcrypto::PrivateKey<lbcrypto::DCRTPoly> privateKey,
                            const KeySwitchingKey& keySwitchingKey, std::vector<FIDESlib::CKKS::Plaintext>& mask_ln, FIDESlib::CKKS::Plaintext& mask_row,
                            std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& weight,
                            std::vector<std::vector<FIDESlib::CKKS::Plaintext>>& bias, int numSlots, int blockSize,
                            const int bStepAcc, bool bts, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& factor) {

        factor.reserve(ctxt.size());
        for (size_t i = 0; i < ctxt.size(); i++) {
            std::vector<FIDESlib::CKKS::Ciphertext> row;
            row.reserve(ctxt[0].size());
            for (size_t j = 0; j < ctxt[0].size(); j++) {
                FIDESlib::CKKS::Ciphertext scale_factor(ctxt[0][0].cc);
                EvalLayerNorm_DelayedInv(ctxt[i][j], ctxt_cpu, privateKey, keySwitchingKey, mask_ln, mask_row, numSlots, blockSize, weight[i][j], bias[i][j], bts, bStepAcc, scale_factor);
                row.emplace_back(std::move(scale_factor));
            }
            factor.emplace_back(std::move(row));
        }
    }
}
