#include "hip/hip_runtime.h"
#include "Transformer.cuh"

namespace FIDESlib::CKKS {

    std::vector<std::vector<lbcrypto::Ciphertext<DCRTPoly>>> ct_tokens;
    lbcrypto::KeyPair<lbcrypto::DCRTPoly> keys_;

    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> encoder(
        PtWeights_GPU& weights_layer, MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
        TransposePrecomputations_GPU& Tprecomp_gpu, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& tokens,
        PtMasks_GPU& masks, EncoderConfiguration& conf, int layerNo) {
        constexpr bool PRINT = false ;

        Context& cc = tokens[0][0].cc;
        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> K, Q, V;
        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> GPUResult_QKT, GPUResult_Sm_V, GPUResult_Output, GPUResult_Up,
            GPUResult_Down;

        dropMatrixLevel(tokens, conf.level_matmul);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(tokens, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "tokens", false);

        if constexpr (PRINT) std::cout << "# limbs, tokens: " << tokens[0][0].getLevel() << " " << tokens[0][0].NoiseLevel << std::endl;

        PCMM_GPU(tokens, weights_layer.Wk, conf.blockSize, K, precomp_gpu, weights_layer.bk);
        PCMM_GPU(tokens, weights_layer.Wq, conf.blockSize, Q, precomp_gpu, weights_layer.bq);
        PCMM_GPU(tokens, weights_layer.Wv, conf.blockSize, V, precomp_gpu, weights_layer.bv);

        if constexpr (PRINT) std::cout << "# limbs, Q: " << Q[0][0].getLevel() << " " << Q[0][0].NoiseLevel << std::endl;

        ////////////////////////////// Multi Head Attention /////////////////////////////////
        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> Sm_V, Sm_V2;

        MatrixBootstrap(Q, conf.numSlots, conf.prescale); 
        MatrixBootstrap(K, conf.numSlots, conf.prescale); 

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> QKT1, QKT2;

        auto Q1 = MatrixMask(Q, masks.head_masks[0]);
        auto Q2 = MatrixMask(Q, masks.head_masks[1]);

        auto K1 = MatrixMask(K, masks.head_masks[0]);
        auto K2 = MatrixMask(K, masks.head_masks[1]);

        auto K1_T = MatrixTranspose_GPU(std::move(K1), conf.blockSize, Tprecomp_gpu);
        auto K2_T = MatrixTranspose_GPU(std::move(K2), conf.blockSize, Tprecomp_gpu);

        CCMM_GPU(Q1, K1_T, conf.blockSize, QKT1, precomp_gpu);
        CCMM_GPU(Q2, K2_T, conf.blockSize, QKT2, precomp_gpu);

        if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT1, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT1: ", false);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT2, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT2: ", false);

        QKT1 = MatrixMask(QKT1, masks.mask_tokens[conf.token_length]);
        QKT2 = MatrixMask(QKT2, masks.mask_tokens[conf.token_length]);

        // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT1, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT1: ", false);
        // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT2, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT2: ", false);

        MatrixRotate(QKT2, - conf.blockSize / conf.num_heads);

        // auto QKT = MatrixAdd(QKT1, QKT2);

        // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT: ", false);

        // if (layerNo == 1){ MatrixAddScalar(QKT, -0.125); }

        MatrixBootstrap(QKT1, conf.numSlots, conf.prescale);
        EvalSoftmax_Matrix(QKT1, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_tokens[conf.token_length], masks.mask_broadcast, masks.mask_layernorm[0], masks.mask_max, conf.numSlots, conf.blockSize, conf.bStepAcc, conf.token_length, true); 
        MatrixBootstrap(QKT1, conf.numSlots, conf.prescale);

        MatrixBootstrap(QKT2, conf.numSlots, conf.prescale);
        EvalSoftmax_Matrix(QKT2, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_tokens[conf.token_length], masks.mask_broadcast, masks.mask_layernorm[0], masks.mask_max, conf.numSlots, conf.blockSize, conf.bStepAcc, conf.token_length, true); 
        MatrixBootstrap(QKT2, conf.numSlots, conf.prescale);

        MatrixRotate(QKT2, conf.blockSize / conf.num_heads);

        if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT1, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT1: ", false);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT2, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT2: ", false);

        auto V1 = MatrixMask(V, masks.head_masks[0]);
        auto V2 = MatrixMask(V, masks.head_masks[1]);
        MatrixRotate(V2, conf.blockSize / conf.num_heads);

        CCMM_GPU(QKT1, V1, conf.blockSize, Sm_V, precomp_gpu);
        CCMM_GPU(QKT2, V2, conf.blockSize, Sm_V2, precomp_gpu);

        if constexpr (PRINT) printMatrix(decryptGPUMatrix(Sm_V, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Sm_V1: ", false);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(Sm_V2, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Sm_V2: ", false);
        MatrixRotate(Sm_V2, - conf.blockSize / conf.num_heads);

        Sm_V = MatrixAdd(Sm_V, Sm_V2);

        //////////////////////////////////////////////////////////////////////////////////////

        // GPUResult_V.clear();
        // GPUResult_QKT.clear();
        if constexpr (PRINT) std::cout << "# limbs: " << Sm_V[0][0].getLevel() << " " << Sm_V[0][0].NoiseLevel << std::endl;
        if constexpr (PRINT) std::cout << "# ------- bts  ------- " << std::endl;
        MatrixBootstrap(Sm_V, conf.numSlots);
        if constexpr (PRINT) std::cout << "# limbs: " << Sm_V[0][0].getLevel() << " " << Sm_V[0][0].NoiseLevel << std::endl;
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(Sm_V, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Sm_V: ", false);

        // Output CCMM
        dropMatrixLevel(Sm_V, conf.level_matmul);
        PCMM_GPU(Sm_V, weights_layer.Wo, conf.blockSize, GPUResult_Output, precomp_gpu, weights_layer.bo);
        if constexpr (PRINT) std::cout << "# limbs O: " << GPUResult_Output[0][0].getLevel() << std::endl;
        if constexpr (PRINT)  printMatrix(decryptGPUMatrix(GPUResult_Output, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Result_output: ", false);

        // Layer Norm
        // MatrixBootstrap(tokens, conf.numSlots, false);
        GPUResult_Output = MatrixAdd(GPUResult_Output, tokens);
        MatrixBootstrap(GPUResult_Output, conf.numSlots);

        tokens.clear();
        EvalLayerNorm_Matrix(GPUResult_Output, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length], weights_layer.Wln1, weights_layer.bln1, conf.numSlots,
                            conf.blockSize, conf.bStepAcc, true);
        if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Output[0][0].getLevel() << std::endl;

        if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
        MatrixBootstrap(GPUResult_Output, conf.numSlots);
        if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Output[0][0].getLevel() << std::endl;

        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Output, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LN: ", false);

        // Up PCMM
        // dropMatrixLevel(GPUResult_Output, conf.level_matmul - 2);
        PCMM_GPU(GPUResult_Output, weights_layer.Wu, conf.blockSize, GPUResult_Up, precomp_gpu, weights_layer.bu);

        // GPUResult_Output.clear();
        if constexpr (PRINT) std::cout << "# limbs U: " << GPUResult_Up[0][0].getLevel() << " " << GPUResult_Up[0][0].NoiseLevel << std::endl;
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Up, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "RELU input: ", false);

        // dropMatrixLevel(GPUResult_Up, conf.level_matmul);

        // ReLU
        EvalGelu_Matrix(GPUResult_Up, cc.GetEvalKey(), conf.numSlots); 
        if constexpr (PRINT) std::cout << "# limbs r: " << GPUResult_Up[0][0].getLevel() << std::endl;
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Up, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "RELU: ", false);
        
        // Down PCMM
        dropMatrixLevel(GPUResult_Up, conf.level_matmul - 4);
        PCMM_GPU(GPUResult_Up, weights_layer.Wd, conf.blockSize, GPUResult_Down, precomp_gpu, weights_layer.bd);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Result_Down: ", false);
        if constexpr (PRINT) std::cout << "# limbs D: " << GPUResult_Down[0][0].getLevel() << std::endl;
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Result_Down: ", false);

        // Layer Norm
        GPUResult_Down = MatrixAdd(GPUResult_Down, GPUResult_Output);
        MatrixBootstrap(GPUResult_Down, conf.numSlots);

        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LN input: ", false);

        EvalLayerNorm_Matrix(GPUResult_Down, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length], weights_layer.Wln2, weights_layer.bln2, conf.numSlots, conf.blockSize, conf.bStepAcc, true);
        if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Down[0][0].getLevel() << std::endl;
        if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
        MatrixBootstrap(GPUResult_Down, conf.numSlots);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LN: ", false);

        if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Down[0][0].getLevel() << std::endl;


        return GPUResult_Down;
    }

    // std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> encoder(
    //     PtWeights_GPU& weights_layer, MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
    //     TransposePrecomputations_GPU& Tprecomp_gpu, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& tokens,
    //     PtMasks_GPU& masks, EncoderConfiguration& conf, int layerNo) {
        
    //     constexpr bool PRINT = false ;

    //     Context& cc = tokens[0][0].cc;
    //     std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> factor;
    //     factor.reserve(tokens.size());
    //     for (size_t i = 0; i < tokens.size(); i++) {
    //         std::vector<FIDESlib::CKKS::Ciphertext> row;
    //         row.reserve(tokens[0].size());
    //         for (size_t j = 0; j < tokens[0].size(); j++) {
    //             row.emplace_back(cc);
    //         }
    //         factor.emplace_back(std::move(row));
    //     }

    //     std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>> K, Q, V, Sm_V_, QKT_heads;
    //     std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> GPUResult_Output, GPUResult_Up, GPUResult_Down;
    //     std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>> QKV_all;

    //     std::vector<std::vector<std::vector<FIDESlib::CKKS::Plaintext>>> WeightsQKV, BiasQKV;
    //     CombineQKV_2D_to_3(weights_layer, cc, WeightsQKV, BiasQKV);

    //     dropMatrixLevel(tokens, conf.level_matmul);
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(tokens, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "tokens", false);
    //     if constexpr (PRINT) std::cout << "# limbs, tokens: " << tokens[0][0].getLevel() << " " << tokens[0][0].NoiseLevel << std::endl;

    //     PCMM_GPU_QKV_merged(tokens, WeightsQKV, conf.blockSize, QKV_all, precomp_gpu, BiasQKV, masks.row_masks[conf.token_length]);    // QKV_all: K, Q, V

    //     if constexpr (PRINT) std::cout << "# limbs, Q: " << QKV_all[0][0][0].getLevel() << " " << QKV_all[0][0][0].NoiseLevel << std::endl;
    //     // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKV_all[0], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "K: ", false);
    //     // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKV_all[1], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Q: ", false);
    //     // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKV_all[2], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "V: ", false);

    //     ////////////////////////////// Multi Head Attention /////////////////////////////////
    //     auto K_T = MatrixTranspose_GPU(std::move(QKV_all[0]), conf.blockSize, Tprecomp_gpu);

    //     CCMM_GPU_double_mask(QKV_all[1], K_T, conf.blockSize, QKT_heads, precomp_gpu, masks, 0, conf.token_length, true);
    //     CCMM_GPU_double_mask(QKV_all[1], K_T, conf.blockSize, QKT_heads, precomp_gpu, masks, 1, conf.token_length, true);

    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT_heads[0], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT1: ", false);
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT_heads[1], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT2: ", false);


    //     MatrixRotate(QKT_heads[1], - conf.blockSize / 2);

    //     auto QKT = MatrixAdd(QKT_heads[0], QKT_heads[1]);

    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT: ", false);

    //     MatrixBootstrap(QKT, conf.numSlots, conf.prescale);
    //     EvalSoftmax_Matrix(QKT, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_tokens[conf.token_length], masks.mask_broadcast, masks.mask_layernorm[0], masks.mask_max, conf.numSlots, conf.blockSize, conf.bStepAcc, conf.token_length, true, 0, layerNo); 
    //     MatrixBootstrap(QKT, conf.numSlots, conf.prescale);

    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT: ", false);
    //     if constexpr (PRINT) std::cout << "# limbs: " << QKT[0][0].getLevel() << " " << QKT[0][0].NoiseLevel << std::endl;

    //     CCMM_GPU_double_mask(QKT, QKV_all[2], conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);
    //     MatrixRotate(QKV_all[2], conf.blockSize / 2);
    //     MatrixRotate(QKT, conf.blockSize / 2);
    //     CCMM_GPU_double_mask(QKT, QKV_all[2], conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);

    //     if constexpr (PRINT) std::cout << "# limbs: " << Sm_V_[0][0][0].getLevel() << " " << Sm_V_[0][0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(Sm_V_[0], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Sm_V1: ", false);
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(Sm_V_[1], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Sm_V2: ", false);
    //     MatrixRotate(Sm_V_[1], - conf.blockSize / conf.num_heads);

    //     auto Sm_V = MatrixAdd(Sm_V_[0], Sm_V_[1]);

    //     //////////////////////////////////////////////////////////////////////////////////////
    //     Sm_V = MatrixMask(Sm_V, masks.row_masks[conf.token_length]);
    //     if constexpr (PRINT) std::cout << "# limbs SmV: " << Sm_V[0][0].getLevel() << " "  << Sm_V[0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT)  printMatrix(decryptGPUMatrix(Sm_V, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "SmV: ", false);

    //     MatrixBootstrap(Sm_V, conf.numSlots);

    //     // Output CCMM
    //     PCMM_GPU(Sm_V, weights_layer.Wo, conf.blockSize, GPUResult_Output, precomp_gpu, weights_layer.bo, masks.row_masks[conf.token_length]);
        
    //     if constexpr (PRINT) std::cout << "# limbs O: " << GPUResult_Output[0][0].getLevel() << " "  << GPUResult_Output[0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT)  printMatrix(decryptGPUMatrix(GPUResult_Output, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Result_output: ", false);

    //     // Layer Norm
    //     GPUResult_Output = MatrixAdd(GPUResult_Output, tokens);

    //     if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
    //     MatrixBootstrap(GPUResult_Output, conf.numSlots);

    //     if constexpr (PRINT) std::cout << "# limbs LNin: " << GPUResult_Output[0][0].getLevel() << " "  << GPUResult_Output[0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT)  printMatrix(decryptGPUMatrix(GPUResult_Output, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNin: ", false);
    //     tokens.clear();

    //     EvalLayerNorm_Matrix(GPUResult_Output, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length], 
    //                         weights_layer.Wln1, weights_layer.bln1, conf.numSlots, conf.blockSize, conf.bStepAcc, true);
    //     if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Output[0][0].getLevel() << " "  << GPUResult_Output[0][0].NoiseLevel << std::endl;

    //     if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
    //     MatrixBootstrap(GPUResult_Output, conf.numSlots);
    //     if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Output[0][0].getLevel() << " "  << GPUResult_Output[0][0].NoiseLevel << std::endl;

    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Output, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNout: ", false);

    //     // Up PCMM
    //     PCMM_GPU(GPUResult_Output, weights_layer.Wu, conf.blockSize, GPUResult_Up, precomp_gpu, weights_layer.bu, masks.row_masks[conf.token_length]);

    //     if constexpr (PRINT) std::cout << "# size U: " << GPUResult_Up.size() << std::endl;
    //     if constexpr (PRINT) std::cout << "# limbs U: " << GPUResult_Up[0][0].getLevel() << " " << GPUResult_Up[0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Up, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Up: ", false);

    //     // ReLU
    //     EvalGelu_Matrix(GPUResult_Up, cc.GetEvalKey(), conf.numSlots); 
    //     if constexpr (PRINT) std::cout << "# limbs r: " << GPUResult_Up[0][0].getLevel() << " "  << GPUResult_Up[0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Up, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "GELU: ", false);
        
    //     // Down PCMM
    //     PCMM_GPU(GPUResult_Up, weights_layer.Wd, conf.blockSize, GPUResult_Down, precomp_gpu, weights_layer.bd, masks.row_masks[conf.token_length]);
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Down: ", false);
    //     if constexpr (PRINT) std::cout << "# limbs D: " << GPUResult_Down[0][0].getLevel() << " "  << GPUResult_Down[0][0].NoiseLevel << std::endl;

    //     // Layer Norm 2
    //     GPUResult_Down = MatrixAdd(GPUResult_Down, GPUResult_Output);
    //     MatrixBootstrap(GPUResult_Down, conf.numSlots);

    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNin: ", false);

    //     EvalLayerNorm_Matrix(GPUResult_Down, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length],
    //                         weights_layer.Wln2, weights_layer.bln2, conf.numSlots, conf.blockSize, conf.bStepAcc, true);
    //     if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Down[0][0].getLevel() << " "  << GPUResult_Down[0][0].NoiseLevel << std::endl;
    //     if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
    //     MatrixBootstrap(GPUResult_Down, conf.numSlots);
    //     if constexpr (PRINT) printMatrix(decryptGPUMatrix(GPUResult_Down, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNout: ", false);
    //     if constexpr (PRINT) std::cout << "# limbs LN: " << GPUResult_Down[0][0].getLevel() << " "  << GPUResult_Down[0][0].NoiseLevel << std::endl;

    //     return GPUResult_Down;
    // }



    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> encoder_helmet(
        PtWeights_GPU& weights_layer, MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
        TransposePrecomputations_GPU& Tprecomp_gpu, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& tokens,
        PtMasks_GPU& masks, EncoderConfiguration& conf, int layerNo, int test_case) {
        
        constexpr bool PRINT = true ;


        Context& cc = tokens[0][0].cc;
        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> factor;

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> K, Q, V, O, U, D, Sm_V;
        std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>> Sm_V_, QKT_heads;

        dropMatrixLevel(tokens, conf.level_matmul-2);

        if constexpr (PRINT) std::cout << "# tokens: " << conf.token_length << std::endl;
        
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(tokens, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "tokens", false);
        if constexpr (PRINT) std::cout << "# limbs, tokens: " << tokens[0][0].getLevel() << " " << tokens[0][0].NoiseLevel << std::endl;

        PCMM_2(tokens, weights_layer.Wk, conf.blockSize, K, precomp_gpu, weights_layer.bk, masks.row_masks[conf.token_length]);

        PCMM_2(tokens, weights_layer.Wq, conf.blockSize, Q, precomp_gpu, weights_layer.bq, masks.row_masks[conf.token_length]);

        PCMM_2(tokens, weights_layer.Wv, conf.blockSize, V, precomp_gpu, weights_layer.bv, masks.row_masks[conf.token_length]);

        if constexpr (PRINT) std::cout << "# limbs, Q: " << Q[0][0].getLevel() << " " << Q[0][0].NoiseLevel << std::endl;

        ////////////////////////////// Multi Head Attention /////////////////////////////////
        auto K_T = MatrixTranspose_GPU(std::move(K), conf.blockSize, Tprecomp_gpu);


        CCMM_GPU_double_mask(Q, K_T, conf.blockSize, QKT_heads, precomp_gpu, masks, 0, conf.token_length, true);
        CCMM_GPU_double_mask(Q, K_T, conf.blockSize, QKT_heads, precomp_gpu, masks, 1, conf.token_length, true);

        if (conf.token_length < 64) {
            
            MatrixRotate(QKT_heads[1], - conf.blockSize / 2);

            auto QKT = MatrixAdd(QKT_heads[0], QKT_heads[1]);

            if constexpr (PRINT) std::cout << "# limbs: " << QKT[0][0].getLevel() << " " << QKT[0][0].NoiseLevel << std::endl;
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT: ", false);

            // MRPC
            if (test_case == 0) {
                if (layerNo == 0){ MatrixAddScalar(QKT, -0.3); } 
                if (layerNo == 1){ MatrixAddScalar(QKT, -0.3); } 
            }

            // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT: ", false);
            // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), conf.token_length, conf.token_length, "QKT: ", false, 3, true);

            if constexpr (PRINT) std::cout << "# limbs: " << QKT[0][0].getLevel() << " " << QKT[0][0].NoiseLevel << std::endl;
            MatrixBootstrap(QKT, conf.numSlots, conf.prescale);
            if constexpr (PRINT) std::cout << "# limbs, before softmax: " << QKT[0][0].getLevel() << " " << QKT[0][0].NoiseLevel << std::endl;

            EvalSoftmax_Matrix(QKT, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_tokens[conf.token_length], masks.mask_broadcast, masks.mask_layernorm[0], masks.mask_max, conf.numSlots, conf.blockSize, conf.bStepAcc, conf.token_length, true, test_case, layerNo); 
            MatrixBootstrap(QKT, conf.numSlots, conf.prescale);

            // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), conf.token_length, conf.token_length, "QKT: ", false, 3, true);
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT: ", false);
            
            if constexpr (PRINT) std::cout << "# limbs, after softmax: " << QKT[0][0].getLevel() << " " << QKT[0][0].NoiseLevel << std::endl;

            dropMatrixLevel(QKT, conf.level_matmul-2);
            dropMatrixLevel(V, conf.level_matmul-2);

            CCMM_GPU_double_mask(QKT, V, conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);
            MatrixRotate(V, conf.blockSize / 2);
            MatrixRotate(QKT, conf.blockSize / 2);
            CCMM_GPU_double_mask(QKT, V, conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);

            if constexpr (PRINT) std::cout << "# limbs Sm: " << Sm_V_[0][0][0].getLevel() << " " << Sm_V_[0][0][0].NoiseLevel << std::endl;
            MatrixRotate(Sm_V_[1], - conf.blockSize / conf.num_heads);

            Sm_V = MatrixAdd(Sm_V_[0], Sm_V_[1]);

        }
        else {

            for (int j = 0; j < conf.num_heads; j++){
                // MRPC
                if (test_case == 0) {
                    if (layerNo == 0){ MatrixAddScalar(QKT_heads[j], -0.3); } 
                    if (layerNo == 1){ MatrixAddScalar(QKT_heads[j], -0.3); } 
                }
                // // RTE
                // if (test_case == 0) {
                //     if (layerNo == 0){ 
                //         MatrixAddScalar(QKT_heads[j], -0.25); 
                //         QKT_heads[j] = MatrixMaskSpecial(QKT_heads[j], masks, -0.25); 
                //         } 
                //     if (layerNo == 1){ 
                //         MatrixAddScalar(QKT_heads[j], -0.25); 
                //         QKT_heads[j] = MatrixMaskSpecial(QKT_heads[j], masks, -0.25); 
                //         } 
                // }

                if constexpr (PRINT) std::cout << "SM iteration: " << j << std::endl;

                if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT_heads[j], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT_heads[j]: ", false);

                MatrixBootstrap(QKT_heads[j], conf.numSlots, conf.prescale);
                EvalSoftmax_Matrix(QKT_heads[j], ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_tokens[conf.token_length], masks.mask_broadcast2, masks.mask_layernorm[0], masks.mask_max, conf.numSlots, conf.blockSize, conf.bStepAcc, conf.token_length, true, test_case, layerNo); 
                MatrixBootstrap(QKT_heads[j], conf.numSlots, conf.prescale);

                if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT_heads[j], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "QKT_heads[j]: ", false);

                if (j == 1) {
                    MatrixRotate(V, conf.blockSize / 2);
                    // MatrixRotate(QKT_heads[j], conf.blockSize / 2);
                }
                // CCMM_GPU_double_mask(QKT_heads[j], V, conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);
                CCMM_GPU_masked(QKT_heads[j], V, conf.blockSize, Sm_V_, precomp_gpu, masks.head_masks[0]);

                // if constexpr (PRINT) printMatrix(decryptGPUMatrix(Sm_V_[j], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Sm_V[j]: ", false);

            }

            MatrixRotate(Sm_V_[1], - conf.blockSize / conf.num_heads);
            Sm_V = MatrixAdd(Sm_V_[0], Sm_V_[1]);
        }

        //////////////////////////////////////////////////////////////////////////////////////
        Sm_V = MatrixMask(Sm_V, masks.row_masks[conf.token_length]);
        // if constexpr (PRINT) std::cout << "# limbs SmV: " << Sm_V[0][0].getLevel() << " "  << Sm_V[0][0].NoiseLevel << std::endl;
        // if constexpr (PRINT)  printMatrix(decryptGPUMatrix(Sm_V, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "SmV: ", false);

        MatrixBootstrap(Sm_V, conf.numSlots);

        // Output CCMM
        dropMatrixLevel(Sm_V, conf.level_matmul);
        PCMM_2(Sm_V, weights_layer.Wo, conf.blockSize, O, precomp_gpu, weights_layer.bo, masks.row_masks[conf.token_length]);
        // PCMM_GPU(Sm_V, weights_layer.Wo, conf.blockSize, O, precomp_gpu, weights_layer.bo, masks.row_masks[conf.token_length]);
        
        // if constexpr (PRINT) std::cout << "# limbs O: " << O[0][0].getLevel() << " "  << O[0][0].NoiseLevel << std::endl;
        // if constexpr (PRINT)  printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Result_output: ", false);

        // Layer Norm
        O = MatrixAdd(O, tokens);

        if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
        MatrixBootstrap(O, conf.numSlots);

        if constexpr (PRINT) std::cout << "# limbs LNin: " << O[0][0].getLevel() << " "  << O[0][0].NoiseLevel << std::endl;
        if constexpr (PRINT)  printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNin: ", false);
        tokens.clear();


        if (test_case == 2) {
            EvalLayerNorm_Matrix_DelayedInv(O, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length], 
                                weights_layer.Wln1, weights_layer.bln1, conf.numSlots, conf.blockSize, conf.bStepAcc, true, factor);
            // if constexpr (PRINT) std::cout << "# limbs LN: " << O[0][0].getLevel() << " "  << O[0][0].NoiseLevel << std::endl;

            // if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
            MatrixBootstrap(O, conf.numSlots);
            if constexpr (PRINT) std::cout << "# limbs LN: " << O[0][0].getLevel() << " "  << O[0][0].NoiseLevel << std::endl;

            if constexpr (PRINT) printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNout: ", false);
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(factor, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "factor: ", false);

            // Up PCMM
            PCMM_GPU_delayedInv(O, weights_layer.Wu, conf.blockSize, U, precomp_gpu, weights_layer.bu, masks.row_masks[conf.token_length], factor);

            // if constexpr (PRINT) std::cout << "# size U: " << U.size() << std::endl;
            if constexpr (PRINT) std::cout << "# limbs U: " << U[0][0].getLevel() << " " << U[0][0].NoiseLevel << std::endl;
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Up: ", false);

            // ReLU
            EvalGelu_Matrix(U, cc.GetEvalKey(), conf.numSlots); 
            if constexpr (PRINT) std::cout << "# limbs r: " << U[0][0].getLevel() << " "  << U[0][0].NoiseLevel << std::endl;
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "GELU: ", false);
            
            // Down PCMM
            dropMatrixLevel(U, conf.level_matmul - 4);
            PCMM_GPU_delayedInv(U, weights_layer.Wd, conf.blockSize, D, precomp_gpu, weights_layer.bd, masks.row_masks[conf.token_length], factor);
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Down: ", false);
            if constexpr (PRINT) std::cout << "# limbs D: " << D[0][0].getLevel() << " "  << D[0][0].NoiseLevel << std::endl;
        }
        else {
            EvalLayerNorm_Matrix(O, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length], 
                                weights_layer.Wln1, weights_layer.bln1, conf.numSlots, conf.blockSize, conf.bStepAcc, true);
            if constexpr (PRINT) std::cout << "# limbs LN: " << O[0][0].getLevel() << " "  << O[0][0].NoiseLevel << std::endl;

            if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
            MatrixBootstrap(O, conf.numSlots);
            if constexpr (PRINT) std::cout << "# limbs LN: " << O[0][0].getLevel() << " "  << O[0][0].NoiseLevel << std::endl;

            if constexpr (PRINT) printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNout: ", false);
        
            // Up PCMM
            PCMM_2(O, weights_layer.Wu, conf.blockSize, U, precomp_gpu, weights_layer.bu, masks.row_masks[conf.token_length]);
            // PCMM_GPU(O, weights_layer.Wu, conf.blockSize, U, precomp_gpu, weights_layer.bu, masks.row_masks[conf.token_length]);
            // MatrixBootstrap(O, conf.numSlots);

            if constexpr (PRINT) std::cout << "# limbs U: " << U[0][0].getLevel() << " " << U[0][0].NoiseLevel << std::endl;
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Up: ", false);

            // ReLU
            EvalGelu_Matrix(U, cc.GetEvalKey(), conf.numSlots); 
            if constexpr (PRINT) std::cout << "# limbs r: " << U[0][0].getLevel() << " "  << U[0][0].NoiseLevel << std::endl;
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "GELU: ", false);
            
            // Down PCMM
            dropMatrixLevel(U, conf.level_matmul - 4);
            PCMM_2(U, weights_layer.Wd, conf.blockSize, D, precomp_gpu, weights_layer.bd, masks.row_masks[conf.token_length]);
            // PCMM_GPU(U, weights_layer.Wd, conf.blockSize, D, precomp_gpu, weights_layer.bd, masks.row_masks[conf.token_length]);
            if constexpr (PRINT) printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "Down: ", false);
            if constexpr (PRINT) std::cout << "# limbs D: " << D[0][0].getLevel() << " "  << D[0][0].NoiseLevel << std::endl;

        }


        // Layer Norm 2
        D = MatrixAdd(D, O);
        MatrixBootstrap(D, conf.numSlots);

        if constexpr (PRINT) printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNin: ", false);

        EvalLayerNorm_Matrix(D, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm, masks.row_masks[conf.token_length],
                            weights_layer.Wln2, weights_layer.bln2, conf.numSlots, conf.blockSize, conf.bStepAcc, true);
        if constexpr (PRINT) std::cout << "# limbs LN: " << D[0][0].getLevel() << " "  << D[0][0].NoiseLevel << std::endl;
        if constexpr (PRINT) std::cout << "# ------- bts ------- " << std::endl;
        MatrixBootstrap(D, conf.numSlots);
        if constexpr (PRINT) printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "LNout: ", false);
        if constexpr (PRINT) std::cout << "# limbs LN: " << D[0][0].getLevel() << " "  << D[0][0].NoiseLevel << std::endl;

        hipDeviceSynchronize();

        return D;
    }



    void process_sentences_from_csv(std::string& file_path,
                                    std::string& output_file,
                                    std::string& model_name,
                                    std::string& model_path,
                                    std::string& output_path,
                                    EncoderConfiguration& base_conf,
                                    lbcrypto::PublicKey<lbcrypto::DCRTPoly>& pk,
                                    FIDESlib::CKKS::Context& GPUcc,
                                    std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>>& ct_tokens,
                                    PtWeights_GPU& weights_layer0,
                                    PtWeights_GPU& weights_layer1,
                                    PtMasks_GPU& masks,
                                    MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
                                    TransposePrecomputations_GPU& Tprecomp_gpu,
                                    lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& cc,
                                    lbcrypto::PrivateKey<lbcrypto::DCRTPoly>& sk,
                                    int test_case) {

        std::ifstream file(file_path);
        std::string line;

        if (!file.is_open()) {
            std::cerr << "ERROR: Could not open file: " << file_path << std::endl;
            return;
        }

        size_t total_counter = 0;
        size_t correct_counter = 0;

        while (std::getline(file, line)) {
            if (line.empty()) continue;

            std::string sentence;
            int label = -1;

            if (line.front() == '"') {
                size_t end_quote = line.find("\",");
                if (end_quote == std::string::npos) continue;
                sentence = line.substr(1, end_quote - 1);

                size_t label_start = end_quote + 2;
                size_t next_comma = line.find(',', label_start);
                if (next_comma != std::string::npos)
                    label = std::stoi(line.substr(label_start, next_comma - label_start));
            } else {
                size_t comma1 = line.find(',');
                if (comma1 == std::string::npos) continue;
                sentence = line.substr(0, comma1);

                size_t comma2 = line.find(',', comma1 + 1);
                if (comma2 != std::string::npos)
                    label = std::stoi(line.substr(comma1 + 1, comma2 - comma1 - 1));
            }

            if (sentence.empty() || label == -1) continue;

            EncoderConfiguration conf = base_conf;
            conf.token_length = tokenizer(sentence, model_name, model_path, output_file);

            std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>> ct_tokens_clone;
            ct_tokens_clone.resize(ct_tokens.size());
            for (int i = 0; i < ct_tokens_clone.size(); i++){
                ct_tokens_clone[i].resize(ct_tokens[i].size());
                for (int j = 0; j < ct_tokens_clone[i].size(); j++){
                    ct_tokens_clone[i][j] = ct_tokens[i][j]->Clone();
                }
            }

            std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> tokens_gpu;
            encryptMatrixtoGPU(model_path + "/" + output_file, tokens_gpu, pk, GPUcc,
                            conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul);

            // file output
            std::ofstream outFile(output_path, std::ios::app);
            outFile << std::endl << "///////////////////////////////////////" << std::endl;
            outFile << "Input: '" << sentence << "'" << ", " << conf.token_length << std::endl << "Label: " << label << std::endl;
            outFile.close();

            // terminal output
            std::cout << std::endl << "///////////////////////////////////////" << std::endl;
            std::cout << "Input: '" << sentence << "'" << ", " << conf.token_length <<  std::endl << "Label: " << label << std::endl;

            hipDeviceSynchronize();
            auto start_gpu = std::chrono::high_resolution_clock::now();
            tokens_gpu = encoder_helmet(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 0, test_case);
            tokens_gpu = encoder_helmet(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 1, test_case);
            uint32_t class_pred = classifier(cc, tokens_gpu, sk, ct_tokens_clone, precomp_gpu, weights_layer1,
                                            masks, conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
            hipDeviceSynchronize();
            auto end_gpu = std::chrono::high_resolution_clock::now();

            tokens_gpu.clear();

            total_counter++;
            if (class_pred == static_cast<uint32_t>(label))
                correct_counter++;


            std::ofstream outFile2(output_path, std::ios::app);
            outFile2 << "took: " 
                    << std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count()
                    << " ms." << std::endl;
            outFile2 << "Accuracy: " << correct_counter << "/" << total_counter << std::endl;
            outFile2.close();
            // terminal output
            std::cout << "Accuracy: " << correct_counter << "/" << total_counter << std::endl;

        }
    }

static inline void ltrim(std::string& s) { s.erase(s.begin(), std::find_if(s.begin(), s.end(), [](int ch){ return !std::isspace(ch); })); }
static inline void rtrim(std::string& s) { s.erase(std::find_if(s.rbegin(), s.rend(), [](int ch){ return !std::isspace(ch); }).base(), s.end()); }
static inline void trim(std::string& s) { ltrim(s); rtrim(s); }

// data: <sentence ending at first '.'><comma><label><comma><optional id or rest>
void process_sentences_from_csv_cola(std::string& file_path,
                                    std::string& output_file,
                                    std::string& model_name,
                                    std::string& model_path,
                                    std::string& output_path,
                                    EncoderConfiguration& base_conf,
                                    lbcrypto::PublicKey<lbcrypto::DCRTPoly>& pk,
                                    FIDESlib::CKKS::Context& GPUcc,
                                    std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>>& ct_tokens,
                                    PtWeights_GPU& weights_layer0,
                                    PtWeights_GPU& weights_layer1,
                                    PtMasks_GPU& masks,
                                    MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
                                    TransposePrecomputations_GPU& Tprecomp_gpu,
                                    lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& cc,
                                    lbcrypto::PrivateKey<lbcrypto::DCRTPoly>& sk,
                                    int test_case) {

        std::ifstream file(file_path);
        std::string line;

        if (!file.is_open()) {
            std::cerr << "ERROR: Could not open file: " << file_path << std::endl;
            return;
        }

        size_t total_counter = 0;
        size_t correct_counter = 0;

        while (std::getline(file, line)) {
            if (line.empty()) continue;

            std::string sentence;
            int label = -1;

            // --- Parse by first '.' as end-of-sentence ---
            // 1) find first '.', include it in the sentence
            size_t dot_pos = line.find('.');
            if (dot_pos == std::string::npos) {
                // No period -> malformed line; skip
                continue;
            }
            sentence = line.substr(0, dot_pos + 1);

            // 2) after the '.', there should be a comma separating the label
            size_t after_dot = dot_pos + 1;
            // skip spaces if any (e.g., ". ,1,260")
            while (after_dot < line.size() && std::isspace(static_cast<unsigned char>(line[after_dot]))) ++after_dot;

            if (after_dot >= line.size() || line[after_dot] != ',') {
                // Expect a comma right after sentence end (allowing spaces). If not present, skip.
                continue;
            }

            size_t label_start = after_dot + 1;
            // skip spaces before label
            while (label_start < line.size() && std::isspace(static_cast<unsigned char>(line[label_start]))) ++label_start;

            // label ends at next comma or end of line
            size_t label_end = line.find(',', label_start);
            std::string label_str = (label_end == std::string::npos)
                                    ? line.substr(label_start)
                                    : line.substr(label_start, label_end - label_start);
            trim(label_str);

            try {
                label = std::stoi(label_str);
            } catch (...) {
                // Non-integer label -> skip
                continue;
            }

            trim(sentence);
            if (sentence.empty() || label == -1) continue;

            EncoderConfiguration conf = base_conf;
            conf.token_length = tokenizer(sentence, model_name, model_path, output_file);

            std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>> ct_tokens_clone;
            ct_tokens_clone.resize(ct_tokens.size());
            for (int i = 0; i < ct_tokens_clone.size(); i++){
                ct_tokens_clone[i].resize(ct_tokens[i].size());
                for (int j = 0; j < ct_tokens_clone[i].size(); j++){
                    ct_tokens_clone[i][j] = ct_tokens[i][j]->Clone();
                }
            }

            std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> tokens_gpu;
            encryptMatrixtoGPU(model_path + "/" + output_file, tokens_gpu, pk, GPUcc,
                            conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul);

            // file output
            std::ofstream outFile(output_path, std::ios::app);
            outFile << std::endl << "///////////////////////////////////////" << std::endl;
            outFile << "Input: '" << sentence << "'" << std::endl << "Label: " << label << std::endl;
            outFile.close();

            // terminal output
            std::cout << std::endl << "///////////////////////////////////////" << std::endl;
            std::cout << "Input: '" << sentence << "'" << std::endl << "Label: " << label << std::endl;

            hipDeviceSynchronize();
            auto start_gpu = std::chrono::high_resolution_clock::now();
            tokens_gpu = encoder_helmet(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 0, test_case);
            tokens_gpu = encoder_helmet(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 1, test_case);
            uint32_t class_pred = classifier(cc, tokens_gpu, sk, ct_tokens_clone, precomp_gpu, weights_layer1,
                                            masks, conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
            hipDeviceSynchronize();
            auto end_gpu = std::chrono::high_resolution_clock::now();

            tokens_gpu.clear();

            total_counter++;
            if (class_pred == static_cast<uint32_t>(label))
                correct_counter++;


            std::ofstream outFile2(output_path, std::ios::app);
            outFile2 << "took: " 
                    << std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count()
                    << " ms." << std::endl;
            outFile2 << "Accuracy: " << correct_counter << "/" << total_counter << std::endl;
            outFile2.close();
            // terminal output
            std::cout << "Accuracy: " << correct_counter << "/" << total_counter << std::endl;

        }
    }


    int32_t classifier(lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& context, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& input, lbcrypto::PrivateKey<lbcrypto::DCRTPoly>& privateKey, 
                                std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>>& ct_tokens, const MatrixMatrixProductPrecomputations_GPU& precomp, PtWeights_GPU& weights_layer, 
                                PtMasks_GPU& masks, int numSlots, int blockSize, int token_length, bool bts, std::string& output_path){

        bool constexpr print = false;

        FIDESlib::CKKS::Context& GPUcc = input[0][0].cc;

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> result, result_f;                                
        PCMM_2(input, weights_layer.Wp, blockSize, result, precomp, weights_layer.bp, masks.row_masks[token_length]);

        if constexpr (print) std::cout << "# limbs: " << result[0][0].getLevel() << " " << result[0][0].NoiseLevel << std::endl;
        if (print) printMatrix(decryptGPUMatrix(result, keys_.secretKey, ct_tokens, numSlots, blockSize), 2, 2, "PCMM", false);

        evalTanh(result[0][0], GPUcc.GetEvalKey(), numSlots, -20, 20, true); // -10, 10 -> -20, 20

        if constexpr (print) std::cout << "# limbs: " << result[0][0].getLevel() << " " << result[0][0].NoiseLevel << std::endl;
        if (print) printMatrix(decryptGPUMatrix(result, keys_.secretKey, ct_tokens, numSlots, blockSize), 2, 2, "Tanh", false);

        FIDESlib::CKKS::Ciphertext result_0(GPUcc), result_1(GPUcc);
        result_0.copy(result[0][0]);
        result_0.multPt(weights_layer.Wc[0][0], false);
        Accumulate(result_0, 4, 1, blockSize);
        result_0.addPt(weights_layer.bc[0][0]);

        FIDESlib::CKKS::RawCipherText raw_res;
        result_0.store(GPUcc, raw_res);
        auto result_gpu0(ct_tokens[0][0]->Clone());
        GetOpenFHECipherText(result_gpu0, raw_res);

        Plaintext weights_rotated(GPUcc), bias_rotated(GPUcc);
        weights_rotated.copy(weights_layer.Wc[0][0]);
        weights_rotated.automorph(blockSize);
        bias_rotated.copy(weights_layer.bc[0][0]);
        bias_rotated.automorph(blockSize);

        result_1.copy(result[0][0]);
        result_1.multPt(weights_rotated, false);
        Accumulate(result_1, 4, 1, blockSize);
        result_1.addPt(bias_rotated);

        FIDESlib::CKKS::RawCipherText raw_res1;
        result_1.store(GPUcc, raw_res1);
        auto result_gpu1(ct_tokens[0][0]->Clone());
        GetOpenFHECipherText(result_gpu1, raw_res1);

        try {
            lbcrypto::Plaintext pt_result_gpu0;
            context->Decrypt(privateKey, result_gpu0, &pt_result_gpu0);
            double result0 = pt_result_gpu0->GetRealPackedValue()[0];

            lbcrypto::Plaintext pt_result_gpu1;
            context->Decrypt(privateKey, result_gpu1, &pt_result_gpu1);
            double result1 = pt_result_gpu1->GetRealPackedValue()[0];

            int yhat = 1;
            if (result0 > result1) {
                yhat = 0;
            }

            std::ofstream outFile(output_path, std::ios::app);
            outFile << "logits: " << result0 << ", " << result1 << std::endl;
            outFile << "Class: " << yhat << std::endl; 
            outFile.close();

            // terminal output
            std::cout << "logits: " << result0 << ", " << result1 << std::endl;
            std::cout << "Class: " << yhat << std::endl; 
            return yhat;

        } catch (const std::exception& e) {
            // std::cerr << "none. Decryption failed: " << e.what() << std::endl;
            return -1;
        } catch (...) {
            // std::cerr << "none. Unknown error occurred during decryption." << std::endl;
            return -1;
        }
        std::cout << std::endl;
    }

    std::vector<int> GenerateRotationIndices_GPU(int blockSize, int bStep, int bStepAcc, int num_heads){

            // JKLS MatMul rotation indices
            std::vector<int32_t> rotation_indices_MM = GenerateMatMulRotationIndices_GPU(blockSize, bStep);
            // Multi-head Attention rotation indices
            std::vector<int32_t> rotation_indices_MHA = GenerateMatMulRotationIndices_GPU(blockSize / num_heads, bStep);

            // Transpose rotation indices
            std::vector<int> rotation_indices_T = GenerateTransposeRotationIndices_GPU(blockSize, bStep);

            std::vector<int> rotsum_indices = {1, 2, 3, 4, 8, 16, 32, 64, 8192, 0, -1, -2, -3, -4, -8, -16, -32, -64, 127, -15, -31, -47, -63, -127}; // 127 is for pooling, -blockSize for Concat

            std::vector<int> accum_indices = FIDESlib::CKKS::GetAccumulateRotationIndices(bStepAcc, 1, blockSize);
            std::vector<int> accum_indices2 = FIDESlib::CKKS::GetAccumulateRotationIndices(bStepAcc, blockSize, blockSize);
            std::vector<int> accum_indices3 = FIDESlib::CKKS::GetAccumulateRotationIndices(bStepAcc, blockSize, blockSize/2);
            std::vector<int> accum_indices4 = FIDESlib::CKKS::GetAccumulateRotationIndices(bStepAcc, blockSize, blockSize/4);
            std::vector<int> broad_indices =  FIDESlib::CKKS::GetbroadcastRotationIndices(bStepAcc, 1, blockSize);
            std::vector<int> broad_indices2 =  FIDESlib::CKKS::GetbroadcastRotationIndices(bStepAcc, 1, blockSize/2);
            std::vector<int> broad_indices3 =  FIDESlib::CKKS::GetbroadcastRotationIndices(bStepAcc, 1, blockSize/4);
            std::vector<int> broad_indices4 =  FIDESlib::CKKS::GetbroadcastRotationIndices(bStepAcc, 1, blockSize/8);
            std::vector<int> broad_indices5 =  FIDESlib::CKKS::GetbroadcastRotationIndices(bStepAcc, 1, blockSize/16);  // 4 
            std::vector<int> broad_indices6 =  FIDESlib::CKKS::GetbroadcastRotationIndices(bStepAcc, blockSize, blockSize*blockSize);
            
            // if (blockSize == 128) rotsum_indices = {128, 256, 512, 1024, 2048, 4096, 8192, 16384};

            // Merge the rotation indices and remove duplicates

            std::set<int32_t> merged_set(rotsum_indices.begin(), rotsum_indices.end());
            merged_set.insert(rotation_indices_MM.begin(), rotation_indices_MM.end());
            merged_set.insert(rotation_indices_MHA.begin(), rotation_indices_MHA.end());
            merged_set.insert(rotation_indices_T.begin(), rotation_indices_T.end());
            merged_set.insert(accum_indices.begin(), accum_indices.end());
            merged_set.insert(accum_indices2.begin(), accum_indices2.end());
            merged_set.insert(accum_indices3.begin(), accum_indices3.end());
            merged_set.insert(accum_indices4.begin(), accum_indices4.end());
            merged_set.insert(broad_indices.begin(), broad_indices.end());
            merged_set.insert(broad_indices2.begin(), broad_indices2.end());
            merged_set.insert(broad_indices3.begin(), broad_indices3.end());
            merged_set.insert(broad_indices4.begin(), broad_indices4.end());
            merged_set.insert(broad_indices5.begin(), broad_indices5.end());
            merged_set.insert(broad_indices6.begin(), broad_indices6.end());
            std::vector<int32_t> rotation_indices(merged_set.begin(), merged_set.end());

            return rotation_indices;
    }

    void MatrixBootstrap(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, int numSlots, bool input_prescaled) {
        for (size_t i = 0; i < matrix.size(); i++) {
            for (size_t j = 0; j < matrix[0].size(); j++) {
                Bootstrap(matrix[i][j], numSlots, input_prescaled);
            }
        }
    }

    void MatrixSquare(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, const KeySwitchingKey& keySwitchingKey){
        for (size_t i=0; i<matrix.size(); i++){
                for (size_t j=0; j<matrix[0].size(); j++){
                    matrix[i][j].mult(matrix[i][j], matrix[i][j], keySwitchingKey);
                }
            }
    }


    void MatrixMultScalar(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, double scale){
        for (size_t i=0; i<matrix.size(); i++){
                for (size_t j=0; j<matrix[0].size(); j++){
                    matrix[i][j].multScalar(scale);
                }
            }
    }

    void MatrixAddScalar(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, double value){
        for (size_t i=0; i<matrix.size(); i++){
                for (size_t j=0; j<matrix[0].size(); j++){
                    matrix[i][j].addScalar(value);
                }
            }
    }

    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> MatrixAdd(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix2){

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> masked_matrix;
        masked_matrix.reserve(matrix.size());
        for (size_t i=0; i<matrix.size(); i++){
            std::vector<FIDESlib::CKKS::Ciphertext> row;
            row.reserve(matrix[0].size());
            for (size_t j=0; j<matrix[0].size(); j++){
                FIDESlib::CKKS::Ciphertext masked_ct(matrix[i][j].cc);
                masked_ct.copy(matrix[i][j]);
                masked_ct.add(matrix2[i][j]);
                row.emplace_back(std::move(masked_ct));
            }
            masked_matrix.emplace_back(std::move(row));
        }
        return masked_matrix;
    }

    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> MatrixMaskSpecial(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, PtMasks_GPU& masks, double scalar){

        FIDESlib::CKKS::Context& cc =  matrix[0][0].cc;
        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> masked_matrix;
        masked_matrix.reserve(matrix.size());
        for (size_t i=0; i<matrix.size(); i++){
            std::vector<FIDESlib::CKKS::Ciphertext> row;
            row.reserve(matrix[0].size());
            for (size_t j=0; j<matrix[0].size(); j++){
                FIDESlib::CKKS::Plaintext mask_temp(cc);
                mask_temp.copy(masks.mask_max); // 1s
                mask_temp.multScalar(scalar, true);          // Subtract except 1st zero

                FIDESlib::CKKS::Ciphertext masked_ct(cc);
                masked_ct.copy(matrix[i][j]);
                masked_ct.addPt(mask_temp);
                row.emplace_back(std::move(masked_ct));
            }
            masked_matrix.emplace_back(std::move(row));
        }
        return masked_matrix;
    }

    void MatrixRotate(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, int index){
        FIDESlib::CKKS::Context& cc = matrix[0][0].cc;
        for (size_t i=0; i<matrix.size(); i++){
                for (size_t j=0; j<matrix[0].size(); j++){
                    matrix[i][j].rotate(index, cc.GetRotationKey(index));
                }
            }
    }
    

    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> MatrixMask(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, FIDESlib::CKKS::Plaintext& mask){

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> masked_matrix;
        masked_matrix.reserve(matrix.size());
        for (size_t i=0; i<matrix.size(); i++){
            std::vector<FIDESlib::CKKS::Ciphertext> row;
            row.reserve(matrix[0].size());
            for (size_t j=0; j<matrix[0].size(); j++){
                FIDESlib::CKKS::Ciphertext masked_ct(matrix[i][j].cc);
                masked_ct.copy(matrix[i][j]);
                masked_ct.multPt(mask);
                // masked_ct.rescale();
                row.emplace_back(std::move(masked_ct));
            }
            masked_matrix.emplace_back(std::move(row));
        }
        return masked_matrix;
    }

    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> MatrixConcat(std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>>& matrices, 
                                                                    std::vector<FIDESlib::CKKS::Plaintext>& masks, int blockSize){

        if (matrices.empty() || masks.size() != matrices.size())
            throw std::invalid_argument("Matrix and mask size mismatch or empty input.");

        const size_t numRows = matrices[0].size();
        const size_t numCols = matrices[0][0].size();

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> result;
        result.reserve(numRows);

        for (size_t i = 0; i < numRows; ++i) {
            std::vector<FIDESlib::CKKS::Ciphertext> row;
            row.reserve(numCols);

            for (size_t j = 0; j < numCols; ++j) {
                // Initialize with a copy of the first masked ciphertext
                FIDESlib::CKKS::Ciphertext sum(matrices[0][i][j].cc);
                sum.copy(matrices[0][i][j]);
                sum.multPt(masks[0]);

                // Accumulate masked ciphertexts from remaining matrices
                for (size_t k = 1; k < matrices.size(); ++k) {
                    FIDESlib::CKKS::Ciphertext tmp(matrices[k][i][j].cc);
                    tmp.copy(matrices[k][i][j]);
                    tmp.multPt(masks[k]);
                    sum.add(tmp);
                }

                row.emplace_back(std::move(sum));
            }
            result.emplace_back(std::move(row));
        }
        return result;
    }

    void MatrixMaskCT(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& matrix, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& mask, const KeySwitchingKey& keySwitchingKey){
        for (size_t i=0; i<matrix.size(); i++){
                for (size_t j=0; j<matrix[0].size(); j++){
                    matrix[i][j].mult(matrix[i][j], mask[i][j], keySwitchingKey);
                }
            }
    }

    void dropMatrixLevel(std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& in, int level) {
        for (auto& row : in)
            for (auto& ct : row) {
                if (ct.NoiseLevel == 2)
                    ct.rescale();
                if (ct.getLevel() > level) {
                    ct.dropToLevel(level);
                    assert(ct.getLevel() == level);
                }
            }
        }

    int tokenizer(const std::string& sentence, const std::string& model_name,
                const std::string& model_path, const std::string& output_filename) {
        std::string script_path = "/projectnb/he/seyda/FIDESlib/src/python/";
        std::string script;

        if (output_filename == "tokens1.txt")
            script = "ExtractEmbeddings1.py";
        else if (output_filename == "tokens2.txt")
            script = "ExtractEmbeddings2.py";
        else if (output_filename == "tokens3.txt")
            script = "ExtractEmbeddings3.py";
        else if (output_filename == "tokens4.txt")
            script = "ExtractEmbeddings4.py";
        else if (output_filename == "tokens5.txt")
            script = "ExtractEmbeddings5.py";

        else if (output_filename == "tokens_test1.txt")
            script = "ExtractEmbeddings_test1.py";
        else if (output_filename == "tokens_test2.txt")
            script = "ExtractEmbeddings_test2.py";
        else if (output_filename == "tokens_test3.txt")
            script = "ExtractEmbeddings_test3.py";
        else if (output_filename == "tokens_test4.txt")
            script = "ExtractEmbeddings_test4.py";
        else if (output_filename == "tokens_test5.txt")
            script = "ExtractEmbeddings_test5.py";

        else if (output_filename == "tokens_cola1.txt")
            script = "ExtractEmbeddings_cola1.py";
        else if (output_filename == "tokens_cola2.txt")
            script = "ExtractEmbeddings_cola2.py";
        else if (output_filename == "tokens_cola3.txt")
            script = "ExtractEmbeddings_cola3.py";
        else if (output_filename == "tokens_cola4.txt")
            script = "ExtractEmbeddings_cola4.py";

        else if (output_filename == "tokens_mrpc1.txt" )
            script = "ExtractEmbeddings_mrpc1.py";
        else if (output_filename == "tokens_mrpc2.txt")
            script = "ExtractEmbeddings_mrpc2.py";
        else if (output_filename == "tokens_mrpc3.txt")
            script = "ExtractEmbeddings_mrpc3.py";
        else if (output_filename == "tokens_mrpc4.txt")
            script = "ExtractEmbeddings_mrpc4.py";

        else if (output_filename == "tokens_rte1.txt" )
            script = "ExtractEmbeddings_rte1.py";
        else if (output_filename == "tokens_rte2.txt")
            script = "ExtractEmbeddings_rte2.py";
        else if (output_filename == "tokens_rte3.txt")
            script = "ExtractEmbeddings_rte3.py";
        else if (output_filename == "tokens_rte4.txt")
            script = "ExtractEmbeddings_rte4.py";


        else
            script = "ExtractEmbeddings.py";

        std::string cmd = "python3 " + script_path + script + " \"" +
                        sentence + "\" \"" + model_name + "\" \"" + model_path + "\" \"" +
                        output_filename + "\"";

        std::array<char, 128> buffer;
        std::string result;
        FILE* pipe = popen(cmd.c_str(), "r");
        if (!pipe)
            throw std::runtime_error("popen() failed to run the tokenizer script");

        while (fgets(buffer.data(), buffer.size(), pipe) != nullptr) {
            result += buffer.data();
        }

        int exitCode = pclose(pipe);
        if (exitCode != 0) {
            throw std::runtime_error("Tokenizer script failed with exit code: " + std::to_string(exitCode));
        }

        try {
            return std::stoi(result);
        } catch (const std::exception& e) {
            std::cerr << "[WARNING] Failed to parse token count from script output: \"" << result << "\"" << std::endl;
            return 0;
        }
    }

    // New: escape double-quotes to keep popen() safe enough for our usage.
    static std::string shell_escape_quotes(std::string s) {
        std::string t;
        t.reserve(s.size());
        for (char c : s) t += (c == '"' ? "\\\"" : std::string(1, c));
        return t;
    }

    int tokenizer_pair(const std::string& sentence1,
                    const std::string& sentence2,
                    const std::string& model_name,
                    const std::string& output_path,
                    const std::string& output_filename) {
        const std::string script_path = "/projectnb/he/seyda/FIDESlib/src/python/";

        const std::string script =
            (model_name == "bert-tiny-mrpc") ? "ExtractEmbeddings_pair.py" :
            (model_name == "bert-tiny-rte")  ? "ExtractEmbeddings_pair_rte.py" :
                                            "";
        const std::string s1 = shell_escape_quotes(sentence1);
        const std::string s2 = shell_escape_quotes(sentence2);
        const std::string m  = shell_escape_quotes(model_name);
        const std::string op = shell_escape_quotes(output_path);
        const std::string of = shell_escape_quotes(output_filename);

        const std::string cmd = "python3 " + script_path + script + " \"" +
                                s1 + "\" \"" + s2 + "\" \"" + m + "\" \"" + op + "\" \"" + of + "\"";

        std::array<char, 128> buffer{};
        std::string result;
        FILE* pipe = popen(cmd.c_str(), "r");
        if (!pipe) throw std::runtime_error("popen() failed to run the tokenizer_pair script");

        while (fgets(buffer.data(), buffer.size(), pipe) != nullptr) {
            result += buffer.data();
        }
        int exitCode = pclose(pipe);
        if (exitCode != 0) {
            throw std::runtime_error("tokenizer_pair script failed, exit code: " + std::to_string(exitCode));
        }

        try {
            return std::stoi(result); // returns seq_len (including special tokens)
        } catch (...) {
            std::cerr << "[WARNING] Failed to parse token count from script output: \"" << result << "\"\n";
            return 0;
        }
    }
    size_t CountNumTokens(const std::string& file_path) {
        std::ifstream file(file_path);
        if (!file.is_open()) {
            std::cerr << "Error: Could not open file " << file_path << std::endl;
            return 0;
        }

        size_t count = 0;
        std::string line;
        while (std::getline(file, line)) {
            // Trim leading/trailing whitespace
            line.erase(0, line.find_first_not_of(" \t\n\r\f\v"));
            line.erase(line.find_last_not_of(" \t\n\r\f\v") + 1);

            if (!line.empty()) {
                ++count;
            }
        }

        file.close();
        return count;
    }


    // TOKENIZER HELPERS

    // --- helpers: trim / strip ---
    static inline void lstrip(std::string& s){
        size_t i = 0; while (i < s.size() && std::isspace(static_cast<unsigned char>(s[i]))) ++i;
        if (i) s.erase(0, i);
    }
    static inline void rstrip(std::string& s){
        size_t i = s.size(); while (i > 0 && std::isspace(static_cast<unsigned char>(s[i-1]))) --i;
        if (i < s.size()) s.erase(i);
    }
    static inline void strip(std::string& s){ rstrip(s); lstrip(s); }

    // --- minimal RFC4180 CSV row parser (returns columns) ---
    static bool parse_csv_row(const std::string& line, std::vector<std::string>& outCols) {
        outCols.clear();
        std::string field;
        bool inQuotes = false;
        for (size_t i = 0; i < line.size(); ++i) {
            char c = line[i];
            if (inQuotes) {
                if (c == '"') {
                    // doubled quote -> literal quote
                    if (i + 1 < line.size() && line[i + 1] == '"') {
                        field.push_back('"');
                        ++i;
                    } else {
                        inQuotes = false;
                    }
                } else {
                    field.push_back(c);
                }
            } else {
                if (c == ',') {
                    outCols.push_back(field);
                    field.clear();
                } else if (c == '"') {
                    // opening quote only valid at field start
                    if (field.empty()) inQuotes = true;
                    else field.push_back('"'); // treat as literal if mid-field
                } else {
                    field.push_back(c);
                }
            }
        }
        outCols.push_back(field);
        return true;
    }
// Collapses spaces, fixes tokenization artifacts, and cleans quotes at boundaries.
static void normalize_tokenized_punct(std::string& s) {
    auto is_space = [](char c){ return std::isspace(static_cast<unsigned char>(c)); };

    // 0) Normalize newlines/tabs to single spaces
    for (char& c : s) if (c == '\t' || c == '\n' || c == '\r') c = ' ';

    // 1) Collapse consecutive spaces
    {
        std::string t; t.reserve(s.size());
        bool prevSpace = false;
        for (char c : s) {
            bool sp = is_space(c);
            if (!(sp && prevSpace)) t.push_back(sp ? ' ' : c);
            prevSpace = sp;
        }
        s.swap(t);
    }

    // 2) Trim outer spaces first
    auto ltrim = [&](std::string& x){
        size_t i = 0; while (i < x.size() && is_space(x[i])) ++i;
        if (i) x.erase(0, i);
    };
    auto rtrim = [&](std::string& x){
        size_t i = x.size(); while (i > 0 && is_space(x[i-1])) --i;
        if (i < x.size()) x.erase(i);
    };
    ltrim(s); rtrim(s);

    // 3) Drop *leading* noise: quotes, commas, spaces (handles: "" The ... ,  ,  " word)
    while (!s.empty() && (s.front() == '"' || s.front() == '\'' || s.front() == ',' || is_space(s.front())))
        s.erase(s.begin());
    // 4) Drop *trailing* noise: quotes, commas, spaces
    while (!s.empty() && (s.back() == '"' || s.back() == '\'' || s.back() == ',' || is_space(s.back())))
        s.pop_back();

    // 5) Remove spaces immediately BEFORE punctuation: "word ,", "word .", etc.
    {
        std::string t; t.reserve(s.size());
        for (size_t i = 0; i < s.size(); ++i) {
            if (s[i] == ' ' && i + 1 < s.size()) {
                char p = s[i + 1];
                if (p == ',' || p == '.' || p == '!' || p == '?' || p == ';' || p == ':') continue; // skip this space
            }
            t.push_back(s[i]);
        }
        s.swap(t);
    }

    // 6) Remove spaces around apostrophes: "doesn ' t" -> "doesn't"
    {
        std::string t; t.reserve(s.size());
        for (size_t i = 0; i < s.size(); ++i) {
            if (s[i] == ' ' && ((i + 1 < s.size() && s[i + 1] == '\'') || (i > 0 && s[i - 1] == '\''))) {
                continue; // drop space adjacent to apostrophe
            }
            t.push_back(s[i]);
        }
        s.swap(t);
    }

    // 7) Tighten spaces right *inside* quotes:
    //    "\" word"  -> "\"word"
    //    "word \""  -> "word\""
    {
        // after opening quote
        std::string t; t.reserve(s.size());
        for (size_t i = 0; i < s.size(); ++i) {
            t.push_back(s[i]);
            if ((s[i] == '"' || s[i] == '\'') && i + 1 < s.size() && s[i + 1] == ' ') {
                // skip all spaces immediately after quote
                size_t j = i + 1;
                while (j < s.size() && s[j] == ' ') ++j;
                if (j < s.size()) t.push_back(s[j]);
                i = j;
            }
        }
        s.swap(t);

        // before closing quote
        std::string u; u.reserve(s.size());
        for (size_t i = 0; i < s.size(); ++i) {
            if (s[i] == ' ' && i + 1 < s.size() && (s[i + 1] == '"' || s[i + 1] == '\'')) continue; // drop space
            u.push_back(s[i]);
        }
        s.swap(u);
    }

    // 8) Final trim (in case step 3/4 created new edges)
    ltrim(s); rtrim(s);
}


    void process_sentences_from_csv_mrpc(std::string& file_path,
                                    std::string& output_file,
                                    std::string& model_name,
                                    std::string& model_path,
                                    std::string& output_path,
                                    EncoderConfiguration& base_conf,
                                    lbcrypto::PublicKey<lbcrypto::DCRTPoly>& pk,
                                    FIDESlib::CKKS::Context& GPUcc,
                                    std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>>& ct_tokens,
                                    PtWeights_GPU& weights_layer0,
                                    PtWeights_GPU& weights_layer1,
                                    PtMasks_GPU& masks,
                                    MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
                                    TransposePrecomputations_GPU& Tprecomp_gpu,
                                    lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& cc,
                                    lbcrypto::PrivateKey<lbcrypto::DCRTPoly>& sk,
                                    int test_case) {

        std::ifstream file(file_path);
        std::string line;

        if (!file.is_open()) {
            std::cerr << "ERROR: Could not open file: " << file_path << std::endl;
            return;
        }

        size_t total_counter = 0;
        size_t correct_counter = 0;

        std::vector<std::string> cols;
        
        while (std::getline(file, line)) {
            rstrip(line);
            if (line.empty()) continue;

            if (!parse_csv_row(line, cols)) continue;
            if (cols.size() < 4) continue;

            std::string s1 = cols[0];
            std::string s2 = cols[1];
            std::string label_str = cols[2];
            std::string idx_str = cols[3];

            strip(s1); strip(s2); strip(label_str); strip(idx_str);
            normalize_tokenized_punct(s1);
            normalize_tokenized_punct(s2);

            int label;
            try { label = std::stoi(label_str); }
            catch (...) { continue; }

            // **pair** tokenization for MRPC!
            EncoderConfiguration conf = base_conf;
            conf.token_length = tokenizer_pair(s1, s2, model_name, model_path, output_file);

            std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>> ct_tokens_clone;
            ct_tokens_clone.resize(ct_tokens.size());
            for (int i = 0; i < ct_tokens_clone.size(); i++){
                ct_tokens_clone[i].resize(ct_tokens[i].size());
                for (int j = 0; j < ct_tokens_clone[i].size(); j++){
                    ct_tokens_clone[i][j] = ct_tokens[i][j]->Clone();
                }
            }

            std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> tokens_gpu;
            int tokens_size = 1 << static_cast<int>(std::ceil(std::log2(conf.token_length)));
            encryptMatrixtoGPU(model_path + "/" + output_file, tokens_gpu, pk, GPUcc,
                            conf.numSlots, conf.blockSize, tokens_size, conf.cols, conf.level_matmul);

            // file output
            {
                std::ofstream outFile(output_path, std::ios::app);
                if (!outFile) {
                    std::cerr << "[ERROR] Could not open file for writing: " << output_path << "\n";
                } else {
                    outFile << "\n///////////////////////////////////////\n";
                    outFile << "Sentence1: \"" << s1 << "\"\n";
                    outFile << "Sentence2: \"" << s2 << "\"\n";
                    outFile << "Label: " << label << "\n";
                    outFile << "Length: " << conf.token_length << "\n";
                }
            }

            // terminal output
            std::cout << "\n///////////////////////////////////////\n";
            std::cout << "Sentence1: \"" << s1 << "\"\n";
            std::cout << "Sentence2: \"" << s2 << "\"\n";
            std::cout << "Label: " << label << "\n";

            hipDeviceSynchronize();
            auto start_gpu = std::chrono::high_resolution_clock::now();
            tokens_gpu = encoder_helmet(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 0, test_case);
            tokens_gpu = encoder_helmet(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 1, test_case);
            uint32_t class_pred = classifier(cc, tokens_gpu, sk, ct_tokens_clone, precomp_gpu, weights_layer1,
                                            masks, conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
            hipDeviceSynchronize();
            auto end_gpu = std::chrono::high_resolution_clock::now();

            tokens_gpu.clear();

            total_counter++;
            if (class_pred == static_cast<uint32_t>(label))
                correct_counter++;


            std::ofstream outFile2(output_path, std::ios::app);
            outFile2 << "took: " 
                    << std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count()
                    << " ms." << std::endl;
            outFile2 << "Accuracy: " << correct_counter << "/" << total_counter << std::endl;
            outFile2.close();
            // terminal output
            std::cout << "Accuracy: " << correct_counter << "/" << total_counter << std::endl;

        }
    }


// --- Local helpers (RTE-only) ------------------------------------------------
static inline void ltrim_rte(std::string& s){
    size_t i = 0; while (i < s.size() && std::isspace(static_cast<unsigned char>(s[i]))) ++i;
    if (i) s.erase(0, i);
}
static inline void rtrim_rte(std::string& s){
    size_t i = s.size(); while (i > 0 && std::isspace(static_cast<unsigned char>(s[i-1]))) --i;
    if (i < s.size()) s.erase(i);
}
static inline void trim_rte(std::string& s){ ltrim_rte(s); rtrim_rte(s); }

// RFC-4180 CSV parser (RTE-local)
static bool csv_parse_row_rte(const std::string& line, std::vector<std::string>& cols){
    cols.clear();
    std::string f; f.reserve(line.size());
    bool inq = false;
    for (size_t i = 0; i < line.size(); ++i){
        const char c = line[i];
        if (inq){
            if (c == '"'){
                if (i + 1 < line.size() && line[i+1] == '"'){ f.push_back('"'); ++i; }
                else { inq = false; }
            } else { f.push_back(c); }
        } else {
            if (c == '"') inq = true;
            else if (c == ','){ cols.push_back(f); f.clear(); }
            else f.push_back(c);
        }
    }
    cols.push_back(f);
    return !inq;
}

// Conservative punctuation normalization (RTE-local)
static void normalize_punct_rte(std::string& s){
    std::string out; out.reserve(s.size());
    for (size_t i = 0; i < s.size(); ++i){
        // remove space before closing punctuation
        if (i + 1 < s.size() && s[i] == ' ' &&
            (s[i+1]=='.' || s[i+1]==',' || s[i+1]==';' || s[i+1]==':' ||
             s[i+1]=='!' || s[i+1]=='?' || s[i+1]==')' || s[i+1]==']' || s[i+1]=='}')){
            continue;
        }
        // remove space after opening punctuation
        if (i + 1 < s.size() && (s[i]=='(' || s[i]=='[' || s[i]=='{') && s[i+1]==' '){
            out.push_back(s[i]); ++i; continue;
        }
        out.push_back(s[i]);
    }
    s.swap(out);
}
// -----------------------------------------------------------------------------

void process_sentences_from_csv_rte(std::string& file_path,
                                std::string& output_file,
                                std::string& model_name,
                                std::string& model_path,
                                std::string& output_path,
                                EncoderConfiguration& base_conf,
                                lbcrypto::PublicKey<lbcrypto::DCRTPoly>& pk,
                                FIDESlib::CKKS::Context& GPUcc,
                                std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>>& ct_tokens,
                                PtWeights_GPU& weights_layer0,
                                PtWeights_GPU& weights_layer1,
                                PtMasks_GPU& masks,
                                MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
                                TransposePrecomputations_GPU& Tprecomp_gpu,
                                lbcrypto::CryptoContext<lbcrypto::DCRTPoly>& cc,
                                lbcrypto::PrivateKey<lbcrypto::DCRTPoly>& sk,
                                int test_case)
{
    std::ifstream fin(file_path);
    if (!fin.is_open()){
        std::cerr << "ERROR: Could not open file: " << file_path << "\n";
        return;
    }

    size_t total_ok = 0, total_all = 0;
    std::string line;
    std::vector<std::string> cols;

    bool header_skipped = false;

    while (std::getline(fin, line)){
        rtrim_rte(line);
        if (line.empty()) continue;

        // one-time header skip
        if (!header_skipped){
            if (csv_parse_row_rte(line, cols) && cols.size() >= 4){
                std::string h0 = cols[0]; trim_rte(h0);
                if (h0 == "sentence1" || h0 == "Sentence1"){ header_skipped = true; continue; }
            }
            // if not a header, fall through and parse below again
        }

        if (!csv_parse_row_rte(line, cols)) continue;
        if (cols.size() < 4) continue;

        std::string s1 = cols[0];
        std::string s2 = cols[1];
        std::string lab = cols[2];
        std::string idx = cols[3];

        trim_rte(s1); trim_rte(s2); trim_rte(lab); trim_rte(idx);
        normalize_punct_rte(s1);
        normalize_punct_rte(s2);

        int label = -1;
        try { label = std::stoi(lab); } catch (...) { continue; }

        EncoderConfiguration conf = base_conf;
        // Pair tokenization (reuse your existing pair tokenizer)
        conf.token_length = tokenizer_pair(s1, s2, model_name, model_path, output_file);
        if (conf.token_length <= 0 || conf.token_length > conf.blockSize) continue;

        // Optional deep-clone original CPU ciphertexts (rename to avoid MRPC names)
        std::vector<std::vector<lbcrypto::Ciphertext<lbcrypto::DCRTPoly>>> ct_tokens_copy;
        ct_tokens_copy.resize(ct_tokens.size());
        for (size_t i = 0; i < ct_tokens.size(); ++i){
            ct_tokens_copy[i].resize(ct_tokens[i].size());
            for (size_t j = 0; j < ct_tokens[i].size(); ++j){
                ct_tokens_copy[i][j] = ct_tokens[i][j]->Clone();
            }
        }

        // Prepare GPU tokens (power-of-two buffer)
        int tokens_pow2 = 1;
        if (conf.token_length > 1){
            tokens_pow2 = 1 << static_cast<int>(std::ceil(std::log2(conf.token_length)));
        }

        std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> tokens_dev;
        encryptMatrixtoGPU(model_path + "/" + output_file, tokens_dev, pk, GPUcc,
                           conf.numSlots, conf.blockSize, tokens_pow2, conf.cols, conf.level_matmul);

        // Logging to file
        {
            std::ofstream out(output_path, std::ios::app);
            if (out){
                out << "\n///////////////////////////////////////\n";
                out << "Sentence1: \"" << s1 << "\"\n";
                out << "Sentence2: \"" << s2 << "\"\n";
                out << "Label: " << label << "\n";
                out << "Length: " << conf.token_length << "\n";
            } else {
                std::cerr << "[ERROR] Could not open file for writing: " << output_path << "\n";
            }
        }

        // Terminal echo
        std::cout << "\n///////////////////////////////////////\n";
        std::cout << "s1 = \"" << s1 << "\"\n";
        std::cout << "s2 = \"" << s2 << "\"\n";
        std::cout << "Label: " << label << "\n";
        std::cout << "Length: " << conf.token_length << "\n";


        hipDeviceSynchronize();
        auto t0 = std::chrono::high_resolution_clock::now();

        tokens_dev = encoder_helmet(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_dev, masks, conf, 0, test_case);
        tokens_dev = encoder_helmet(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_dev, masks, conf, 1, test_case);

        uint32_t pred =
            classifier(cc, tokens_dev, sk, ct_tokens_copy, precomp_gpu, weights_layer1,
                       masks, conf.numSlots, conf.blockSize, conf.token_length,
                       true, output_path);

        hipDeviceSynchronize();
        auto t1 = std::chrono::high_resolution_clock::now();

        tokens_dev.clear(); // release GPU-side matrix

        ++total_all;
        if (pred == static_cast<uint32_t>(label)) ++total_ok;

        {
            std::ofstream out(output_path, std::ios::app);
            if (out){
                out << "took: "
                    << std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count()
                    << " ms.\n";
                out << "Accuracy: " << total_ok << "/" << total_all << "\n";
            }
        }
        std::cout << "Accuracy: " << total_ok << "/" << total_all << "\n";

        header_skipped = true; // after first processed record
    }
}




// //// CARLOS ENCODER

// std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> encoder_helmet(
//     PtWeights_GPU& weights_layer, MatrixMatrixProductPrecomputations_GPU& precomp_gpu,
//     TransposePrecomputations_GPU& Tprecomp_gpu, std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>& tokens,
//     PtMasks_GPU& masks, EncoderConfiguration& conf, int layerNo, int test_case) {

//     constexpr bool PRINT = false;

//     Context& cc = tokens[0][0].cc;
//     std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> factor;

//     std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> K, Q, V, O, U, D, Sm_V;
//     std::vector<std::vector<std::vector<FIDESlib::CKKS::Ciphertext>>> Sm_V_, QKT_heads;

//     dropMatrixLevel(tokens, conf.level_matmul);

//     if constexpr (PRINT)
//         printMatrix(decryptGPUMatrix(tokens, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2, "tokens",
//                     false);
//     if constexpr (PRINT)
//         std::cout << "# limbs, tokens: " << tokens[0][0].getLevel() << " " << tokens[0][0].NoiseLevel << std::endl;

//     PCMM_2(tokens, weights_layer.Wk, conf.blockSize, K, precomp_gpu, weights_layer.bk,
//            masks.row_masks[conf.token_length]);

//     PCMM_2(tokens, weights_layer.Wq, conf.blockSize, Q, precomp_gpu, weights_layer.bq,
//            masks.row_masks[conf.token_length]);

//     PCMM_2(tokens, weights_layer.Wv, conf.blockSize, V, precomp_gpu, weights_layer.bv,
//            masks.row_masks[conf.token_length]);

//     if constexpr (PRINT)
//         std::cout << "# limbs, Q: " << Q[0][0].getLevel() << " " << Q[0][0].NoiseLevel << std::endl;

//     ////////////////////////////// Multi Head Attention /////////////////////////////////
//     auto K_T = MatrixTranspose_GPU(std::move(K), conf.blockSize, Tprecomp_gpu);

//     CCMM_GPU_double_mask(Q, K_T, conf.blockSize, QKT_heads, precomp_gpu, masks, 0, conf.token_length, true);
//     CCMM_GPU_double_mask(Q, K_T, conf.blockSize, QKT_heads, precomp_gpu, masks, 1, conf.token_length, true);

//     if (conf.token_length < 64) {

//         MatrixRotate(QKT_heads[1], -conf.blockSize / 2);

//         auto QKT = MatrixAdd(QKT_heads[0], QKT_heads[1]);

//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "QKT: ", false);

//         if (test_case == 0) {
//             if (layerNo == 0) {
//                 MatrixAddScalar(QKT, -0.2);
//             }
//             if (layerNo == 1) {
//                 MatrixAddScalar(QKT, -0.3);
//             }
//         }

//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "QKT: ", false);
//         // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), conf.token_length, conf.token_length, "QKT: ", false, 3, true);

//         MatrixBootstrap(QKT, conf.numSlots, conf.prescale);
//         EvalSoftmax_Matrix(QKT, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_tokens[conf.token_length],
//                            masks.mask_broadcast, masks.mask_layernorm[0], masks.mask_max, conf.numSlots, conf.blockSize,
//                            conf.bStepAcc, conf.token_length, true, test_case, layerNo);
//         MatrixBootstrap(QKT, conf.numSlots, conf.prescale);

//         // if constexpr (PRINT) printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), conf.token_length, conf.token_length, "QKT: ", false, 3, true);
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(QKT, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "QKT: ", false);

//         if constexpr (PRINT)
//             std::cout << "# limbs: " << QKT[0][0].getLevel() << " " << QKT[0][0].NoiseLevel << std::endl;

//         CCMM_GPU_double_mask(QKT, V, conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);
//         MatrixRotate(V, conf.blockSize / 2);
//         MatrixRotate(QKT, conf.blockSize / 2);
//         CCMM_GPU_double_mask(QKT, V, conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);

//         if constexpr (PRINT)
//             std::cout << "# limbs: " << Sm_V_[0][0][0].getLevel() << " " << Sm_V_[0][0][0].NoiseLevel << std::endl;
//         MatrixRotate(Sm_V_[1], -conf.blockSize / conf.num_heads);

//         Sm_V = MatrixAdd(Sm_V_[0], Sm_V_[1]);

//     } else {

//         for (int j = 0; j < conf.num_heads; j++) {
//             if (test_case == 0) {
//                 if (layerNo == 0) {
//                     MatrixAddScalar(QKT_heads[j], -0.2);
//                 }
//                 if (layerNo == 1) {
//                     MatrixAddScalar(QKT_heads[j], -0.3);
//                 }
//             }

//             if constexpr (PRINT)
//                 std::cout << "SM iteration: " << j << std::endl;

//             if constexpr (PRINT)
//                 printMatrix(decryptGPUMatrix(QKT_heads[j], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize),
//                             2, 2, "QKT_heads[j]: ", false);

//             MatrixBootstrap(QKT_heads[j], conf.numSlots, conf.prescale);
//             EvalSoftmax_Matrix(QKT_heads[j], ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(),
//                                masks.mask_tokens[conf.token_length], masks.mask_broadcast2, masks.mask_layernorm[0],
//                                masks.mask_max, conf.numSlots, conf.blockSize, conf.bStepAcc, conf.token_length, true,
//                                test_case, layerNo);
//             MatrixBootstrap(QKT_heads[j], conf.numSlots, conf.prescale);

//             if constexpr (PRINT)
//                 printMatrix(decryptGPUMatrix(QKT_heads[j], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize),
//                             2, 2, "QKT_heads[j]: ", false);

//             if (j == 1) {
//                 MatrixRotate(V, conf.blockSize / 2);
//                 // MatrixRotate(QKT_heads[j], conf.blockSize / 2);
//             }
//             // CCMM_GPU_double_mask(QKT_heads[j], V, conf.blockSize, Sm_V_, precomp_gpu, masks, 0, conf.token_length, false);
//             CCMM_GPU_masked(QKT_heads[j], V, conf.blockSize, Sm_V_, precomp_gpu, masks.head_masks[0]);

//             if constexpr (PRINT)
//                 printMatrix(decryptGPUMatrix(Sm_V_[j], keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                             "Sm_V[j]: ", false);
//         }

//         MatrixRotate(Sm_V_[1], -conf.blockSize / conf.num_heads);
//         Sm_V = MatrixAdd(Sm_V_[0], Sm_V_[1]);
//     }

//     //////////////////////////////////////////////////////////////////////////////////////
//     Sm_V = MatrixMask(Sm_V, masks.row_masks[conf.token_length]);
//     if constexpr (PRINT)
//         std::cout << "# limbs SmV: " << Sm_V[0][0].getLevel() << " " << Sm_V[0][0].NoiseLevel << std::endl;
//     if constexpr (PRINT)
//         printMatrix(decryptGPUMatrix(Sm_V, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                     "SmV: ", false);

//     MatrixBootstrap(Sm_V, conf.numSlots);

//     // Output CCMM
//     dropMatrixLevel(Sm_V, conf.level_matmul);
//     PCMM_2(Sm_V, weights_layer.Wo, conf.blockSize, O, precomp_gpu, weights_layer.bo,
//            masks.row_masks[conf.token_length]);
//     // PCMM_GPU(Sm_V, weights_layer.Wo, conf.blockSize, O, precomp_gpu, weights_layer.bo, masks.row_masks[conf.token_length]);

//     if constexpr (PRINT)
//         std::cout << "# limbs O: " << O[0][0].getLevel() << " " << O[0][0].NoiseLevel << std::endl;
//     if constexpr (PRINT)
//         printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                     "Result_output: ", false);

//     // Layer Norm
//     O = MatrixAdd(O, tokens);

//     if constexpr (PRINT)
//         std::cout << "# ------- bts ------- " << std::endl;
//     MatrixBootstrap(O, conf.numSlots);

//     if constexpr (PRINT)
//         std::cout << "# limbs LNin: " << O[0][0].getLevel() << " " << O[0][0].NoiseLevel << std::endl;
//     if constexpr (PRINT)
//         printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                     "LNin: ", false);
//     tokens.clear();



//     if (test_case == 2) {
//         EvalLayerNorm_Matrix_DelayedInv(O, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm,
//                                         masks.row_masks[conf.token_length], weights_layer.Wln1, weights_layer.bln1,
//                                         conf.numSlots, conf.blockSize, conf.bStepAcc, true, factor);
//         if constexpr (PRINT)
//             std::cout << "# limbs LN: " << O[0][0].getLevel() << " " << O[0][0].NoiseLevel << std::endl;

//         if constexpr (PRINT)
//             std::cout << "# ------- bts ------- " << std::endl;
//         MatrixBootstrap(O, conf.numSlots);
//         if constexpr (PRINT)
//             std::cout << "# limbs LN: " << O[0][0].getLevel() << " " << O[0][0].NoiseLevel << std::endl;

//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "LNout: ", false);
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(factor, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "factor: ", false);

//         // Up PCMM
//         PCMM_GPU_delayedInv(O, weights_layer.Wu, conf.blockSize, U, precomp_gpu, weights_layer.bu,
//                             masks.row_masks[conf.token_length], factor);

//         if constexpr (PRINT)
//             std::cout << "# size U: " << U.size() << std::endl;
//         if constexpr (PRINT)
//             std::cout << "# limbs U: " << U[0][0].getLevel() << " " << U[0][0].NoiseLevel << std::endl;
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "Up: ", false);

//         // ReLU
//         EvalGelu_Matrix(U, cc.GetEvalKey(), conf.numSlots);
//         if constexpr (PRINT)
//             std::cout << "# limbs r: " << U[0][0].getLevel() << " " << U[0][0].NoiseLevel << std::endl;
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "GELU: ", false);

//         // Down PCMM
//         PCMM_GPU_delayedInv(U, weights_layer.Wd, conf.blockSize, D, precomp_gpu, weights_layer.bd,
//                             masks.row_masks[conf.token_length], factor);
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "Down: ", false);
//         if constexpr (PRINT)
//             std::cout << "# limbs D: " << D[0][0].getLevel() << " " << D[0][0].NoiseLevel << std::endl;
//     } else {
//         EvalLayerNorm_Matrix(O, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm,
//                              masks.row_masks[conf.token_length], weights_layer.Wln1, weights_layer.bln1, conf.numSlots,
//                              conf.blockSize, conf.bStepAcc, true);
//         if constexpr (PRINT)
//             std::cout << "# limbs LN: " << O[0][0].getLevel() << " " << O[0][0].NoiseLevel << std::endl;

//         if constexpr (PRINT)
//             std::cout << "# ------- bts ------- " << std::endl;
//         MatrixBootstrap(O, conf.numSlots);
//         if constexpr (PRINT)
//             std::cout << "# limbs LN: " << O[0][0].getLevel() << " " << O[0][0].NoiseLevel << std::endl;

//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(O, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "LNout: ", false);

//         // Up PCMM
//         PCMM_2(O, weights_layer.Wu, conf.blockSize, U, precomp_gpu, weights_layer.bu,
//                masks.row_masks[conf.token_length]);
//         // PCMM_GPU(O, weights_layer.Wu, conf.blockSize, U, precomp_gpu, weights_layer.bu, masks.row_masks[conf.token_length]);

//         if constexpr (PRINT)
//             std::cout << "# limbs U: " << U[0][0].getLevel() << " " << U[0][0].NoiseLevel << std::endl;
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "Up: ", false);

//         // ReLU
//         EvalGelu_Matrix(U, cc.GetEvalKey(), conf.numSlots);
//         if constexpr (PRINT)
//             std::cout << "# limbs r: " << U[0][0].getLevel() << " " << U[0][0].NoiseLevel << std::endl;
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(U, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "GELU: ", false);

//         // Down PCMM
//         dropMatrixLevel(U, conf.level_matmul - 4);
//         PCMM_2(U, weights_layer.Wd, conf.blockSize, D, precomp_gpu, weights_layer.bd,
//                masks.row_masks[conf.token_length]);
//         // PCMM_GPU(U, weights_layer.Wd, conf.blockSize, D, precomp_gpu, weights_layer.bd, masks.row_masks[conf.token_length]);
//         if constexpr (PRINT)
//             printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                         "Down: ", false);
//         if constexpr (PRINT)
//             std::cout << "# limbs D: " << D[0][0].getLevel() << " " << D[0][0].NoiseLevel << std::endl;
//     }

//     // Layer Norm 2
//     D = MatrixAdd(D, O);
//     MatrixBootstrap(D, conf.numSlots);

//     if constexpr (PRINT)
//         printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                     "LNin: ", false);

//     EvalLayerNorm_Matrix(D, ct_tokens[0][0], keys_.secretKey, cc.GetEvalKey(), masks.mask_layernorm,
//                          masks.row_masks[conf.token_length], weights_layer.Wln2, weights_layer.bln2, conf.numSlots,
//                          conf.blockSize, conf.bStepAcc, true);
//     if constexpr (PRINT)
//         std::cout << "# limbs LN: " << D[0][0].getLevel() << " " << D[0][0].NoiseLevel << std::endl;
//     if constexpr (PRINT)
//         std::cout << "# ------- bts ------- " << std::endl;
//     MatrixBootstrap(D, conf.numSlots);
//     if constexpr (PRINT)
//         printMatrix(decryptGPUMatrix(D, keys_.secretKey, ct_tokens, conf.numSlots, conf.blockSize), 2, 2,
//                     "LNout: ", false);
//     if constexpr (PRINT)
//         std::cout << "# limbs LN: " << D[0][0].getLevel() << " " << D[0][0].NoiseLevel << std::endl;

//     hipDeviceSynchronize();

//     return D;
// }
}
