#include "hip/hip_runtime.h"
//
// Created by carlosad on 7/05/25.
//
#include <source_location>
#include "CKKS/Ciphertext.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/LinearTransform.cuh"
#include "CKKS/Plaintext.cuh"
#include "CudaUtils.cuh"

void FIDESlib::CKKS::LinearTransform(Ciphertext& ctxt, int rowSize, int bStep, std::vector<Plaintext*> pts, int stride,
                                     int offset) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts.size() >= rowSize);
    for (auto i : pts) {
        assert(i != nullptr);
    }
    Context& cc = ctxt.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt.NoiseLevel == 2)
        ctxt.rescale();

    std::vector<Ciphertext> fastRotation;

    for (int i = 0; i < bStep; ++i)
        fastRotation.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 0; i < bStep; ++i) {
        fastRotationPtr.push_back(&fastRotation[i]);
        keys.push_back(i == 0 ? nullptr : &cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    bool ext = true;
    if (bStep == 1)
        ext = false;
    for (auto& i : pts) {
        if (!i->c0.isModUp()) {
            ext = false;
        }
    }

    ctxt.rotate_hoisted(keys, indexes, fastRotationPtr, ext);

    Ciphertext inner(cc);


    for (uint32_t j = gStep - 1; j < gStep; --j) {

        // if (j == gStep - 1) {
        //     std::cout << "PT ct: " << fastRotation[0].getLevel() << ", " << fastRotation[0].NoiseLevel << std::endl;
        //     std::cout << "PT pt: " << (*pts[1]).c0.getLevel() << ", " << (*pts[1]).NoiseLevel << std::endl;
        // }
        int n = 1;
        //inner.multPt(ctxt, A[bStep * j], false);
        for (uint32_t i = 1; i < bStep; i++) {
            if (bStep * j + i < rowSize) {
                n++;
                //inner.addMultPt(fastRotation[i - 1], A[bStep * j + i], false);
            }
        }

        if (fastRotation[0].getLevel() > inner.getLevel()) {
            inner.c0.grow(fastRotation[0].getLevel(), true);
            inner.c1.grow(fastRotation[0].getLevel(), true);
        } else {
            inner.dropToLevel(fastRotation[0].getLevel());
        }

        inner.dotProductPt(fastRotation.data(), (const Plaintext**)pts.data() + j * bStep, n, ext);

        if (ext)
            inner.modDown(false);

        if (j > 0) {
            if (j == gStep - 1) {
                ctxt.copy(inner);
            } else {
                ctxt.add(inner);
            }
            ctxt.rotate((int)stride * bStep, cc.GetRotationKey((int)stride * bStep));
        } else {
            if (gStep == 1) {
                ctxt.copy(inner);
            } else {
                ctxt.add(inner);
            }
        }
    }

    if (offset != 0) {
        ctxt.rotate(offset, cc.GetRotationKey(offset));
    }
}

// Masked CCMM
void FIDESlib::CKKS::LinearTransform(Ciphertext& ctxt, int rowSize, int bStep, std::vector<Plaintext*> pts, int stride,
                                     int offset, Plaintext& mask_pt) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts.size() >= rowSize);
    for (auto i : pts) {
        assert(i != nullptr);
    }
    Context& cc = ctxt.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt.NoiseLevel == 2)
        ctxt.rescale();

    std::vector<Ciphertext> fastRotation;

    for (int i = 0; i < bStep; ++i) {
        fastRotation.emplace_back(cc);
    }

    std::vector<Ciphertext*> fastRotationPtr;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 0; i < bStep; ++i) {
        fastRotationPtr.push_back(&fastRotation[i]);

        keys.push_back(i == 0 ? nullptr : &cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    bool ext = true;
    if (bStep == 1)
        ext = false;


    // int i = 0;
    for (const Plaintext* p : pts) {
        ext &= p->c0.isModUp();
    }

    ctxt.rotate_hoisted(keys, indexes, fastRotationPtr, ext);
    Ciphertext inner(cc);

    Plaintext mask_pt_rot(cc);
    mask_pt_rot.copy(mask_pt);

    for (uint32_t j = gStep - 1; j < gStep; --j) {
        // std::cout << "# j: " << j << std::endl;


        int n = 1;
        //inner.multPt(ctxt, A[bStep * j], false);
        for (uint32_t i = 1; i < bStep; i++) {
            if (bStep * j + i < rowSize) {
                n++;
                //inner.addMultPt(fastRotation[i - 1], A[bStep * j + i], false);
            }
        }

        // masked pts !
        std::vector<Plaintext> tmp_storage;
        tmp_storage.reserve(n);
        for (int i = 0; i < n; i++) {
            Plaintext tmp(cc);
            tmp.copy(mask_pt_rot);
            tmp.automorph(indexes[i]);
            tmp.multPt(*pts[j * bStep + i], true);

            // tmp.copy(*pts[j * bStep + i]);
            tmp_storage.emplace_back(std::move(tmp));
        }
        std::vector<Plaintext*> tmp_pts;
        tmp_pts.reserve(tmp_storage.size());
        for (auto& x : tmp_storage) tmp_pts.push_back(&x);

        if (fastRotation[0].getLevel() > inner.getLevel()) {
            inner.c0.grow(fastRotation[0].getLevel(), true);
            inner.c1.grow(fastRotation[0].getLevel(), true);
        } else {
            inner.dropToLevel(fastRotation[0].getLevel());
        }

        // inner.dotProductPt(fastRotation.data(), (const Plaintext**)pts.data() + j * bStep, n, ext);
        inner.dotProductPt(fastRotation.data(), (const Plaintext**)tmp_pts.data(), n, ext);

        // std::cout << "inner: " << inner.getLevel() << ", " << inner.NoiseLevel << std::endl;

        if (ext)
            inner.modDown(false);

        if (j > 0) {
            if (j == gStep - 1) {
                ctxt.copy(inner);
            } else {
                ctxt.add(inner);
            }
            ctxt.rotate((int)stride * bStep, cc.GetRotationKey((int)stride * bStep));
        } else {
            if (gStep == 1) {
                ctxt.copy(inner);
            } else {
                ctxt.add(inner);
            }
        }
    }

    if (offset != 0) {
        ctxt.rotate(offset, cc.GetRotationKey(offset));
    }
}

void FIDESlib::CKKS::LinearTransformSpecial(FIDESlib::CKKS::Ciphertext& ctxt1, FIDESlib::CKKS::Ciphertext& ctxt2,
                                            FIDESlib::CKKS::Ciphertext& ctxt3, int rowSize, int bStep,
                                            std::vector<Plaintext*> pts1, std::vector<Plaintext*> pts2, int stride,
                                            int stride3) {
    constexpr bool PRINT = false;
    if constexpr (PRINT)
        std::cout << std::endl << "LinearTransformSpecial " << std::endl;

    if constexpr (PRINT) {
        std::cout << "ctxt1 ";
        for (auto& j : ctxt1.c0.GPU)
            for (auto& k : j.limb) {
                SWITCH(k, printThisLimb(1));
            }
        std::cout << std::endl;
        for (auto& j : ctxt1.c0.GPU)
            for (auto& k : j.SPECIALlimb) {
                SWITCH(k, printThisLimb(1));
            }
        std::cout << std::endl;
    }

    if constexpr (PRINT) {
        std::cout << "ctxt2 ";
        for (auto& j : ctxt2.c0.GPU)
            for (auto& k : j.limb) {
                SWITCH(k, printThisLimb(1));
            }
        std::cout << std::endl;
        for (auto& j : ctxt2.c0.GPU)
            for (auto& k : j.SPECIALlimb) {
                SWITCH(k, printThisLimb(1));
            }
        std::cout << std::endl;
    }

    if constexpr (PRINT) {
        std::cout << "ctxt3 ";
        for (auto& j : ctxt3.c0.GPU)
            for (auto& k : j.limb) {
                SWITCH(k, printThisLimb(1));
            }
        std::cout << std::endl;
        for (auto& j : ctxt3.c0.GPU)
            for (auto& k : j.SPECIALlimb) {
                SWITCH(k, printThisLimb(1));
            }
        std::cout << std::endl;
    }

    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts1.size() >= rowSize);
    for (auto i : pts1) {
        assert(i != nullptr);
    }
    assert(pts2.size() >= rowSize);

    Context& cc = ctxt1.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt1.NoiseLevel == 2)
        ctxt1.rescale();
    if (ctxt2.NoiseLevel == 2)
        ctxt2.rescale();
    if (ctxt3.NoiseLevel == 2)
        ctxt3.rescale();

    std::vector<Ciphertext> fastRotation2;
    std::vector<Ciphertext> fastRotation1;
    for (int i = 0; i < bStep - 1; ++i) {
        fastRotation2.emplace_back(cc);
        fastRotation1.emplace_back(cc);
    }

    std::vector<Ciphertext*> fastRotationPtr2;
    std::vector<Ciphertext*> fastRotationPtr1;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr1.push_back(&fastRotation1[i - 1]);
        fastRotationPtr2.push_back(&fastRotation2[i - 1]);
        keys.push_back(&cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    ctxt1.rotate_hoisted(keys, indexes, fastRotationPtr1, false);
    ctxt2.rotate_hoisted(keys, indexes, fastRotationPtr2, false);

    std::vector<Ciphertext> fastRotation3;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation3.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr3;
    std::vector<int> indexes3;
    std::vector<KeySwitchingKey*> keys3;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr3.push_back(&fastRotation3[i - 1]);
        keys3.push_back(&cc.GetRotationKey(i * stride3));
        indexes3.push_back(i * stride3);
    }

    Ciphertext result(cc);
    Ciphertext inner(cc);
    Ciphertext aux(cc);

    if ((gStep - 1) * bStep * (rowSize - 1) != 0)
        ctxt3.rotate((gStep - 1) * bStep * (rowSize - 1), cc.GetRotationKey((gStep - 1) * bStep * (rowSize - 1)));

    for (uint32_t j = gStep - 1; j < gStep; --j) {

        inner.multPt(ctxt1, *pts1[bStep * j], false);
        if (bStep * j != 0)
            inner.addMultPt(ctxt2, *pts2[bStep * j], true);
        inner.mult(ctxt3, cc.GetEvalKey(), false);

        if constexpr (PRINT) {
            std::cout << "inner " << bStep * j << " ";
            for (auto& j : inner.c0.GPU)
                for (auto& k : j.limb) {
                    SWITCH(k, printThisLimb(1));
                }
            std::cout << std::endl;
            for (auto& j : inner.c0.GPU)
                for (auto& k : j.SPECIALlimb) {
                    SWITCH(k, printThisLimb(1));
                }
            std::cout << std::endl;
        }

        for (uint32_t i = 1; i < bStep; i++) {
            if (bStep * j + i < rowSize) {
                if (i == 1) {
                    int size = std::min((int)bStep - 1, (int)(rowSize - (bStep * j + i)));
                    if (size < bStep - 1) {
                        auto keys3_ = keys3;
                        auto indexes3_ = indexes3;
                        auto fastRotationPtr3_ = fastRotationPtr3;

                        keys3_.resize(size);
                        indexes3_.resize(size);
                        fastRotationPtr3_.resize(size);

                        ctxt3.rotate_hoisted(keys3_, indexes3_, fastRotationPtr3_, false);
                    } else {
                        ctxt3.rotate_hoisted(keys3, indexes3, fastRotationPtr3, false);
                    }
                }
                aux.multPt(fastRotation1[i - 1], *pts1[bStep * j + i], false);
                aux.addMultPt(fastRotation2[i - 1], *pts2[bStep * j + i], true);
                aux.mult(fastRotation3[i - 1], cc.GetEvalKey(), false);
                if constexpr (PRINT) {
                    std::cout << "inner " << bStep * j + i << " ";
                    for (auto& j : aux.c0.GPU)
                        for (auto& k : j.limb) {
                            SWITCH(k, printThisLimb(1));
                        }
                    std::cout << std::endl;
                    for (auto& j : aux.c0.GPU)
                        for (auto& k : j.SPECIALlimb) {
                            SWITCH(k, printThisLimb(1));
                        }
                    std::cout << std::endl;
                }
                inner.add(aux);
            }
        }

        if (j > 0) {
            if (j == gStep - 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
            result.rotate(stride * bStep, cc.GetRotationKey(stride * bStep));
            ctxt3.rotate(-bStep * (rowSize - 1), cc.GetRotationKey(-bStep * (rowSize - 1)));
        } else {
            if (gStep == 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
        }
    }

    ctxt1.copy(result);
    CudaCheckErrorModNoSync;
}

// Masked CCMM
void FIDESlib::CKKS::LinearTransformSpecial(FIDESlib::CKKS::Ciphertext& ctxt1, FIDESlib::CKKS::Ciphertext& ctxt2,
                                            FIDESlib::CKKS::Ciphertext& ctxt3, int rowSize, int bStep,
                                            std::vector<Plaintext*> pts1, std::vector<Plaintext*> pts2, int stride,
                                            int stride3, Plaintext& mask_pt) {
    constexpr bool PRINT = false;
    
    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts1.size() >= rowSize);
    for (auto i : pts1) {
        assert(i != nullptr);
    }
    assert(pts2.size() >= rowSize);

    Context& cc = ctxt1.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt1.NoiseLevel == 2)
        ctxt1.rescale();
    if (ctxt2.NoiseLevel == 2)
        ctxt2.rescale();
    if (ctxt3.NoiseLevel == 2)
        ctxt3.rescale();

    std::vector<Ciphertext> fastRotation2;
    std::vector<Ciphertext> fastRotation1;
    for (int i = 0; i < bStep - 1; ++i) {
        fastRotation2.emplace_back(cc);
        fastRotation1.emplace_back(cc);
    }

    std::vector<Ciphertext*> fastRotationPtr2;
    std::vector<Ciphertext*> fastRotationPtr1;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr1.push_back(&fastRotation1[i - 1]);
        fastRotationPtr2.push_back(&fastRotation2[i - 1]);
        keys.push_back(&cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    ctxt1.rotate_hoisted(keys, indexes, fastRotationPtr1, false);
    ctxt2.rotate_hoisted(keys, indexes, fastRotationPtr2, false);

    std::vector<Ciphertext> fastRotation3;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation3.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr3;
    std::vector<int> indexes3;
    std::vector<KeySwitchingKey*> keys3;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr3.push_back(&fastRotation3[i - 1]);
        keys3.push_back(&cc.GetRotationKey(i * stride3));
        indexes3.push_back(i * stride3);
    }

    Ciphertext result(cc);
    Ciphertext inner(cc);
    Ciphertext aux(cc);

    if ((gStep - 1) * bStep * (rowSize - 1) != 0)
        ctxt3.rotate((gStep - 1) * bStep * (rowSize - 1), cc.GetRotationKey((gStep - 1) * bStep * (rowSize - 1)));

    Plaintext mask_temp(cc);
    mask_temp.copy(mask_pt);

    for (uint32_t j = gStep - 1; j < gStep; --j) {
        // if (j != gStep - 1 && j > 0) {
        //     mask_temp.automorph(- stride * bStep);
        // }

        Plaintext mask_temp2(cc);
        mask_temp2.copy(mask_pt);
        mask_temp2.automorph(- j * stride * bStep);

        Plaintext tmp1(cc);
        tmp1.copy(*pts1[bStep * j]);
        tmp1.multPt(tmp1, mask_temp2, true);

        // std::cout << ctxt1.getLevel() << tmp1.c0.getLevel() << std::endl;

        inner.multPt(ctxt1, tmp1, false);

        if (bStep * j != 0){
            Plaintext tmp2(cc);
            tmp2.copy(*pts2[bStep * j]);
            tmp2.multPt(tmp2, mask_temp2, true);

            // std::cout << ctxt2.getLevel() << tmp2.c0.getLevel() << std::endl;

            inner.addMultPt(ctxt2, tmp2, false);
        }

        inner.mult(ctxt3, cc.GetEvalKey(), false);

        for (uint32_t i = 1; i < bStep; i++) {
            if (bStep * j + i < rowSize) {
                if (i == 1) {
                    int size = std::min((int)bStep - 1, (int)(rowSize - (bStep * j + i)));
                    if (size < bStep - 1) {
                        auto keys3_ = keys3;
                        auto indexes3_ = indexes3;
                        auto fastRotationPtr3_ = fastRotationPtr3;

                        keys3_.resize(size);
                        indexes3_.resize(size);
                        fastRotationPtr3_.resize(size);

                        ctxt3.rotate_hoisted(keys3_, indexes3_, fastRotationPtr3_, false);
                    } else {
                        ctxt3.rotate_hoisted(keys3, indexes3, fastRotationPtr3, false);
                    }
                }

                Plaintext temp1(cc);
                temp1.copy(*pts1[bStep * j + i]);
                temp1.multPt(temp1, mask_temp2, true);
                // std::cout << fastRotation1[i - 1].getLevel() << temp1.c0.getLevel() << std::endl;
                aux.multPt(fastRotation1[i - 1], temp1, false);

                Plaintext temp2(cc);
                temp2.copy(*pts2[bStep * j + i]);
                temp2.multPt(temp2, mask_temp2, true);
                // std::cout << fastRotation2[i - 1].getLevel() << temp2.c0.getLevel() << std::endl;
                aux.addMultPt(fastRotation2[i - 1], temp2);

                aux.mult(fastRotation3[i - 1], cc.GetEvalKey(), false);

                inner.add(aux);
            }
        }

        if (j > 0) {
            if (j == gStep - 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
            result.rotate(stride * bStep, cc.GetRotationKey(stride * bStep));
            ctxt3.rotate(-bStep * (rowSize - 1), cc.GetRotationKey(-bStep * (rowSize - 1)));
        } else {
            if (gStep == 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
        }
    }

    ctxt1.copy(result);
    CudaCheckErrorModNoSync;
}

std::vector<int> FIDESlib::CKKS::GetLinearTransformRotationIndices(int bStep, int stride, int offset) {
    std::vector<int> res(bStep + (offset != 0));
    for (int i = 1; i <= bStep; ++i)
        res[i - 1] = i * stride;
    if (offset != 0)
        res[bStep] = offset;
    return res;
}

std::vector<int> FIDESlib::CKKS::GetLinearTransformPlaintextRotationIndices(int rowSize, int bStep, int stride,
                                                                            int offset) {
    std::vector<int> res(rowSize);
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    for (int j = 0; j < gStep; j++) {
        for (int i = 0; i < bStep; ++i) {
            if (i + j * bStep < rowSize)
                res[i + j * bStep] = -bStep * j * stride - offset;
        }
    }
    return res;
}

void FIDESlib::CKKS::LinearTransformSpecialPt(FIDESlib::CKKS::Ciphertext& ctxt1, FIDESlib::CKKS::Ciphertext& ctxt2,
                                              FIDESlib::CKKS::Plaintext& ptxt, int rowSize, int bStep,
                                              std::vector<Plaintext*> pts1, std::vector<Plaintext*> pts2, int stride,
                                              int stride3) {
    constexpr bool PRINT = true;

    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts1.size() >= rowSize);
    for (auto i : pts1) {
        assert(i != nullptr);
    }
    assert(pts2.size() >= rowSize);

    Context& cc = ctxt1.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt1.NoiseLevel == 2)
        ctxt1.rescale();
    if (ctxt2.NoiseLevel == 2)
        ctxt2.rescale();

    std::vector<Ciphertext> fastRotation1;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation1.emplace_back(cc);
    std::vector<Ciphertext> fastRotation2;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation2.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr1;
    std::vector<Ciphertext*> fastRotationPtr2;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr1.push_back(&fastRotation1[i - 1]);
        fastRotationPtr2.push_back(&fastRotation2[i - 1]);
        keys.push_back(&cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    ctxt1.rotate_hoisted(keys, indexes, fastRotationPtr1, false);
    ctxt2.rotate_hoisted(keys, indexes, fastRotationPtr2, false);

    std::vector<Plaintext> fastRotation3;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation3.emplace_back(cc);

    std::vector<Plaintext*> fastRotationPtr3;
    std::vector<int> indexes3;
    //std::vector<KeySwitchingKey*> keys3;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr3.push_back(&fastRotation3[i - 1]);
        //keys3.push_back(&cc.GetRotationKey(i * stride3));
        indexes3.push_back(i * stride3);
    }

    Ciphertext result(cc);
    Ciphertext inner(cc);
    Ciphertext aux(cc);

    if ((gStep - 1) * bStep * (rowSize - 1) != 0)
        // ctxt2.rotate((gStep - 1) * bStep * (rowSize - 1), cc.GetRotationKey((gStep - 1) * bStep * (rowSize - 1)));
        ptxt.automorph((gStep - 1) * bStep * (rowSize - 1));

    for (uint32_t j = gStep - 1; j < gStep; --j) {

        Plaintext tmp1(cc);
        tmp1.copy(*pts1[bStep * j]);
        tmp1.multPt(tmp1, ptxt, true);
        inner.multPt(ctxt1, tmp1, false);

        if (bStep * j != 0){
            Plaintext tmp2(cc);
            tmp2.copy(*pts2[bStep * j]);
            tmp2.multPt(tmp2, ptxt, true);

            inner.addMultPt(ctxt2, tmp2, false);
        }
        for (uint32_t i = 1; i < bStep; i++) {

            if (bStep * j + i < rowSize) {

                if (i == 1) {
                    int size = std::min((int)bStep - 1, (int)(rowSize - (bStep * j + i)));
                    if (size < bStep - 1) {
                        //auto keys3_ = keys3;
                        auto indexes3_ = indexes3;

                        //keys3_.resize(size);
                        indexes3_.resize(size);
                        // ctxt2.rotate_hoisted(keys3_, indexes3_, fastRotationPtr3);
                        ptxt.rotate_hoisted(indexes3_, fastRotationPtr3);
                    } else {
                        // ctxt2.rotate_hoisted(keys3, indexes3, fastRotationPtr3);
                        ptxt.rotate_hoisted(indexes3, fastRotationPtr3);
                    }
                }
                // (*pts1[bStep * j + i]).multPt(*pts1[bStep * j + i], fastRotation3[i - 1], true);
                Plaintext temp1(cc);
                temp1.copy(*pts1[bStep * j + i]);
                temp1.multPt(temp1, fastRotation3[i - 1], true);
                aux.multPt(fastRotation1[i - 1], temp1, false);

                // (*pts2[bStep * j + i]).multPt(*pts2[bStep * j + i], fastRotation3[i - 1], true);
                Plaintext temp2(cc);
                temp2.copy(*pts2[bStep * j + i]);
                temp2.multPt(temp2, fastRotation3[i - 1], true);
                aux.addMultPt(fastRotation2[i - 1], temp2);

                inner.add(aux);
            }
        }

        if (j > 0) {

            if (j == gStep - 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
            result.rotate(stride * bStep, cc.GetRotationKey(stride * bStep));
            // ctxt2.rotate(-bStep * (rowSize - 1), cc.GetRotationKey(-bStep * (rowSize - 1)));
            ptxt.automorph(-bStep * (rowSize - 1));
        } else {
            if (gStep == 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
        }
    }

    ctxt1.copy(result);
    hipDeviceSynchronize();
}

// for merged PCMM
void FIDESlib::CKKS::LinearTransformSpecialPt(FIDESlib::CKKS::Ciphertext& ctxt1, FIDESlib::CKKS::Ciphertext& ctxt2,
                                              FIDESlib::CKKS::Plaintext& ptxt, int rowSize, int bStep,
                                              std::vector<Plaintext*> pts1, std::vector<Plaintext*> pts2, int stride,
                                              int stride3, FIDESlib::CKKS::Ciphertext& cProduct) {
    constexpr bool PRINT = true;

    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts1.size() >= rowSize);
    for (auto i : pts1) {
        assert(i != nullptr);
    }
    assert(pts2.size() >= rowSize);

    Context& cc = ctxt1.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt1.NoiseLevel == 2)
        ctxt1.rescale();
    if (ctxt2.NoiseLevel == 2)
        ctxt2.rescale();

    std::vector<Ciphertext> fastRotation1;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation1.emplace_back(cc);
    std::vector<Ciphertext> fastRotation2;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation2.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr1;
    std::vector<Ciphertext*> fastRotationPtr2;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr1.push_back(&fastRotation1[i - 1]);
        fastRotationPtr2.push_back(&fastRotation2[i - 1]);
        keys.push_back(&cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    ctxt1.rotate_hoisted(keys, indexes, fastRotationPtr1, false);
    ctxt2.rotate_hoisted(keys, indexes, fastRotationPtr2, false);

    std::vector<Plaintext> fastRotation3;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation3.emplace_back(cc);

    std::vector<Plaintext*> fastRotationPtr3;
    std::vector<int> indexes3;
    //std::vector<KeySwitchingKey*> keys3;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr3.push_back(&fastRotation3[i - 1]);
        //keys3.push_back(&cc.GetRotationKey(i * stride3));
        indexes3.push_back(i * stride3);
    }

    Ciphertext result(cc);
    Ciphertext inner(cc);
    Ciphertext aux(cc);

    if ((gStep - 1) * bStep * (rowSize - 1) != 0)
        // ctxt2.rotate((gStep - 1) * bStep * (rowSize - 1), cc.GetRotationKey((gStep - 1) * bStep * (rowSize - 1)));
        ptxt.automorph((gStep - 1) * bStep * (rowSize - 1));

    for (uint32_t j = gStep - 1; j < gStep; --j) {

        Plaintext tmp1(cc);
        tmp1.copy(*pts1[bStep * j]);
        tmp1.multPt(tmp1, ptxt, true);
        inner.multPt(ctxt1, tmp1, false);

        if (bStep * j != 0){
            Plaintext tmp2(cc);
            tmp2.copy(*pts2[bStep * j]);
            tmp2.multPt(tmp2, ptxt, true);

            inner.addMultPt(ctxt2, tmp2, false);
        }
        for (uint32_t i = 1; i < bStep; i++) {

            if (bStep * j + i < rowSize) {

                if (i == 1) {
                    int size = std::min((int)bStep - 1, (int)(rowSize - (bStep * j + i)));
                    if (size < bStep - 1) {
                        //auto keys3_ = keys3;
                        auto indexes3_ = indexes3;

                        //keys3_.resize(size);
                        indexes3_.resize(size);
                        // ctxt2.rotate_hoisted(keys3_, indexes3_, fastRotationPtr3);
                        ptxt.rotate_hoisted(indexes3_, fastRotationPtr3);
                    } else {
                        // ctxt2.rotate_hoisted(keys3, indexes3, fastRotationPtr3);
                        ptxt.rotate_hoisted(indexes3, fastRotationPtr3);
                    }
                }
                // (*pts1[bStep * j + i]).multPt(*pts1[bStep * j + i], fastRotation3[i - 1], true);
                Plaintext temp1(cc);
                temp1.copy(*pts1[bStep * j + i]);
                temp1.multPt(temp1, fastRotation3[i - 1], true);
                aux.multPt(fastRotation1[i - 1], temp1, false);

                // (*pts2[bStep * j + i]).multPt(*pts2[bStep * j + i], fastRotation3[i - 1], true);
                Plaintext temp2(cc);
                temp2.copy(*pts2[bStep * j + i]);
                temp2.multPt(temp2, fastRotation3[i - 1], true);
                aux.addMultPt(fastRotation2[i - 1], temp2);

                inner.add(aux);
            }
        }

        if (j > 0) {

            if (j == gStep - 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
            result.rotate(stride * bStep, cc.GetRotationKey(stride * bStep));
            // ctxt2.rotate(-bStep * (rowSize - 1), cc.GetRotationKey(-bStep * (rowSize - 1)));
            ptxt.automorph(-bStep * (rowSize - 1));
        } else {
            if (gStep == 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
        }
    }

    cProduct.copy(result);
    hipDeviceSynchronize();
}

void FIDESlib::CKKS::LinearTransformSpecialPt_2(FIDESlib::CKKS::Ciphertext& ctxt1, FIDESlib::CKKS::Ciphertext& ctxt2,
                                              FIDESlib::CKKS::Plaintext& ptxt, int rowSize, int bStep,
                                              std::vector<Plaintext*> pts1, std::vector<Plaintext*> pts2, int stride,
                                              int stride3) {
    constexpr bool PRINT = true;

    CudaNvtxRange r(std::string{std::source_location::current().function_name()});
    assert(pts1.size() >= rowSize);
    for (auto i : pts1) {
        assert(i != nullptr);
    }
    assert(pts2.size() >= rowSize);

    Context& cc = ctxt1.cc;
    uint32_t gStep = ceil(static_cast<double>(rowSize) / bStep);

    if (ctxt1.NoiseLevel == 2)
        ctxt1.rescale();
    if (ctxt2.NoiseLevel == 2)
        ctxt2.rescale();

    std::vector<Ciphertext> fastRotation1;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation1.emplace_back(cc);
    std::vector<Ciphertext> fastRotation2;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation2.emplace_back(cc);

    std::vector<Ciphertext*> fastRotationPtr1;
    std::vector<Ciphertext*> fastRotationPtr2;
    std::vector<int> indexes;
    std::vector<KeySwitchingKey*> keys;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr1.push_back(&fastRotation1[i - 1]);
        fastRotationPtr2.push_back(&fastRotation2[i - 1]);
        keys.push_back(&cc.GetRotationKey(i * stride));
        indexes.push_back(i * stride);
    }

    ctxt1.rotate_hoisted(keys, indexes, fastRotationPtr1, false);
    ctxt2.rotate_hoisted(keys, indexes, fastRotationPtr2, false);

    std::vector<Plaintext> fastRotation3;
    for (int i = 0; i < bStep - 1; ++i)
        fastRotation3.emplace_back(cc);

    std::vector<Plaintext*> fastRotationPtr3;
    std::vector<int> indexes3;
    for (int i = 1; i < bStep; ++i) {
        fastRotationPtr3.push_back(&fastRotation3[i - 1]);
        indexes3.push_back(i * stride3);
    }

    Ciphertext result(cc);
    Ciphertext inner(cc);
    Ciphertext aux(cc);

    if ((gStep - 1) * bStep * (rowSize - 1) != 0)
        ptxt.automorph((gStep - 1) * bStep * (rowSize - 1));

    for (uint32_t j = gStep - 1; j < gStep; --j) {

        // mask 1
        Plaintext tmp2(cc);

        tmp2.copy(*pts1[bStep * j]);
        tmp2.multPt(ptxt, true);
        BroadcastPtMM(tmp2, cc, rowSize);    // broadcast!!!

        inner.multPt(ctxt1, tmp2, false);

        if (bStep * j != 0){
            Plaintext tmp(cc);
            tmp.copy(*pts2[bStep * j]);

            tmp.multPt(ptxt, true);
            BroadcastPtMM(tmp, cc, rowSize);    // broadcast!!!

            inner.addMultPt(ctxt2, tmp, false);
        }


        for (uint32_t i = 1; i < bStep; i++) {

            if (bStep * j + i < rowSize) {

                if (i == 1) {
                    int size = std::min((int)bStep - 1, (int)(rowSize - (bStep * j + i)));
                    if (size < bStep - 1) {
                        auto indexes3_ = indexes3;
                        indexes3_.resize(size);
                        ptxt.rotate_hoisted(indexes3_, fastRotationPtr3);
                    } else {
                        ptxt.rotate_hoisted(indexes3, fastRotationPtr3);
                    }
                }
                // mask 1
                Plaintext tmp(cc);

                tmp.copy(*pts1[bStep * j + i]);
                tmp.multPt(tmp, fastRotation3[i - 1], true);
                BroadcastPtMM(tmp, cc, rowSize);    // broadcast!!!

                aux.multPt(fastRotation1[i - 1], tmp, false);

                Plaintext tmp2(cc);

                tmp2.copy(*pts2[bStep * j + i]);
                tmp2.multPt(tmp2, fastRotation3[i - 1], true);
                BroadcastPtMM(tmp2, cc, rowSize);    // broadcast!!!    // Carlos suggestion:    maybe try reordering to reduce the broadcast call to 1   

                aux.addMultPt(fastRotation2[i - 1], tmp2);

                inner.add(aux);
            }
        }

        if (j > 0) {

            if (j == gStep - 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
            result.rotate(stride * bStep, cc.GetRotationKey(stride * bStep));
            ptxt.automorph(-bStep * (rowSize - 1));
        } else {
            if (gStep == 1) {
                result.copy(inner);
            } else {
                result.add(inner);
            }
        }
    }
    ctxt1.copy(result);

    hipDeviceSynchronize();
}
