#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <cstdlib>
#include <filesystem>
#include <string>

#include <CKKS/KeySwitchingKey.cuh>
#include "CKKS/Ciphertext.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/openfhe-interface/RawCiphertext.cuh"

#include "MatMul.cuh"
#include "PolyApprox.cuh"
#include "Transformer.cuh"
#include "Transpose.cuh"

#include "CKKS/ApproxModEval.cuh"
#include "CKKS/Bootstrap.cuh"
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/CoeffsToSlots.cuh"
#include "CKKS/AccumulateBroadcast.cuh"

#include "utils.cuh"

namespace fs = std::filesystem; 
using namespace FIDESlib::CKKS;
std::vector<int> devices{0};

int main(const int argc, char **argv) {

    // Context 
	create_cpu_context();
	lbcrypto::KeyPair<lbcrypto::DCRTPoly> keys = cc->KeyGen();
    EncoderConfiguration conf{.numSlots = cc->GetEncodingParams()->GetBatchSize(), .blockSize = int(sqrt(cc->GetEncodingParams()->GetBatchSize())), .token_length = 10};
	prepare_cpu_context(keys, conf.numSlots, conf.blockSize);
    keys_ = keys;
	auto raw_params = FIDESlib::CKKS::GetRawParams(cc);
	auto adapted_params = params.adaptTo(raw_params);
	FIDESlib::CKKS::Context GPUcc(adapted_params, std::vector<int>(devices.begin(), devices.end()));
	prepare_gpu_context_bert(GPUcc, keys, conf.numSlots, conf.blockSize);

    // Paths
    std::string model_path = std::string(root_dir) + "weights/weights-bert-tiny-sst2";
    std::string path = model_path + "/sst2_validation.csv";
    std::string output_path = "a.txt";


    // Loading weights and biases
    struct PtMasks_GPU masks = GetPtMasks_GPU(GPUcc, cc, conf.numSlots, conf.blockSize, conf.token_length, conf.level_matmul+1);

    struct PtWeights_GPU weights_layer0 = GetPtWeightsGPU(GPUcc, keys.publicKey, model_path, 0, conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul+1, conf.num_heads);
    struct PtWeights_GPU weights_layer1 = GetPtWeightsGPU(GPUcc, keys.publicKey, model_path, 1, conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul+1, conf.num_heads);

    struct MatrixMatrixProductPrecomputations_GPU precomp_gpu = getMatrixMatrixProductPrecomputations_GPU(
        GPUcc, cc, masks, conf.blockSize, conf.bStep, conf.level_matmul+1, conf.level_matmul+1, conf.prescale, conf.numSlots);

    TransposePrecomputations_GPU Tprecomp_gpu = getMatrixTransposePrecomputations_GPU(GPUcc, cc, conf.blockSize, conf.bStep, conf.level_matmul);

    // Loading tokens
    ct_tokens = encryptMatrixtoCPU(std::string(model_path + "/tokens.txt"), keys.publicKey, conf.numSlots, conf.blockSize, conf.rows, conf.cols);
    std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> tokens_gpu, tokens_gpu2;
    encryptMatrixtoGPU(std::string(model_path + "/tokens.txt"), tokens_gpu, keys.publicKey, GPUcc, conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul);
    encryptMatrixtoGPU(std::string(model_path + "/tokens.txt"), tokens_gpu2, keys.publicKey, GPUcc, conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul);
    

    // Baseline
    auto start_gpu = std::chrono::high_resolution_clock::now();
    tokens_gpu = encoder(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 0);
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();

    tokens_gpu = encoder(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 1);
    uint32_t class_pred = classifier(cc, tokens_gpu, keys.secretKey, ct_tokens, precomp_gpu, weights_layer1, masks, conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
    
    std::cout << "took: " << (std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count()) << " ms." << std::endl;


    // Proposed Method
    int test_case = 0;  // 0: Asymmetrical PCMM, 1: Statistical Softmax, 3: LN with delayed norm
    start_gpu = std::chrono::high_resolution_clock::now();
    tokens_gpu2 = encoder_helmet(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu2, masks, conf, 0, test_case);
    hipDeviceSynchronize();
    end_gpu = std::chrono::high_resolution_clock::now();
    
    tokens_gpu2 = encoder_helmet(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu2, masks, conf, 1, test_case);
    class_pred = classifier(cc, tokens_gpu2, keys.secretKey, ct_tokens, precomp_gpu, weights_layer1, masks, conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
 
    std::cout << "took: " << (std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count()) << " ms." << std::endl;


	return 0;
}

// #include <gtest/gtest.h>
// #include <cstdlib>
// #include <filesystem>
// #include <string>
// #include <CKKS/KeySwitchingKey.cuh>
// #include "CKKS/Ciphertext.cuh"
// #include "CKKS/Context.cuh"
// #include "CKKS/openfhe-interface/RawCiphertext.cuh"
// #include "MatMul.cuh"
// #include "PolyApprox.cuh"
// #include "Transformer.cuh"
// #include "Transpose.cuh"
// #include "CKKS/AccumulateBroadcast.cuh"
// #include "CKKS/ApproxModEval.cuh"
// #include "CKKS/Bootstrap.cuh"
// #include "CKKS/BootstrapPrecomputation.cuh"
// #include "CKKS/CoeffsToSlots.cuh"
// #include "utils.cuh"
// namespace fs = std::filesystem;
// using namespace FIDESlib::CKKS;
// std::vector<int> devices{0};
// int main(const int argc, char** argv) {
//     // Context
//     create_cpu_context();
//     lbcrypto::KeyPair<lbcrypto::DCRTPoly> keys = cc->KeyGen();
//     EncoderConfiguration conf{.numSlots = cc->GetEncodingParams()->GetBatchSize(),
//                               .blockSize = int(sqrt(cc->GetEncodingParams()->GetBatchSize())),
//                               .token_length = 10,
//                               .bStep = 16,
//                               .bStepBoot = 16,
//                               .bStepAcc = 4};
//     prepare_cpu_context(keys, conf.numSlots, conf.blockSize);
//     keys_ = keys;
//     auto raw_params = FIDESlib::CKKS::GetRawParams(cc);
//     auto adapted_params = params.adaptTo(raw_params);
//     FIDESlib::CKKS::Context GPUcc(adapted_params, std::vector<int>(devices.begin(), devices.end()));
//     prepare_gpu_context_bert(GPUcc, keys, conf);
//     GPUcc.batch = 100;
//     // Paths
//     std::string model_path = std::string(root_dir) + "weights/weights-bert-tiny-sst2";
//     std::string path = model_path + "/sst2_validation.csv";
//     std::string output_path = "a.txt";
//     // Loading weights and biases
//     struct PtMasks_GPU masks =
//         GetPtMasks_GPU(GPUcc, cc, conf.numSlots, conf.blockSize, conf.token_length, conf.level_matmul + 1);
//     struct PtWeights_GPU weights_layer0 =
//         GetPtWeightsGPU(GPUcc, keys.publicKey, model_path, 0, conf.numSlots, conf.blockSize, conf.rows, conf.cols,
//                         conf.level_matmul + 1, conf.num_heads);
//     struct PtWeights_GPU weights_layer1 =
//         GetPtWeightsGPU(GPUcc, keys.publicKey, model_path, 1, conf.numSlots, conf.blockSize, conf.rows, conf.cols,
//                         conf.level_matmul + 1, conf.num_heads);
//     struct MatrixMatrixProductPrecomputations_GPU precomp_gpu =
//         getMatrixMatrixProductPrecomputations_GPU(GPUcc, cc, masks, conf.blockSize, conf.bStep, conf.level_matmul + 1,
//                                                   conf.level_matmul + 1, conf.prescale, conf.numSlots);
//     TransposePrecomputations_GPU Tprecomp_gpu =
//         getMatrixTransposePrecomputations_GPU(GPUcc, cc, conf.blockSize, conf.bStep, conf.level_matmul);
//     // Loading tokens
//     ct_tokens = encryptMatrixtoCPU(std::string(model_path + "/tokens.txt"), keys.publicKey, conf.numSlots,
//                                    conf.blockSize, conf.rows, conf.cols);
//     std::vector<std::vector<FIDESlib::CKKS::Ciphertext>> tokens_gpu, tokens_gpu2;
//     encryptMatrixtoGPU(std::string(model_path + "/tokens.txt"), tokens_gpu, keys.publicKey, GPUcc, conf.numSlots,
//                        conf.blockSize, conf.rows, conf.cols, conf.level_matmul);
//     encryptMatrixtoGPU(std::string(model_path + "/tokens.txt"), tokens_gpu2, keys.publicKey, GPUcc, conf.numSlots,
//                        conf.blockSize, conf.rows, conf.cols, conf.level_matmul);
//     if (1) {
//         // Baseline
//         hipDeviceSynchronize();
//         auto start_gpu = std::chrono::high_resolution_clock::now();
//         tokens_gpu = encoder(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 0);
//         hipDeviceSynchronize();
//         auto end_gpu = std::chrono::high_resolution_clock::now();
//         tokens_gpu = encoder(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu, masks, conf, 1);
//         uint32_t class_pred = classifier(cc, tokens_gpu, keys.secretKey, ct_tokens, precomp_gpu, weights_layer1, masks,
//                                          conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
//         std::cout << "took: " << (std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count())
//                   << " ms."
//                      ""
//                   << std::endl;
//     }
//     // Proposed Method
//     for (int test_case : {0, 1, 2}) {
//         // 0: Asymmetrical PCMM, 1: Statistical Softmax, 2: LN with delayed norm
//         hipDeviceSynchronize();
//         auto start_gpu = std::chrono::high_resolution_clock::now();
//         tokens_gpu2 = encoder_helmet(weights_layer0, precomp_gpu, Tprecomp_gpu, tokens_gpu2, masks, conf, 0, test_case);
//         hipDeviceSynchronize();
//         auto end_gpu = std::chrono::high_resolution_clock::now();
//         tokens_gpu2 = encoder_helmet(weights_layer1, precomp_gpu, Tprecomp_gpu, tokens_gpu2, masks, conf, 1, test_case);
//         uint32_t class_pred = classifier(cc, tokens_gpu2, keys.secretKey, ct_tokens, precomp_gpu, weights_layer1, masks,
//                                          conf.numSlots, conf.blockSize, conf.token_length, true, output_path);
//         std::cout << "took: " << (std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu - start_gpu).count())
//                   << " ms." << std::endl;
//     }
//     return 0;
}