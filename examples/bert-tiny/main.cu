#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <cstdlib>
#include <filesystem>
#include <string>

#include <CKKS/KeySwitchingKey.cuh>
#include "CKKS/Ciphertext.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/openfhe-interface/RawCiphertext.cuh"

#include "MatMul.cuh"
#include "PolyApprox.cuh"
#include "Transformer.cuh"
#include "Transpose.cuh"

#include "CKKS/ApproxModEval.cuh"
#include "CKKS/Bootstrap.cuh"
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/CoeffsToSlots.cuh"
#include "CKKS/AccumulateBroadcast.cuh"

#include "utils.cuh"

namespace fs = std::filesystem; 
using namespace FIDESlib::CKKS;
std::vector<int> devices{0};

int main(const int argc, char **argv) {

    std::string model_name = "bert-tiny-mrpc";
    std::string model_path = std::string(root_dir) + "weights/weights-bert-tiny-mrpc";

    int test_case   = 1;
    std::string outputBase  = ".";   
    std::string outputPrefix = "out";

    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "-c" && i + 1 < argc) {
            test_case = std::stoi(argv[++i]);
        } else if (arg == "-w" && i + 1 < argc) {
            outputBase = argv[++i];
        }
    }

    fs::path outDir(outputBase);
    std::error_code ec;
    fs::create_directories(outDir, ec); // ok if already exists

    int task_id = test_case;
    if (const char* p = std::getenv("SGE_TASK_ID")) {
        try { task_id = std::stoi(p); } catch (...) { /* keep test_case */ }
    }

    std::string output_file = "tokens_mrpc" + std::to_string(test_case+1) + ".txt";

    std::string path = model_path + "/mrpc_validation.csv";

    if (outputBase == "mrpc12_seyda/") {
        path = model_path + "/mrpc_validation12.csv";
    }
    else {
        output_file = "tokens_mrpc" + std::to_string(test_case+4) + ".txt";
    }
    std::cout << "input file: " << path << std::endl;
    std::cout << "token file: " << output_file << std::endl;

    fs::path output_path = outDir / ("out_" + std::to_string(test_case) + ".txt");

    std::ofstream outFile(output_path); // creates the file if it doesn't exist
    if (!outFile.is_open()) {
        std::cerr << "Error: could not create file " << output_path << std::endl;
        return 1;
    }

    outFile << "Experiment: " << test_case << std::endl;
    outFile.close();
 
	create_cpu_context();
	lbcrypto::KeyPair<lbcrypto::DCRTPoly> keys = cc->KeyGen();
    EncoderConfiguration conf{.numSlots = cc->GetEncodingParams()->GetBatchSize(), .blockSize = int(sqrt(cc->GetEncodingParams()->GetBatchSize())), .token_length = 63};
	prepare_cpu_context(keys, conf.numSlots, conf.blockSize);
	auto raw_params = FIDESlib::CKKS::GetRawParams(cc);
	auto adapted_params = params.adaptTo(raw_params);
	FIDESlib::CKKS::Context GPUcc(adapted_params, std::vector<int>(devices.begin(), devices.end()));
	prepare_gpu_context_bert(GPUcc, keys, conf.numSlots, conf.blockSize);

    // Loading weights and biases

    struct PtMasks_GPU masks = GetPtMasks_GPU(GPUcc, cc, conf.numSlots, conf.blockSize, conf.token_length, conf.level_matmul+1);

    struct PtWeights_GPU weights_layer0 = GetPtWeightsGPU(GPUcc, keys.publicKey, model_path, 0, conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul+1, conf.num_heads);
    struct PtWeights_GPU weights_layer1 = GetPtWeightsGPU(GPUcc, keys.publicKey, model_path, 1, conf.numSlots, conf.blockSize, conf.rows, conf.cols, conf.level_matmul+1, conf.num_heads);

    struct MatrixMatrixProductPrecomputations_GPU precomp_gpu = getMatrixMatrixProductPrecomputations_GPU(
        GPUcc, cc, masks, conf.blockSize, conf.bStep, conf.level_matmul+1, conf.level_matmul+1, conf.prescale, conf.numSlots);

    TransposePrecomputations_GPU Tprecomp_gpu = getMatrixTransposePrecomputations_GPU(GPUcc, cc, conf.blockSize, conf.bStep, conf.level_matmul);

    ct_tokens = encryptMatrixtoCPU(std::string(model_path + "/" + output_file), keys.publicKey, conf.numSlots, conf.blockSize, conf.rows, conf.cols);

    std::string output_path_str = output_path.string(); 
    process_sentences_from_csv_mrpc(path, output_file,
                            model_name, model_path, output_path_str, conf,
                            keys.publicKey, GPUcc, ct_tokens,
                            weights_layer0, weights_layer1, masks,
                            precomp_gpu, Tprecomp_gpu,
                            cc, keys.secretKey, test_case);

	return 0;
}